#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "bit_line_maker.h"

#include <chrono>
#include <cmath>
#include <iostream>
#include <memory>
#include <random>

#include "cuda_math_util.h"
#include "cuda_util.h"
#include "io_iterator.h"
#include <hipcub/hipcub.hpp>

namespace RPU {

#define LASTK32MASK 0xFFFFFFFF >> ((nK32 << 5) - Kplus1)

#define RPU_BLM_DEFINE_NK32                                                                        \
  const int nK32 = (Kplus1 + 31) >> 5;                                                             \
  const int nK32m1 = nK32 - 1;

#define COMMA ,

#define RPU_BLM_SWITCH_TRANS_TEMPLATE(X_TRANS, D_TRANS, OUT_TRANS, KERNEL, ARGS, ADDTEMP)          \
  if (OUT_TRANS) {                                                                                 \
    if (X_TRANS & D_TRANS) {                                                                       \
      KERNEL<T, true, true, true ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                      \
    } else if (!X_TRANS & !D_TRANS) {                                                              \
      KERNEL<T, false, false, true ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                    \
    } else if (!X_TRANS & D_TRANS) {                                                               \
      KERNEL<T, false, true, true ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                     \
    } else {                                                                                       \
      KERNEL<T, true, false, true ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                     \
    }                                                                                              \
  } else {                                                                                         \
    if (X_TRANS & D_TRANS) {                                                                       \
      KERNEL<T, true, true, false ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                     \
    } else if (!X_TRANS & !D_TRANS) {                                                              \
      KERNEL<T, false, false, false ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                   \
    } else if (!X_TRANS & D_TRANS) {                                                               \
      KERNEL<T, false, true, false ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                    \
    } else {                                                                                       \
      KERNEL<T, true, false, false ADDTEMP><<<nblocks, nthreads_, 0, s>>> ARGS;                    \
    }                                                                                              \
  }

#define RPU_BLM_SWITCH_TRANS_TEMPLATE_UM(X_TRANS, D_TRANS, OUT_TRANS, UM, UBLM, KERNEL, ARGS)      \
  if (UM && UBLM) {                                                                                \
    RPU_BLM_SWITCH_TRANS_TEMPLATE(                                                                 \
        X_TRANS, D_TRANS, OUT_TRANS, KERNEL, ARGS, COMMA true COMMA true);                         \
  } else if (UM && !UBLM) {                                                                        \
    RPU_BLM_SWITCH_TRANS_TEMPLATE(                                                                 \
        X_TRANS, D_TRANS, OUT_TRANS, KERNEL, ARGS, COMMA true COMMA false);                        \
  } else if (!UM && !UBLM) {                                                                       \
    RPU_BLM_SWITCH_TRANS_TEMPLATE(                                                                 \
        X_TRANS, D_TRANS, OUT_TRANS, KERNEL, ARGS, COMMA false COMMA false);                       \
  } else {                                                                                         \
    RPU_BLM_SWITCH_TRANS_TEMPLATE(                                                                 \
        X_TRANS, D_TRANS, OUT_TRANS, KERNEL, ARGS, COMMA true COMMA false);                        \
  }

#define RPU_BLM_DEBUG_DEFINE_K                                                                     \
  int Kplus1 = K + 1;                                                                              \
  int nK32 = (Kplus1 + 31) / 32;                                                                   \
  T resolution = 0.01;                                                                             \
  bool sto_round = false;

#define RPU_BLM_DEBUG_DEFINE_K_BATCH                                                               \
  RPU_BLM_DEBUG_DEFINE_K;                                                                          \
  int m_batch = 1;

#define RPU_BLM_DEBUG_INIT(NSTATES)                                                                \
                                                                                                   \
  CudaContext c{-1, false};                                                                        \
  CudaArray<uint32_t> dev_counts(&c, size *nK32);                                                  \
  CudaArray<T> dev_indata(&c, size, indata);                                                       \
                                                                                                   \
  CudaArray<hiprandState> dev_states(&c, NSTATES);                                                  \
  curandSetup(dev_states, size, fake_seed);                                                        \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  hipEvent_t start, stop;                                                                         \
  CUDA_CALL(hipEventCreate(&start));                                                              \
  CUDA_CALL(hipEventCreate(&stop));                                                               \
  CUDA_CALL(hipEventRecord(start, c.getStream()));

#define RPU_BLM_DEBUG_FINISH                                                                       \
  CUDA_CALL(hipEventRecord(stop, c.getStream()));                                                 \
  hipEventSynchronize(stop);                                                                      \
  CUDA_CALL(hipPeekAtLastError());                                                                \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  float milliseconds = 0;                                                                          \
  hipEventElapsedTime(&milliseconds, start, stop);                                                \
                                                                                                   \
  *timing = milliseconds;                                                                          \
  dev_counts.copyTo(counts);                                                                       \
  CUDA_CALL(hipDeviceSynchronize());

#define RPU_BLM_DEBUG_BATCH_INIT(NSTATES, COUNTSTYPE)                                              \
  CudaContext c{-1, false};                                                                        \
  CudaArray<COUNTSTYPE> dev_counts(&c, size *m_batch *nK32);                                       \
  CudaArray<COUNTSTYPE> dev_counts2(&c, size *m_batch *nK32);                                      \
  dev_counts.setConst(0);                                                                          \
  dev_counts2.setConst(0);                                                                         \
  T *tmp = new T[size * m_batch];                                                                  \
  for (int i = 0; i < m_batch; i++) {                                                              \
    for (int j = 0; j < size; j++) {                                                               \
      tmp[i * size + j] = indata[j];                                                               \
    }                                                                                              \
  }                                                                                                \
  CudaArray<T> dev_indata(&c, size *m_batch, tmp);                                                 \
  CudaArray<T> dev_indata2(&c, size *m_batch, tmp);                                                \
                                                                                                   \
  CudaArray<hiprandState> dev_states(&c, NSTATES);                                                  \
  curandSetup(dev_states, size, fake_seed);                                                        \
                                                                                                   \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  hipEvent_t start, stop;                                                                         \
  CUDA_CALL(hipEventCreate(&start));                                                              \
  CUDA_CALL(hipEventCreate(&stop));                                                               \
  CUDA_CALL(hipEventRecord(start, c.getStream()));

#define RPU_BLM_DEBUG_BATCH_FINISH(COUNTT)                                                         \
  CUDA_CALL(hipEventRecord(stop, c.getStream()));                                                 \
  hipEventSynchronize(stop);                                                                      \
  CUDA_CALL(hipPeekAtLastError());                                                                \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  float milliseconds = 0;                                                                          \
  hipEventElapsedTime(&milliseconds, start, stop);                                                \
                                                                                                   \
  *timing = milliseconds;                                                                          \
                                                                                                   \
  COUNTT *tmp32 = new COUNTT[size * m_batch * nK32];                                               \
  dev_counts.copyTo(tmp32);                                                                        \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  int batch_idx = m_batch - 1;                                                                     \
  for (int j = 0; j < size * nK32; j++) {                                                          \
    counts[j] = tmp32[(batch_idx)*size * nK32 + j];                                                \
  }                                                                                                \
                                                                                                   \
  CUDA_CALL(hipDeviceSynchronize());                                                              \
                                                                                                   \
  delete[] tmp;                                                                                    \
  delete[] tmp32;

template <bool trans, bool out_trans, typename count_t>
__device__ __forceinline__ int
getCountsIdx(int idx, int sz, int m_batch, int count_offset, int K = 0, kagg_t Kc = 0, int nB = 0);

template <bool trans> __device__ __forceinline__ int getBatchIdx(int idx, int sz, int m_batch);

template <typename T, bool um>
__device__ __forceinline__ T getScale(const T *scale_values, int batch_idx);

template <bool ublm>
__device__ __forceinline__ int getK(const int *K_values, int batch_idx, int Kplus1);

template <typename T, bool ublm>
__device__ __forceinline__ T getScaleProb(const T scaleprob, const int K, const T lr_div_dwmin);

template <bool ublm, typename count_t>
__device__ __forceinline__ int getnB(const kagg_t *Kn, int m_batch, int Kplus1);

template <bool ublm, typename count_t>
__device__ __forceinline__ int getKc(const kagg_t *Kc_values, int batch_idx, int Kplus1);

template <typename count_t>
__device__ __forceinline__ void getCountsSimpleLoop(
    float value,
    bool negative,
    count_t *c,
    int nK32m1,
    int K,
    hiprandState &local_state,
    int nK32,
    int sz,
    kagg_t Kc);

#define DISCRETIZE_VALUE_STOCH_DEFINITIONS                                                         \
  T res = resolution;                                                                              \
  bool sr = sto_round & (res > 0);                                                                 \
  T stoch_value;

#define DISCRETIZE_VALUE(RES)                                                                      \
  if (RES > 0) {                                                                                   \
    value /= RES;                                                                                  \
    value = RES * RPU_ROUNDFUN(value);                                                             \
  }

#define DISCRETIZE_VALUE_STOCH(STATEVAR)                                                           \
  if (sr)                                                                                          \
    stoch_value = hiprand_uniform(&STATEVAR);                                                       \
                                                                                                   \
  if (res > 0) {                                                                                   \
    value /= res;                                                                                  \
    if (sr)                                                                                        \
      value += stoch_value - 0.5;                                                                  \
    value = res * RPU_ROUNDFUN(value);                                                             \
  }

namespace test_helper {
// helper function for debugging

int getCounts(uint32_t *counts, int i, int K, int size, bool negtest) {
  int icounts = 0;
  int nK32 = (K + 1 + 31) / 32;
  uint32_t one = 1;
  uint32_t negative = counts[i] & one;
  for (int j = 0; j < nK32; j++) {
    uint32_t c = counts[i + j * size];
    for (int l = 0; l < 32; l++) {
      if ((c & (one << l)) != 0) {
        icounts++;
      };
    }
  }

  if (negtest && (negative == 1))
    return -icounts + 1;
  else
    return icounts;
}

template <typename T>
void checkCounts(
    const T *x_input,
    int x_size,
    const T *d_input,
    int d_size,
    int BL,
    T A,
    T B,
    CudaArray<uint32_t> *dev_x_counts,
    CudaArray<uint32_t> *dev_d_counts) {
  T *host_x_input = new T[x_size];
  T *host_d_input = new T[d_size];

  CUDA_CALL(hipMemcpy(host_d_input, d_input, d_size * sizeof(T), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(host_x_input, x_input, x_size * sizeof(T), hipMemcpyDeviceToHost));

  uint32_t *x_counts = new uint32_t[dev_x_counts->getSize()];
  uint32_t *d_counts = new uint32_t[dev_d_counts->getSize()];

  dev_d_counts->copyTo(d_counts);
  dev_x_counts->copyTo(x_counts);
  int dzero = 0;
  int xzero = 0;

  for (int i = 0; i < d_size; i++) {
    int d = getCounts(d_counts, i, BL, d_size, true);
    if (i < 100)
      std::cout << "D[" << i << "]: " << d << " (" << (T)d / BL << ")"
                << " in: " << host_d_input[i] * A << std::endl;
    if (fabs(d) == 0)
      dzero++;
  }

  for (int i = 0; i < x_size; i++) {
    int x = getCounts(x_counts, i, BL, x_size, true);

    if (fabs(x) == 0)
      xzero++;

    if (i < 100)
      std::cout << "X[" << i << "]: " << x << " (" << (T)x / BL << ")"
                << " in: " << host_x_input[i] * B << std::endl;
  }

  delete[] x_counts;
  delete[] d_counts;
  delete[] host_x_input;
  delete[] host_d_input;
}
} // namespace test_helper

/******************************************************************************
 * COUNT KERNELS *
 ******************************************************************************/

// *********************************************************************************
// kernelUpdateGetCounts_Linear
template <typename T, int ITEMS_PER_THREAD, typename InputIterator>
__global__ void kernelUpdateGetCounts_Linear(
    InputIterator source_value,
    int size_in,
    T scaleprob,
    uint32_t *counts,
    int Kplus1,
    hiprandState *random_states,
    T resolution,
    bool sto_round) {
  // call <<
  // size*Kplus1/ITEMS_PER_THREAD/RPU_THREADS_PER_BLOCK_UPDATE,RPU_THREADS_PER_BLOCK_UPDATE>>

  // 2) assume that Kplus1<= 32 !!
  // 3) assume that ITEM_PER_THREAD is power of 2
  // 4) NEEDS Kplus1/ITEM_PERT_PER_THREAD*size random states !
  // 5) assume ITEMS_PER_THREAD <= Kplus1 and mod(Kplus1,ITEMS_PER_THREAD)==0 !!

  const int Kp1 = Kplus1;
  const int size = size_in;
  const int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int nKthreads = (Kp1 / ITEMS_PER_THREAD);
  const int sourceId = tid / nKthreads;
  const int kidsub = (tid * ITEMS_PER_THREAD) % Kp1;
  const int kidthread = tid % nKthreads;
  const uint32_t one = 1;

  if (tid >= (size * nKthreads))
    return;

  DISCRETIZE_VALUE_STOCH_DEFINITIONS;

  hiprandState local_state;
  T value;
  bool negative;

  if (sourceId < size) {

    value = source_value[sourceId]; // not memory optimized but good  for K=32 (broadcasted)
    local_state = random_states[tid];

    // input management
    negative = value < 0;
    value = (negative) ? -value : value;

    value *= scaleprob;

    if (kidsub == 0) {      // only once per K
      counts[sourceId] = 0; // need to set zero (all Kthreads need to be inside a warp!)

      DISCRETIZE_VALUE_STOCH(local_state);
    }
    // need to broadcast value to within K threads
    value = __shfl_up_sync(0xFFFFFFFF, value, kidthread);

    uint32_t bitwise = 0;
#pragma unroll
    for (int i = 0; i < ITEMS_PER_THREAD; i++) {
      stoch_value = hiprand_uniform(&local_state);
      if (stoch_value < value) {
        bitwise |= (1 << (kidsub + i));
      }
    }
    random_states[tid] = local_state;

    if (kidsub == 0) {
      bitwise = (negative) ? (bitwise | one) : (bitwise & ~one);
    }
    if (bitwise > 0)
      atomicOr(&counts[sourceId], bitwise);
  }
}

namespace test_helper {

template <typename T, int ITEMS_PER_THREAD>
int debugKernelUpdateGetCounts_Linear(
    T *indata, int size, T scaleprob, uint32_t *counts, int K, T *timing, bool fake_seed) {
  // counts should be: size*nk32;
  RPU_BLM_DEBUG_DEFINE_K;

  if ((nK32 > 1) || ((Kplus1 % ITEMS_PER_THREAD) != 0) || (Kplus1 < ITEMS_PER_THREAD) ||
      (32 % Kplus1 != 0)) {
    std::cerr << "Kplus1: ITEM_PER_THREAD combination not supported. ";
    return 1;
  }
  int n = size * (Kplus1 / ITEMS_PER_THREAD);
  dim3 nthreads = dim3(RPU_THREADS_PER_BLOCK_UPDATE);
  int nblocks = (n + RPU_THREADS_PER_BLOCK_UPDATE - 1) / RPU_THREADS_PER_BLOCK_UPDATE;

  RPU_BLM_DEBUG_INIT(n);

  kernelUpdateGetCounts_Linear<T, ITEMS_PER_THREAD, const T *>
      <<<nblocks, nthreads, 0, c.getStream()>>>(
          dev_indata.getData(), size, scaleprob, dev_counts.getData(), Kplus1, dev_states.getData(),
          resolution, sto_round);

  RPU_BLM_DEBUG_FINISH;
  return 0;
}

template int
debugKernelUpdateGetCounts_Linear<float, 1>(float *, int, float, uint32_t *, int, float *, bool);
template int
debugKernelUpdateGetCounts_Linear<float, 2>(float *, int, float, uint32_t *, int, float *, bool);
template int
debugKernelUpdateGetCounts_Linear<float, 4>(float *, int, float, uint32_t *, int, float *, bool);
template int
debugKernelUpdateGetCounts_Linear<float, 8>(float *, int, float, uint32_t *, int, float *, bool);
template int
debugKernelUpdateGetCounts_Linear<float, 16>(float *, int, float, uint32_t *, int, float *, bool);
template int
debugKernelUpdateGetCounts_Linear<float, 32>(float *, int, float, uint32_t *, int, float *, bool);

#ifdef RPU_USE_DOUBLE
template int debugKernelUpdateGetCounts_Linear<double, 1>(
    double *, int, double, uint32_t *, int, double *, bool);
template int debugKernelUpdateGetCounts_Linear<double, 2>(
    double *, int, double, uint32_t *, int, double *, bool);
template int debugKernelUpdateGetCounts_Linear<double, 4>(
    double *, int, double, uint32_t *, int, double *, bool);
template int debugKernelUpdateGetCounts_Linear<double, 8>(
    double *, int, double, uint32_t *, int, double *, bool);
template int debugKernelUpdateGetCounts_Linear<double, 16>(
    double *, int, double, uint32_t *, int, double *, bool);
template int debugKernelUpdateGetCounts_Linear<double, 32>(
    double *, int, double, uint32_t *, int, double *, bool);
#endif
} // namespace test_helper

// *********************************************************************************
// kernelUpdateGetCountsBatch_Loop2

#define GET_COUNTS_INNER_LOOP(SCALEPROB)                                                           \
                                                                                                   \
  negative = value < 0;                                                                            \
  value = (negative) ? -value : value;                                                             \
                                                                                                   \
  value *= SCALEPROB;                                                                              \
                                                                                                   \
  if (laneId == 0) {                                                                               \
    DISCRETIZE_VALUE_STOCH(local_state);                                                           \
  }                                                                                                \
  value = __shfl_sync(0xFFFFFFFF, value, 0);                                                       \
                                                                                                   \
  int isize = 0;                                                                                   \
                                                                                                   \
  PRAGMA(unroll)                                                                                   \
  for (int i = 0; i < nK32; i++) {                                                                 \
                                                                                                   \
    stoch_value = hiprand_uniform(&local_state);                                                    \
                                                                                                   \
    ballot = __ballot_sync(0xFFFFFFFF, stoch_value < value);                                       \
                                                                                                   \
    if (laneId == 0) {                                                                             \
      if (i == 0) {                                                                                \
        ballot = (negative) ? (ballot | one) : (ballot & ~one);                                    \
      }                                                                                            \
                                                                                                   \
      if (i == nK32m1) {                                                                           \
        ballot = ballot & lastK32mask;                                                             \
      }                                                                                            \
                                                                                                   \
      *(c + isize) = ballot;                                                                       \
      isize += sz;                                                                                 \
    }                                                                                              \
  }

#define GET_COUNTS_LOOP(PROB, SIZE, COUNTS, SCALEPROB)                                             \
  sz = SIZE;                                                                                       \
  if (sourceId < sz) {                                                                             \
    value = PROB[sourceId];                                                                        \
                                                                                                   \
    c = &COUNTS[sourceId];                                                                         \
                                                                                                   \
    GET_COUNTS_INNER_LOOP(SCALEPROB);                                                              \
  }

#define GET_COUNTS_LOOP_BATCH(PROB, SIZE, COUNTS, SCALEPROB, TRANS, OUTTRANS)                      \
  {                                                                                                \
    sz = SIZE;                                                                                     \
    int counts_offset = nK32 * sz;                                                                 \
    int n = m_batch * sz;                                                                          \
    int n32 = n << 5;                                                                              \
    for (int i_stride = 0; i_stride < n32; i_stride += total_threads) {                            \
      int sourceId = (tid + i_stride) >> 5;                                                        \
      if (sourceId < n) {                                                                          \
        value = PROB[sourceId];                                                                    \
        c = &COUNTS[getCountsIdx<TRANS, OUTTRANS, uint32_t>(                                       \
            sourceId, sz, m_batch, counts_offset)];                                                \
        GET_COUNTS_INNER_LOOP(SCALEPROB);                                                          \
      }                                                                                            \
    }                                                                                              \
  }

template <
    typename T,
    bool x_trans,
    bool d_trans,
    bool out_trans,
    typename XInputIteratorT,
    typename DInputIteratorT>
__global__ void kernelUpdateGetCountsBatch_Loop2(
    XInputIteratorT x_prob,
    int x_size_in,
    T x_scaleprob,
    uint32_t *x_counts,
    DInputIteratorT d_prob,
    int d_size_in,
    T d_scaleprob,
    uint32_t *d_counts,
    int Kplus1,
    int m_batch_in,
    hiprandState *random_states,
    T resolution,
    bool sto_round) {
  // call << (size (of states)/numwarpsperblock,1),warpSize*numwarpsperblock >>

  // -- let each warp compute 32 K values
  // -- no limit for K , however BLocked design might be better for larger K

  const int tid = blockDim.x * blockIdx.x + threadIdx.x; // can be larger that x_size or d_size
  const int total_threads = blockDim.x * gridDim.x;
  const int x_size = x_size_in;
  const int d_size = d_size_in;
  const int m_batch = m_batch_in;

  const int max_size = (((d_size > x_size) ? d_size : x_size) * m_batch) << 5;

  if (tid >= max_size)
    return;

  hiprandState local_state;
  local_state = random_states[tid];

  RPU_BLM_DEFINE_NK32;
  const uint32_t lastK32mask = LASTK32MASK;
  const uint32_t one = 1;

  const int laneId = threadIdx.x & 0x1f;

  DISCRETIZE_VALUE_STOCH_DEFINITIONS;

  uint32_t ballot = 0;

  T value;
  uint32_t *c;
  bool negative;
  int sz;

  // NOTE: need to re-order in update from SIZE*nK32 format, when it is trans!

  // x input
  GET_COUNTS_LOOP_BATCH(x_prob, x_size, x_counts, x_scaleprob, x_trans, out_trans);

  // d input
  GET_COUNTS_LOOP_BATCH(d_prob, d_size, d_counts, d_scaleprob, d_trans, out_trans);

  // save new random states
  random_states[tid] = local_state;
}

namespace test_helper {
template <typename T>
int debugKernelUpdateGetCountsBatch_Loop2(
    T *indata, int size, T scaleprob, uint32_t *counts, int K, T *timing, bool fake_seed) {
  // counts should be: size*nk32 allocated !
  RPU_BLM_DEBUG_DEFINE_K_BATCH;

  int nthreads = RPU_THREADS_PER_BLOCK_UPDATE;
  int numwarpsperblock = RPU_THREADS_PER_BLOCK_UPDATE / 32;

  int n_items = 12;
  int m =
      MIN((size * m_batch + n_items - 1) / n_items, numwarpsperblock * 8); // stripped per thread
  int nblocks = (m + numwarpsperblock - 1) / numwarpsperblock;
  int n = m * 32;
  RPU_BLM_DEBUG_BATCH_INIT(n, uint32_t);

  kernelUpdateGetCountsBatch_Loop2<T, false, false, false><<<nblocks, nthreads, 0, c.getStream()>>>(
      dev_indata.getData(), size, scaleprob, dev_counts.getData(), dev_indata2.getData(), size,
      scaleprob, dev_counts2.getData(), Kplus1, m_batch, dev_states.getData(), resolution,
      sto_round);

  RPU_BLM_DEBUG_BATCH_FINISH(uint32_t);
  return 0;
}
template int debugKernelUpdateGetCountsBatch_Loop2<float>(
    float *, int, float, unsigned int *, int, float *, bool);
#ifdef RPU_USE_DOUBLE
template int debugKernelUpdateGetCountsBatch_Loop2<double>(
    double *, int, double, unsigned int *, int, double *, bool);
#endif

} // namespace test_helper

// *********************************************************************************
// kernelUpdateGetCountsBatch_SimpleLoop2

template <>
__device__ __forceinline__ int getCountsIdx<false, false, uint32_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {

  return idx / sz * counts_offset + (idx % sz);
}

template <>
__device__ __forceinline__ int getCountsIdx<false, true, uint32_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  // batchidx = idx/sz;
  // x_idx = idx % sz
  int transposed_idx = idx / sz + m_batch * (idx % sz);
  return (transposed_idx / sz) * counts_offset + (transposed_idx % sz);
}

template <>
__device__ __forceinline__ int getCountsIdx<true, false, uint32_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  // batchidx * count_offset  + i_x
  return (idx % m_batch) * counts_offset + (idx / m_batch);
}

template <>
__device__ __forceinline__ int getCountsIdx<true, true, uint32_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  // idx already transposed. leave it
  return idx / sz * counts_offset + (idx % sz);
}

template <>
__device__ __forceinline__ int getCountsIdx<true, true, uint64_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  int iB = Kc >> 5;           // start word [new batch idx]
  int xd_idx = idx / m_batch; // NOTE: nK32==1 REQUIRED (and not checked!!)
  return xd_idx * nB + iB;
}
template <>
__device__ __forceinline__ int getCountsIdx<false, true, uint64_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  int iB = Kc >> 5;      // start word [new batch idx]
  int xd_idx = idx % sz; // NOTE: nK32==1 REQUIRED (and not checked!!)
  return xd_idx * nB + iB;
}

template <>
__device__ __forceinline__ int getCountsIdx<true, false, uint64_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  printf("ERROR. Not implemented\n");
  return 0;
}
template <>
__device__ __forceinline__ int getCountsIdx<false, false, uint64_t>(
    int idx, int sz, int m_batch, int counts_offset, int K, kagg_t Kc, int nB) {
  printf("ERROR. Not implemented\n");
  return 0;
}

template <> __device__ __forceinline__ int getBatchIdx<true>(int idx, int sz, int m_batch) {
  return idx % m_batch;
}

template <> __device__ __forceinline__ int getBatchIdx<false>(int idx, int sz, int m_batch) {
  return idx / sz;
}

template <>
__device__ __forceinline__ float getScale<float, true>(const float *scale_values, int batch_idx) {
  // UM + inverse (for B (that is x))
  return scale_values[batch_idx];
}

template <>
__device__ __forceinline__ float getScale<float, false>(const float *scale_values, int batch_idx) {
  return 1.0;
}

#ifdef RPU_USE_DOUBLE
template <>
__device__ __forceinline__ double
getScale<double, true>(const double *scale_values, int batch_idx) {
  // UM + inverse (for B (that is x))
  return scale_values[batch_idx];
}

template <>
__device__ __forceinline__ double
getScale<double, false>(const double *scale_values, int batch_idx) {
  return 1.0;
}
#endif

template <> // UBLM
__device__ __forceinline__ int getK<true>(const int *K_values, int batch_idx, int Kplus1) {
  return K_values[batch_idx];
}

template <> // UBLM
__device__ __forceinline__ int getK<false>(const int *K_values, int batch_idx, int Kplus1) {
  return Kplus1 - 1;
}

template <> // UBLM
__device__ __forceinline__ int
getKc<true, uint64_t>(const kagg_t *Kc_values, int batch_idx, int Kplus1) {
  return Kc_values[batch_idx];
}

template <> // UBLM
__device__ __forceinline__ int
getKc<false, uint64_t>(const kagg_t *Kc_values, int batch_idx, int Kplus1) {
  return batch_idx * (Kplus1 - 1);
}

template <> // UBLM
__device__ __forceinline__ int
getKc<true, uint32_t>(const kagg_t *Kc_values, int batch_idx, int Kplus1) {
  return 0; // dummy
}

template <> // UBLM
__device__ __forceinline__ int
getKc<false, uint32_t>(const kagg_t *Kc_values, int batch_idx, int Kplus1) {
  return 0; // dummy
}

template <> // UBLM
__device__ __forceinline__ int getnB<true, uint64_t>(const kagg_t *Kn, int m_batch, int Kplus1) {
  return ((*Kn) + 31) >> 5;
}

template <> // UBLM
__device__ __forceinline__ int getnB<false, uint64_t>(const kagg_t *Kn, int m_batch, int Kplus1) {
  return (m_batch * (Kplus1 - 1) + 31) >> 5;
}

template <> // UBLM
__device__ __forceinline__ int getnB<false, uint32_t>(const kagg_t *Kn, int m_batch, int Kplus1) {
  return 1; // dummy
}
template <> // UBLM
__device__ __forceinline__ int getnB<true, uint32_t>(const kagg_t *Kn, int m_batch, int Kplus1) {
  return 1; // dummy
}

template <>
__device__ __forceinline__ float
getScaleProb<float, true>(const float scaleprob, const int K, const float lr_div_dwmin) {
  return sqrt(lr_div_dwmin / K);
};

// UBLM
template <>
__device__ __forceinline__ float
getScaleProb<float, false>(const float scaleprob, const int K, const float lr_div_dwmin) {
  return scaleprob;
};

#ifdef RPU_USE_DOUBLE
template <>
__device__ __forceinline__ double
getScaleProb<double, true>(const double scaleprob, const int K, const double lr_div_dwmin) {
  return sqrt(lr_div_dwmin / K);
};

template <>
__device__ __forceinline__ double
getScaleProb<double, false>(const double scaleprob, const int K, const double lr_div_dwmin) {
  return scaleprob;
};

#endif

template <>
__device__ __forceinline__ void getCountsSimpleLoop<uint32_t>(
    float value,
    bool negative,
    uint32_t *c,
    int nK32m1,
    int K,
    hiprandState &local_state,
    int nK32,
    int sz,
    kagg_t Kc) {

  uint32_t ballot = (negative) ? 1 : 0;
  int nK32m1_local = MIN(K >> 5, nK32m1);
  int nn = (nK32m1_local > 0) ? 31 : K;
  PRAGMA(unroll)
  for (int j = 1; j <= nn; j++) {
    float stoch_value = hiprand_uniform(&local_state);
    ballot |= (stoch_value < value) ? (((uint32_t)1) << j) : (uint32_t)0;
  }
  *c = ballot;
  if (nK32 > 1) {
    ballot = 0;
    int offset = 0;
    PRAGMA(unroll)
    for (int i = 1; i < nK32; i++) {
      offset += sz;
      if (i > nK32m1_local) {
        *(c + offset) = 0;
      } else {
        ballot = 0;
        nn = (i == nK32m1_local) ? (K & 0x1f) : 31;
        PRAGMA(unroll)
        for (int j = 0; j <= nn; j++) {
          float stoch_value = hiprand_uniform(&local_state);
          ballot |= (stoch_value < value) ? (((uint32_t)1) << j) : (uint32_t)0;
        }
        *(c + offset) = ballot;
      }
    }
  }
}

template <> // count_t
__device__ __forceinline__ void getCountsSimpleLoop<uint64_t>(
    float value,
    bool negative,
    uint64_t *c,
    int nK32m1,
    int K,
    hiprandState &local_state,
    int nK32,
    int sz,
    kagg_t Kc) {
  static_assert(sizeof(uint64_t) == sizeof(unsigned long long int), "uint64 issue");

  // nK32m1 NEEDS TO BE 0 (otherwise not supported)
  // needs OUTTRANS!!
  int bit_pos_start = Kc & 0x1f;

  uint32_t ballot = 0;
  uint32_t neg_word = (negative) ? (0xffffffff >> (32 - K)) : 0;

  PRAGMA(unroll)
  for (int j = 0; j < K; j++) { // start from zero (no negative bit)
    float stoch_value = hiprand_uniform(&local_state);
    ballot |= (stoch_value < value) ? (((uint32_t)1) << j) : (uint32_t)0;
  }

  uint64_t ballot64 = (uint64_t)(ballot << bit_pos_start); // may overflow upper bits
  ballot64 |= (((uint64_t)(neg_word << bit_pos_start)) << 32);
  atomicOr((unsigned long long int *)c, (unsigned long long int)ballot64); // save

  if (bit_pos_start + K > 32) {
    // in case of overflow
    ballot64 = (ballot >> (32 - bit_pos_start));
    ballot64 |= (((uint64_t)(neg_word >> (32 - bit_pos_start))) << 32);

    atomicOr((unsigned long long int *)(c + 1), (unsigned long long int)ballot64); // save
  }
}

#define GET_COUNTS_SIMPLE_LOOP_BATCH(                                                              \
    PROB, SIZE, COUNTS, SCALEPROB, TRANS, OUTTRANS, SPROPOP, TIDSTART, TIDEND, TIDN)               \
  {                                                                                                \
    if ((tid >= TIDSTART) && (tid < TIDEND)) {                                                     \
      int sz = SIZE;                                                                               \
      int counts_offset = nK32 * sz;                                                               \
      int n = m_batch * sz;                                                                        \
                                                                                                   \
      for (int i_stride = 0; i_stride < n; i_stride += TIDN) {                                     \
                                                                                                   \
        int idx = (tid - TIDSTART + i_stride);                                                     \
        if (idx < n) {                                                                             \
          T value = PROB[idx];                                                                     \
          int batch_idx = getBatchIdx<TRANS>(idx, sz, m_batch);                                    \
          int K = getK<update_bl_management>(K_values, batch_idx, Kplus1);                         \
          if ((K == 0) || (value == 0)) {                                                          \
            continue;                                                                              \
          }                                                                                        \
          kagg_t Kc = getKc<update_bl_management, count_t>(Kc_values, batch_idx, Kplus1);          \
          T scaleprob = getScaleProb<T, update_bl_management>(SCALEPROB, K, lr_div_dwmin);         \
          T scale = getScale<T, update_management>(scale_values, batch_idx);                       \
          count_t *c = &COUNTS[getCountsIdx<TRANS, OUTTRANS, count_t>(                             \
              idx, sz, m_batch, counts_offset, K, Kc, nB)];                                        \
          T sprob = scaleprob SPROPOP scale;                                                       \
          bool negative = value < 0;                                                               \
          value = (negative) ? -value : value;                                                     \
          value *= sprob;                                                                          \
          DISCRETIZE_VALUE_STOCH(local_state);                                                     \
                                                                                                   \
          getCountsSimpleLoop<count_t>(value, negative, c, nK32m1, K, local_state, nK32, sz, Kc);  \
        }                                                                                          \
      }                                                                                            \
    }                                                                                              \
  }

template <
    typename T,
    bool x_trans,
    bool d_trans,
    bool out_trans,
    bool update_management,
    bool update_bl_management,
    typename count_t,
    typename XInputIteratorT,
    typename DInputIteratorT>
__global__ void kernelUpdateGetCountsBatch_SimpleLoop2(
    XInputIteratorT x_prob,
    int x_size_in,
    T x_scaleprob_in,
    count_t *x_counts,
    DInputIteratorT d_prob,
    int d_size_in,
    T d_scaleprob_in,
    count_t *d_counts,
    int Kplus1_in,
    int m_batch_in,
    hiprandState *random_states,
    T resolution,
    bool sto_round,
    const T *scale_values = nullptr,
    const int *K_values = nullptr,
    const T lr_div_dwmin_in = 1.0,
    const kagg_t *Kc_values = nullptr,
    const kagg_t *Kn = nullptr)

{
  // -- each thread computes all the  K values
  // -- no limit for number of threads. However, occupy all stream processors once should reduce
  // overhead
  // -- for UM: scale values should be sqrt(amax_x/amax_d) for D
  // -- for UM: scale values has to be strictly POSITIVE (NON-zero)!!
  // -- ASSUMES: NGRID>1 !! (nblocks>1)
  // -- RANDOMSTATES need to have 1 for each tid.
  // -- CAUTION: counts should be set to zero!!!
  //
  // In the case of uint64_t:
  // -- using atomics to save the counts. CAUTION: counts should be set to zero!!!
  // -- ONLY for out_trans=true
  // -- only K <= 31 supported (nK32==1)
  // -- CAUTION: some bit boundardy issue? Very seldom 64 version seems one bit off.. ignore. Has no
  // relevance with noise
  //             could be just a rounding issue

  const int tid = blockDim.x * blockIdx.x + threadIdx.x; // can be larger that x_size or d_size
  const int x_size = x_size_in;
  const int d_size = d_size_in;
  const int m_batch = m_batch_in;
  const int Kplus1 = Kplus1_in;
  const int nB = getnB<update_bl_management, count_t>(Kn, m_batch, Kplus1);

  const T x_scaleprob = x_scaleprob_in;
  const T d_scaleprob = d_scaleprob_in;

  int nx_blocks = ceil(gridDim.x * ((T)x_size / (T)(x_size + d_size)));
  int nd_blocks = gridDim.x - nx_blocks;
  if ((nd_blocks <= 0) && (d_size > 0)) {
    nx_blocks = gridDim.x - 1; // ASSUMES gridDim.x>1 !~
    nd_blocks = 1;
  }
  const int tid_nx = nx_blocks * blockDim.x;
  const int tid_nd = nd_blocks * blockDim.x;

  if ((tid < tid_nx) && (tid > x_size * m_batch))
    return;
  if ((tid >= tid_nx) && (tid - tid_nx > d_size * m_batch))
    return;

  const T lr_div_dwmin = lr_div_dwmin_in;

  hiprandState local_state = random_states[tid];
  RPU_BLM_DEFINE_NK32;

  DISCRETIZE_VALUE_STOCH_DEFINITIONS;

  // x input
  GET_COUNTS_SIMPLE_LOOP_BATCH(
      x_prob, x_size, x_counts, x_scaleprob, x_trans, out_trans, /, 0, tid_nx, tid_nx);

  // d input
  GET_COUNTS_SIMPLE_LOOP_BATCH(
      d_prob, d_size, d_counts, d_scaleprob, d_trans, out_trans, *, tid_nx, tid_nx + tid_nd,
      tid_nd);

  // save new random states
  random_states[tid] = local_state;
}

namespace test_helper {
template <typename T>
int debugKernelUpdateGetCountsBatch_SimpleLoop2(
    T *indata, int size, T scaleprob, uint32_t *counts, int K, T *timing, bool fake_seed) {
  // counts should be: size*nk32 allocated !
  RPU_BLM_DEBUG_DEFINE_K_BATCH;

  int nthreads = RPU_THREADS_PER_BLOCK_UPDATE;

  int m = MIN(size * m_batch, nthreads * 12);
  int nblocks = MAX((m + nthreads - 1) / nthreads, 2);
  std::cout << "nblocks, nthreads: " << nblocks << ", " << nthreads << std::endl;
  int n = m;
  RPU_BLM_DEBUG_BATCH_INIT(n, uint32_t);

  kernelUpdateGetCountsBatch_SimpleLoop2<T, false, false, false, false, false>
      <<<nblocks, nthreads, 0, c.getStream()>>>(
          dev_indata.getData(), size, scaleprob, dev_counts.getData(), dev_indata2.getData(), size,
          scaleprob, dev_counts2.getData(), Kplus1, m_batch, dev_states.getData(), resolution,
          sto_round);

  RPU_BLM_DEBUG_BATCH_FINISH(uint32_t);
  return 0;
}
template int debugKernelUpdateGetCountsBatch_SimpleLoop2<float>(
    float *, int, float, unsigned int *, int, float *, bool);
#ifdef RPU_USE_DOUBLE
template int debugKernelUpdateGetCountsBatch_SimpleLoop2<double>(
    double *, int, double, unsigned int *, int, double *, bool);
#endif
} // namespace test_helper

// *********************************************************************************
// kernelUpdateGetCounts_Loop2
template <typename T, typename XInputIteratorT, typename DInputIteratorT>
__global__ void kernelUpdateGetCounts_Loop2(
    XInputIteratorT x_prob,
    int x_size_in,
    T x_scaleprob,
    uint32_t *x_counts,
    DInputIteratorT d_prob,
    int d_size_in,
    T d_scaleprob,
    uint32_t *d_counts,
    int Kplus1,
    hiprandState *random_states,
    T resolution,
    bool sto_round) {
  // call << (size/numwarpsperblock,1),warpSize*numwarpsperblock >>

  // -- let each warp compute 32 K values
  // -- no limit for K , however BLocked design might be better for larger K

  volatile int tid = blockDim.x * blockIdx.x + threadIdx.x;
  const int x_size = x_size_in;
  const int d_size = d_size_in;

  const int max_size = ((x_size > d_size) ? x_size : d_size) << 5;

  if (tid >= max_size)
    return;

  hiprandState local_state = random_states[tid];

  RPU_BLM_DEFINE_NK32;
  const uint32_t one = 1;
  const uint32_t lastK32mask = LASTK32MASK;

  const int laneId = threadIdx.x & 0x1f;
  // const uint32_t sourceId =  blockIdx.x*warps_per_block + warpId;
  const int sourceId = blockIdx.x * (blockDim.x >> 5) + (threadIdx.x >> 5);

  DISCRETIZE_VALUE_STOCH_DEFINITIONS;

  uint32_t ballot = 0;

  T value;
  uint32_t *c;
  bool negative;
  int sz;

  // x input
  GET_COUNTS_LOOP(x_prob, x_size, x_counts, x_scaleprob);

  // d input
  GET_COUNTS_LOOP(d_prob, d_size, d_counts, d_scaleprob);

  // save new random states
  random_states[tid] = local_state;
}

namespace test_helper {
template <typename T>
int debugKernelUpdateGetCounts_Loop2(
    T *indata, int size, T scaleprob, uint32_t *counts, int K, T *timing, bool fake_seed) {
  // counts should be: size*nk32 allocated !
  RPU_BLM_DEBUG_DEFINE_K;

  dim3 nthreads = dim3(RPU_THREADS_PER_BLOCK_UPDATE);
  int numwarpsperblock = RPU_THREADS_PER_BLOCK_UPDATE / 32;
  dim3 nblocks = dim3((size + numwarpsperblock - 1) / numwarpsperblock);

  int n = size * 32;

  RPU_BLM_DEBUG_INIT(n);

  kernelUpdateGetCounts_Loop2<<<nblocks, nthreads, 0, c.getStream()>>>(
      dev_indata.getData(), size, scaleprob, dev_counts.getData(), dev_indata.getData(), 0,
      scaleprob, dev_counts.getData(), Kplus1, dev_states.getData(), resolution, sto_round);

  RPU_BLM_DEBUG_FINISH;
  return 0;
}
template int
debugKernelUpdateGetCounts_Loop2<float>(float *, int, float, unsigned int *, int, float *, bool);
#ifdef RPU_USE_DOUBLE
template int debugKernelUpdateGetCounts_Loop2<double>(
    double *, int, double, unsigned int *, int, double *, bool);
#endif

} // namespace test_helper

/****************************************************************************************************************/
/* BITLINEMAKER */
/******************************************************************************************************************/

#define RPU_BLM_ITEMS_PER_THREAD 4
#define RPU_BLM_BL_TO_SELECT_SIMPLE_LOOP 0
#define RPU_BLM_BLOCKS_PER_SM 2

template <typename T>
BitLineMaker<T>::BitLineMaker(CudaContext *c, int x_size, int d_size)
    : context_{c}, x_size_{x_size}, d_size_{d_size}, umh_{nullptr}, buffer_m_batch_{0} {
  max_block_count_ = context_->getSMCount() * RPU_BLM_BLOCKS_PER_SM;
  nthreads_ = RPU_THREADS_PER_BLOCK_UPDATE;
}

template <typename T>
BLMOutputFormat BitLineMaker<T>::getFormat(int use_bo64, bool implicit_pulses) {

  if (implicit_pulses && use_bo64 == 0) {
    return BLMOutputFormat::FP;
  } else if (use_bo64 == 1 && !implicit_pulses) {
    return BLMOutputFormat::BO64;
  } else if (use_bo64 == 2 && !implicit_pulses) {
    return BLMOutputFormat::UI32BO64;
  } else if (use_bo64 == 0 && !implicit_pulses) {
    return BLMOutputFormat::UI32;
  } else {
    RPU_FATAL("Not able to determine BLM output format");
  }
}

template <typename T> T *BitLineMaker<T>::getXData() const {
  return format_ == BLMOutputFormat::FP ? dev_x_->getData() : nullptr;
};

template <typename T> T *BitLineMaker<T>::getDData() const {
  return format_ == BLMOutputFormat::FP ? dev_d_->getData() : nullptr;
};

template <typename T> uint32_t *BitLineMaker<T>::getXCountsData() const {
  return (format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64)
             ? dev_x_counts_->getData()
             : nullptr;
};

template <typename T> uint32_t *BitLineMaker<T>::getDCountsData() const {
  return (format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64)
             ? dev_d_counts_->getData()
             : nullptr;
};

template <typename T> uint64_t *BitLineMaker<T>::getXCountsBo64Data() const {
  return (format_ == BLMOutputFormat::BO64 || format_ == BLMOutputFormat::UI32BO64)
             ? dev_x_counts_bo64_->getData()
             : nullptr;
};

template <typename T> uint64_t *BitLineMaker<T>::getDCountsBo64Data() const {
  return (format_ == BLMOutputFormat::BO64 || format_ == BLMOutputFormat::UI32BO64)
             ? dev_d_counts_bo64_->getData()
             : nullptr;
};

template <typename T> kagg_t *BitLineMaker<T>::getKnData(bool ublm) const {
  return umh_->getKnData(ublm);
};

template <typename T> int BitLineMaker<T>::getBo64Batch(int m_batch) const {
  return umh_->getBo64Batch(m_batch, current_BL_);
};

template <typename T> void BitLineMaker<T>::copyXCountsToHost(uint32_t *dest) const {
  if (!(format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64)) {
    RPU_FATAL("Wrong format!");
  }
  dev_x_counts_->copyTo(dest);
};

template <typename T> void BitLineMaker<T>::copyDCountsToHost(uint32_t *dest) const {
  if (!(format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64)) {
    RPU_FATAL("Wrong format!");
  }
  dev_d_counts_->copyTo(dest);
};

template <typename T> void BitLineMaker<T>::copyXCountsBo64ToHost(uint64_t *dest) const {
  if (!(format_ == BLMOutputFormat::BO64 || format_ == BLMOutputFormat::UI32BO64)) {
    RPU_FATAL("Wrong format!");
  }
  dev_x_counts_bo64_->copyTo(dest);
};

template <typename T> void BitLineMaker<T>::copyDCountsBo64ToHost(uint64_t *dest) const {
  if (!(format_ == BLMOutputFormat::BO64 || format_ == BLMOutputFormat::UI32BO64)) {
    RPU_FATAL("Wrong format!");
  }
  dev_d_counts_bo64_->copyTo(dest);
};

template <typename T> void BitLineMaker<T>::initializeBLBuffers(int m_batch, int BL, int use_bo64) {

  buffer_m_batch_ = m_batch;
  buffer_BL_ = BL;
  format_ = getFormat(use_bo64);

  if (format_ == BLMOutputFormat::FP) {
    dev_d_ = RPU::make_unique<CudaArray<T>>(context_, d_size_ * m_batch);
    dev_x_ = RPU::make_unique<CudaArray<T>>(context_, x_size_ * m_batch);

  } else {
    int nK32 = BL / 32 + 1; // equivalent to ((BL+1) + 31)/32
    if (format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64) {
      dev_d_counts_ = RPU::make_unique<CudaArray<uint32_t>>(context_, d_size_ * (nK32)*m_batch);
      dev_x_counts_ = RPU::make_unique<CudaArray<uint32_t>>(context_, x_size_ * (nK32)*m_batch);
    }

    if (format_ == BLMOutputFormat::BO64 || format_ == BLMOutputFormat::UI32BO64) {
      if (nK32 > 1) {
        RPU_FATAL("BL>31 is not supported for BO64");
      }
      dev_d_counts_bo64_ = RPU::make_unique<CudaArray<uint64_t>>(context_, d_size_ * m_batch);
      dev_x_counts_bo64_ = RPU::make_unique<CudaArray<uint64_t>>(context_, x_size_ * m_batch);
    }
  }
  context_->synchronize();
  // std::cout << "BLM init BL buffers with batch " << m_batch << " and BL " << BL << ".\n";
}

template <typename T> void BitLineMaker<T>::getCountsDebug(uint32_t *x_counts, uint32_t *d_counts) {

  if (!(format_ == BLMOutputFormat::UI32 || format_ == BLMOutputFormat::UI32BO64)) {
    RPU_FATAL("Wrong format output requested!");
  }

  dev_x_counts_->copyTo(x_counts);
  dev_d_counts_->copyTo(d_counts);
}

#define RPU_BLM_START_KERNEL_LINEAR(ITEM_PER_THREAD)                                               \
  int n = (Kplus1 / ITEM_PER_THREAD);                                                              \
                                                                                                   \
  int nblocks = context_->getNBlocks(x_size_ * n, nthreads_);                                      \
  kernelUpdateGetCounts_Linear<T, ITEM_PER_THREAD><<<nblocks, nthreads_, 0, s>>>(                  \
      x_in, x_size_, B, dev_x_counts_->getData(), Kplus1,                                          \
      context_->getRandomStates(nthreads_ * nblocks), res, sr);                                    \
                                                                                                   \
  nblocks = context_->getNBlocks(d_size_ * n, nthreads_);                                          \
  kernelUpdateGetCounts_Linear<T, ITEM_PER_THREAD><<<nblocks, nthreads_, 0, s>>>(                  \
      d_in, d_size_, A, dev_d_counts_->getData(), Kplus1,                                          \
      context_->getRandomStates(nthreads_ * nblocks), res, sr);

template <typename T>
template <typename XInputIteratorT, typename DInputIteratorT>
void BitLineMaker<T>::makeCounts(
    XInputIteratorT x_in,
    DInputIteratorT d_in,
    const PulsedUpdateMetaParameter<T> &up,
    const T dw_min,
    const T lr,
    const int m_batch,
    const bool x_trans,
    const bool d_trans,
    const bool out_trans,
    const int use_bo64) {
  // use_bo64==1 : direct bo64
  // use_bo64==2 : translate into bo64

  // update management
  T A = 0;
  T B = 0;
  up.calculateBlAB(current_BL_, A, B, lr, dw_min);
  current_lr_ = lr; // save for rpu device if needed

  bool update_management = up.update_management;
  bool update_bl_management = up.update_bl_management;
  bool sr = up.sto_round;
  T res = up.res;

  hipStream_t s = this->context_->getStream();

  if (format_ != getFormat(use_bo64) || (buffer_BL_ / 32 < current_BL_ / 32) ||
      (buffer_m_batch_ < m_batch)) {
    initializeBLBuffers(m_batch, current_BL_, use_bo64);
  }

  if ((use_bo64 > 0) && !out_trans) {
    RPU_FATAL("out_trans=false not supported for BO64");
  }

  bool um_if = update_management || update_bl_management;
  T *scale_values = nullptr;
  int *K_values = nullptr;

  if (um_if || use_bo64 > 0) {
    if (umh_ == nullptr) {
      umh_ = RPU::make_unique<UpdateManagementHelper<T>>(context_, x_size_, d_size_);
    }
    if (um_if) {
      umh_->computeKandScaleValues(
          x_in, d_in, dw_min, lr, update_management, update_bl_management, m_batch, x_trans,
          d_trans, current_BL_);

      scale_values = umh_->getScaleValueData();
      K_values = umh_->getKValueData();
    }
  }

  // ------- generate the requested bit lines

  switch (up.pulse_type) {

  case PulseType::StochasticCompressed: {
    // here we generate stochastic bitlines. These are either in 64
    // bit format (32 bits for sign and 32 bits for data) or
    // standard 32 bit format. In the latter case the first bit is
    // the sign bit, which iis the same for the whole word. Longer
    // bitlines can be accomodated by adding words (nK32>1; only in
    // 32-bit case).

    if (format_ == BLMOutputFormat::FP) {
      RPU_FATAL("Expects to be NOT in floating point mode!");
    }

    int Kplus1 = current_BL_ + 1;
    bool possible_linear = (Kplus1 <= 32) && ((32 % Kplus1) == 0);
    // always do simple_loop: fastest in any case
    bool simple_loop = (Kplus1 > RPU_BLM_BL_TO_SELECT_SIMPLE_LOOP) || um_if;
    simple_loop |= use_bo64 == 1; // direct bo64 only supported in simple loop

    if ((m_batch == 1) && (possible_linear && (!simple_loop))) {

      if (!possible_linear) {

        // one block  is a little bit faster than TWOBLOCKS
        int nblocks = context_->getNBlocks(MAX(d_size_, x_size_) * 32, nthreads_);

        kernelUpdateGetCounts_Loop2<<<nblocks, nthreads_, 0, s>>>(
            x_in, x_size_, B, dev_x_counts_->getData(), d_in, d_size_, A, dev_d_counts_->getData(),
            Kplus1, context_->getRandomStates(nthreads_ * nblocks), res, sr);

      } else { // fast path for smaller K values (needs to be K<=32!)

        if ((Kplus1 % RPU_BLM_ITEMS_PER_THREAD) != 0) {
          // just set to 2 (smallest possible)

          RPU_BLM_START_KERNEL_LINEAR(2);

        } else {

          RPU_BLM_START_KERNEL_LINEAR(RPU_BLM_ITEMS_PER_THREAD);
        }
      }

    } else {
      // batch or single batch with simple loop

      if (simple_loop) {

        int m = (d_size_ + x_size_) * m_batch;
        int nblocks = context_->getNBlocks(m, nthreads_);
        nblocks = MAX(MIN(max_block_count_, nblocks), 2);

        if (use_bo64 == 1) {

          // need to set buffers to zero
          dev_x_counts_bo64_->setConst(0);
          dev_d_counts_bo64_->setConst(0);

          kagg_t *Kc_values = nullptr;
          if (update_bl_management) {
            umh_->computeKc(m_batch);
            umh_->computeKn(m_batch);
            Kc_values = umh_->getKcValueData();
          }

          RPU_BLM_SWITCH_TRANS_TEMPLATE_UM(
              x_trans, d_trans, out_trans, update_management, update_bl_management,
              kernelUpdateGetCountsBatch_SimpleLoop2,
              (x_in, x_size_, B, dev_x_counts_bo64_->getData(), d_in, d_size_, A,
               dev_d_counts_bo64_->getData(), current_BL_ + 1, m_batch,
               context_->getRandomStates(nthreads_ * nblocks), res, sr, scale_values, K_values,
               lr / dw_min, Kc_values, umh_->getKnData(update_bl_management)));

        } else {

          // need to set buffers to zero for zero short-cut
          dev_x_counts_->setConst(0);
          dev_d_counts_->setConst(0);

          RPU_BLM_SWITCH_TRANS_TEMPLATE_UM(
              x_trans, d_trans, out_trans, update_management, update_bl_management,
              kernelUpdateGetCountsBatch_SimpleLoop2,
              (x_in, x_size_, B, dev_x_counts_->getData(), d_in, d_size_, A,
               dev_d_counts_->getData(), current_BL_ + 1, m_batch,
               context_->getRandomStates(nthreads_ * nblocks), res, sr, scale_values, K_values,
               lr / dw_min));
        }
      } else {
        int m = MAX(d_size_, x_size_) * m_batch * 32;
        int nblocks = context_->getNBlocks(m, nthreads_);
        nblocks = MIN(max_block_count_, nblocks);

        RPU_BLM_SWITCH_TRANS_TEMPLATE(
            x_trans, d_trans, out_trans, kernelUpdateGetCountsBatch_Loop2,
            (x_in, x_size_, B, dev_x_counts_->getData(), d_in, d_size_, A, dev_d_counts_->getData(),
             current_BL_ + 1, m_batch, context_->getRandomStates(nthreads_ * nblocks), res, sr), );
      }
    }

    // translate to BO64 if necessary
    if (use_bo64 > 1) {
      umh_->translateTransToBatchOrder64(
          dev_x_counts_bo64_->getData(), dev_d_counts_bo64_->getData(), dev_x_counts_->getData(),
          dev_d_counts_->getData(), m_batch, current_BL_, update_bl_management);
    }

    DEBUG_CALL(context_->synchronizeDevice(); CudaArray<T> dev_x(context_, x_size_);
               CudaArray<T> dev_d(context_, d_size_);
               RPU::math::copyWithIterator(context_, dev_x.getData(), x_in, x_size_);
               RPU::math::copyWithIterator(context_, dev_d.getData(), d_in, d_size_);
               context_->synchronizeDevice(); test_helper::checkCounts(
                   dev_x.getData(), x_size_, dev_d.getData(), d_size_, current_BL_, A, B,
                   &*dev_x_counts_, &*dev_d_counts_);
               context_->synchronizeDevice(););
  } break;

  default:
    RPU_FATAL("PulseType not supported by BitLineMaker");
  }
};

template class BitLineMaker<float>;
#ifdef RPU_USE_DOUBLE
template class BitLineMaker<double>;
#endif

#define RPU_BLM_ITER_TEMPLATE(NUM_T, XITERT, DITERT)                                               \
  template void BitLineMaker<NUM_T>::makeCounts(                                                   \
      XITERT, DITERT, const PulsedUpdateMetaParameter<NUM_T> &, const NUM_T, const NUM_T,          \
      const int, const bool, const bool, const bool, const int);

#define TRANSFLOAT(TRANS) TRANS, float

RPU_BLM_ITER_TEMPLATE(float, const float *, const float *);
RPU_BLM_ITER_TEMPLATE(float, float *, float *);
RPU_BLM_ITER_TEMPLATE(float, IndexReaderInputIterator<float>, const float *);
RPU_BLM_ITER_TEMPLATE(float, IndexReaderTransInputIterator<float>, const float *);
RPU_BLM_ITER_TEMPLATE(
    float, IndexReaderTransInputIterator<float>, PermuterTransInputIterator<float>);
RPU_BLM_ITER_TEMPLATE(float, const float *, PermuterTransInputIterator<float>);

#undef TRANSFLOAT

#ifdef RPU_USE_DOUBLE
#define TRANSDOUBLE(TRANS) TRANS, double

RPU_BLM_ITER_TEMPLATE(double, const double *, const double *);
RPU_BLM_ITER_TEMPLATE(double, double *, double *);
RPU_BLM_ITER_TEMPLATE(double, IndexReaderInputIterator<double>, const double *);
RPU_BLM_ITER_TEMPLATE(double, IndexReaderTransInputIterator<double>, const double *);
RPU_BLM_ITER_TEMPLATE(
    double, IndexReaderTransInputIterator<double>, PermuterTransInputIterator<double>);
RPU_BLM_ITER_TEMPLATE(double, const double *, PermuterTransInputIterator<double>);

#undef TRANSDOUBLE
#endif

#undef RPU_BLM_ITER_TEMPLATE

#undef RPU_BLM_SWITCH_TRANS_TEMPLATE
#undef RPU_BLM_SWITCH_TRANS_TEMPLATE_UM
#undef RPU_BLM_ITEMS_PER_THREAD
#undef RPU_BLM_START_KERNEL_LINEAR
#undef RPU_BLM_BL_TO_SELECT_SIMPLE_LOOP
#undef LASTK32MASK
#undef RPU_BLM_DEFINE_NK32
#undef RPU_BLM_DEFINE_NK32_BATCH
#undef COMMA
#undef RPU_BLM_DEBUG_INIT
#undef RPU_BLM_DEBUG_FINISH
#undef RPU_BLM_DEBUG_BATCH_INIT
#undef RPU_BLM_DEBUG_BATCH_FINISH
#undef DISCRETIZE_VALUE_STOCH_DEFINITIONS
#undef DISCRETIZE_VALUE_STOCH
#undef DISCRETIZE_VALUE
#undef GET_COUNTS_INNER_LOOP
#undef GET_COUNTS_LOOP
#undef GET_COUNTS_LOOP_BATCH
#undef GET_COUNTS_SIMPLE_LOOP_BATCH
#undef RPU_BLM_BLOCKS_PER_SM
} // namespace RPU
