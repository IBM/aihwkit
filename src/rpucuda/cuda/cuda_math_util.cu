#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021, 2022 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "cuda_math_util.h"
#include "cuda_util.h"
#include "io_iterator.h"

#define RPU_GET_CUBLAS_HANDLE                                                                      \
  hipblasHandle_t handle = context->getBlasHandle();                                                \
  CUBLAS_CALL(hipblasSetStream(handle, context->getStream()))

#define RPU_SET_CUBLAS_POINTER_MODE_DEVICE                                                         \
  hipblasPointerMode_t p_mode;                                                                      \
  CUBLAS_CALL(hipblasGetPointerMode(handle, &p_mode));                                              \
  CUBLAS_CALL(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE))

#define RPU_SET_CUBLAS_POINTER_MODE_HOST                                                           \
  hipblasPointerMode_t p_mode;                                                                      \
  CUBLAS_CALL(hipblasGetPointerMode(handle, &p_mode));                                              \
  CUBLAS_CALL(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST))

#define RPU_RESTORE_CUBLAS_POINTER_MODE CUBLAS_CALL(hipblasSetPointerMode(handle, p_mode))

namespace RPU {
namespace math {

template <>
void gemm<float>(
    const CudaContextPtr context,
    const bool TransA,
    const bool TransB,
    const int M,
    const int N,
    const int K,
    const float alpha,
    const float *A,
    const int lda,
    const float *B,
    const int ldb,
    const float beta,
    float *C,
    const int ldc) {
  RPU_GET_CUBLAS_HANDLE;

  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasSgemm(
      handle, TransA ? HIPBLAS_OP_T : HIPBLAS_OP_N, TransB ? HIPBLAS_OP_T : HIPBLAS_OP_N, M, N, K,
      &alpha, A, lda, B, ldb, &beta, C, ldc));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
};

template <>
void gemm<double>(
    const CudaContextPtr context,
    const bool TransA,
    const bool TransB,
    const int M,
    const int N,
    const int K,
    const double alpha,
    const double *A,
    const int lda,
    const double *B,
    const int ldb,
    const double beta,
    double *C,
    const int ldc) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasDgemm(
      handle, TransA ? HIPBLAS_OP_T : HIPBLAS_OP_N, TransB ? HIPBLAS_OP_T : HIPBLAS_OP_N, M, N, K,
      &alpha, A, lda, B, ldb, &beta, C, ldc));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
};

template <>
int iamax<float>(const CudaContextPtr context, const int N, const float *X, const int incX) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  int result = 0;
  CUBLAS_CALL(hipblasIsamax(handle, N, X, incX, &result));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
  return result - 1; // make 0 based index !!
};

template <>
int iamax<double>(const CudaContextPtr context, const int N, const double *X, const int incX) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  int result;
  CUBLAS_CALL(hipblasIdamax(handle, N, X, incX, &result));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
  return result - 1; // make 0 based index
};

template <>
void copy<float>(
    const CudaContextPtr context,
    const int N,
    const float *X,
    const int incX,
    float *Y,
    const int incY) {
  RPU_GET_CUBLAS_HANDLE;
  CUBLAS_CALL(hipblasScopy(handle, N, X, incX, Y, incY));
}

template <>
void copy<double>(
    const CudaContextPtr context,
    const int N,
    const double *X,
    const int incX,
    double *Y,
    const int incY) {
  RPU_GET_CUBLAS_HANDLE;
  CUBLAS_CALL(hipblasDcopy(handle, N, X, incX, Y, incY));
}

template <>
void scal<float>(
    const CudaContextPtr context, const int N, const float alpha, float *X, const int incX) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasSscal(handle, N, &alpha, X, incX));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}

template <>
void scal<double>(
    const CudaContextPtr context, const int N, const double alpha, double *X, const int incX) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasDscal(handle, N, &alpha, X, incX));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}

template <>
void nrm2<float>(
    const CudaContextPtr context, const int N, const float *X, const int incX, float *res) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_DEVICE;
  CUBLAS_CALL(hipblasSnrm2(handle, N, X, incX, res));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}

template <>
void nrm2<double>(
    const CudaContextPtr context, const int N, const double *X, const int incX, double *res) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_DEVICE;
  CUBLAS_CALL(hipblasDnrm2(handle, N, X, incX, res));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}

template <>
void gemv<float>(
    const CudaContextPtr context,
    const bool TransA,
    const int M,
    const int N,
    const float alpha,
    const float *A,
    const int lda,
    const float *X,
    const int incX,
    const float beta,
    float *Y,
    const int incY) {
  RPU_GET_CUBLAS_HANDLE;
  // col major !!
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasSgemv(
      handle, TransA ? HIPBLAS_OP_T : HIPBLAS_OP_N, M, N, &alpha, A, lda, X, incX, &beta, Y, incY));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}
template <>
void gemv<double>(
    const CudaContextPtr context,
    const bool TransA,
    const int M,
    const int N,
    const double alpha,
    const double *A,
    const int lda,
    const double *X,
    const int incX,
    const double beta,
    double *Y,
    const int incY) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasDgemv(
      handle, TransA ? HIPBLAS_OP_T : HIPBLAS_OP_N, M, N, &alpha, A, lda, X, incX, &beta, Y, incY));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
}

template <>
void ger<float>(
    const CudaContextPtr context,
    const int M,
    const int N,
    const float alpha,
    const float *X,
    const int incX,
    const float *Y,
    const int incY,
    float *A,
    const int lda) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasSger(handle, M, N, &alpha, X, incX, Y, incY, A, lda));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
};

template <>
void ger<double>(
    const CudaContextPtr context,
    const int M,
    const int N,
    const double alpha,
    const double *X,
    const int incX,
    const double *Y,
    const int incY,
    double *A,
    const int lda) {
  RPU_GET_CUBLAS_HANDLE;
  RPU_SET_CUBLAS_POINTER_MODE_HOST;
  CUBLAS_CALL(hipblasDger(handle, M, N, &alpha, X, incX, Y, incY, A, lda));
  RPU_RESTORE_CUBLAS_POINTER_MODE;
};

// W += A
template <typename T> __global__ void kernelElemAdd(T *dev_W, const int size, const T *dev_A) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_W[idx] += dev_A[idx]; }
}

template <typename T>
void elemadd(const CudaContextPtr context, T *dev_W, const int size, const T *dev_A) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAdd<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A);
}
template void elemadd<float>(const CudaContextPtr, float *, const int, const float *);
#ifdef RPU_USE_DOUBLE
template void elemadd<double>(const CudaContextPtr, double *, const int, const double *);
#endif

// W = W.*W
template <typename T> __global__ void kernelElemPow2(T *dev_W, const int size, const T *W_in) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T x = W_in[idx];
    dev_W[idx] = x * x;
  }
}
template <typename T>
void elempow2(const CudaContextPtr context, T *dev_W, const int size, const T *dev_W_in) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemPow2<T><<<nblocks, nthreads, 0, context->getStream()>>>(
      dev_W, size, dev_W_in == nullptr ? dev_W : dev_W_in);
}
template void elempow2<float>(const CudaContextPtr, float *, const int, const float *);
#ifdef RPU_USE_DOUBLE
template void elempow2<double>(const CudaContextPtr, double *, const int, const double *);
#endif

// V = abs(W )
template <typename T> __global__ void kernelElemAbs(T *dev_V, const T *dev_W, const int size) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_V[idx] = fabs(dev_W[idx]); }
}
template <typename T>
void elemabs(const CudaContextPtr context, T *dev_V, const T *dev_W, const int size) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAbs<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_V, dev_W, size);
}
template void elemabs<float>(const CudaContextPtr, float *, const float *, const int);
#ifdef RPU_USE_DOUBLE
template void elemabs<double>(const CudaContextPtr, double *, const double *, const int);
#endif

// W += beta*A
template <typename T, typename T_A>
__global__ void kernelElemAddScale(T *dev_W, const int size, const T_A *dev_A, const T beta) {

  T b = beta;
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = dev_W[idx];
    T a = dev_A[idx];

    w += a * b;
    dev_W[idx] = w;
  }
}
template <typename T, typename T_A>
void elemaddscale(
    const CudaContextPtr context, T *dev_W, const int size, const T_A *dev_A, const T beta) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAddScale<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, beta);
}
template void
elemaddscale<float, float>(const CudaContextPtr, float *, const int, const float *, const float);
#ifdef RPU_USE_DOUBLE
template void elemaddscale<double, double>(
    const CudaContextPtr, double *, const int, const double *, const double);
template void
elemaddscale<double, float>(const CudaContextPtr, double *, const int, const float *, const double);
#endif

// W += A.*B
template <typename T>
__global__ void kernelElemAddScale(T *dev_W, const int size, const T *dev_A, const T *dev_B) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = dev_W[idx];
    T a = dev_A[idx];
    T b = dev_B[idx];

    w += a * b;
    dev_W[idx] = w;
  }
}
template <typename T>
void elemaddscale(
    const CudaContextPtr context, T *dev_W, const int size, const T *dev_A, const T *dev_B) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAddScale<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, dev_B);
}
template void
elemaddscale<float>(const CudaContextPtr, float *, const int, const float *, const float *);
#ifdef RPU_USE_DOUBLE
template void
elemaddscale<double>(const CudaContextPtr, double *, const int, const double *, const double *);
#endif

// W += sat(A.*B)
template <typename T, typename T_A>
__global__ void
kernelElemASB02(T *dev_W, const int size, const T_A *dev_A, const T *dev_B, float *dev_4params) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {

    T w = dev_W[idx];
    T a = dev_A[idx];
    T b = dev_B[idx];
    float4 parij = reinterpret_cast<float4 *>(dev_4params)[idx];

    w += a * b;
    // check bounds
    w = (w > parij.z) ? parij.z : w;
    w = (w < parij.x) ? parij.x : w;

    dev_W[idx] = w;
  }
}
template <typename T, typename T_A>
void elemasb02(
    const CudaContextPtr context,
    T *dev_W,
    const int size,
    const T_A *dev_A,
    const T *dev_B,
    float *dev_4params) {
  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemASB02<T, T_A>
      <<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, dev_B, dev_4params);
}
template void elemasb02<float, float>(
    const CudaContextPtr, float *, const int, const float *, const float *, float *);
#ifdef RPU_USE_DOUBLE
template void elemasb02<double, double>(
    const CudaContextPtr, double *, const int, const double *, const double *, float *);
template void elemasb02<double, float>(
    const CudaContextPtr, double *, const int, const float *, const double *, float *);
#endif

// sat(W *= A) (w/shift)
template <typename T>
__global__ void kernelElemScaleSat(
    T *dev_W, const int size, const T *dev_A, float *dev_4params, const T *dev_shift) {

  bool with_shift = dev_shift != nullptr;
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = dev_W[idx];
    T a = dev_A[idx];
    T s = with_shift ? dev_shift[idx] : 0.0;
    float4 parij = reinterpret_cast<float4 *>(dev_4params)[idx];

    w = (w - s) * a + s;
    // check bounds
    w = (w > parij.z) ? parij.z : w;
    w = (w < parij.x) ? parij.x : w;

    dev_W[idx] = w;
  }
}
// W *= A (w/shift)
template <typename T>
__global__ void kernelElemScale(T *dev_W, const int size, const T *dev_A, const T *dev_shift) {
  bool with_shift = dev_shift != nullptr;
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = dev_W[idx];
    T a = dev_A[idx];
    T s = with_shift ? dev_shift[idx] : 0.0;
    w = (w - s) * a + s;
    dev_W[idx] = w;
  }
}

template <typename T>
void elemscale(
    const CudaContextPtr context,
    T *dev_W,
    const int size,
    const T *dev_A,
    float *dev_4params,
    const T *dev_shift) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  if (dev_4params != nullptr) {
    kernelElemScaleSat<T><<<nblocks, nthreads, 0, context->getStream()>>>(
        dev_W, size, dev_A, dev_4params, dev_shift);
  } else {
    kernelElemScale<T>
        <<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, dev_shift);
  }
}
template void
elemscale<float>(const CudaContextPtr, float *, const int, const float *, float *, const float *);
#ifdef RPU_USE_DOUBLE
template void elemscale<double>(
    const CudaContextPtr, double *, const int, const double *, float *, const double *);
#endif

// C = A.*B
template <typename T>
__global__ void kernelElemMul(T *dev_C, const int size, const T *dev_A, const T *dev_B) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_C[idx] = dev_A[idx] * dev_B[idx]; }
}

template <typename T>
void elemmul(
    const CudaContextPtr context, T *dev_C, const int size, const T *dev_A, const T *dev_B) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemMul<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_C, size, dev_A, dev_B);
}
template void
elemmul<float>(const CudaContextPtr, float *, const int, const float *, const float *);
#ifdef RPU_USE_DOUBLE
template void
elemmul<double>(const CudaContextPtr, double *, const int, const double *, const double *);
#endif

// sat(W)
template <typename T> __global__ void kernelElemSat(T *dev_W, const int size, float *dev_4params) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {

    T w = dev_W[idx];
    float4 parij = reinterpret_cast<float4 *>(dev_4params)[idx];
    // check bounds
    w = (w > parij.z) ? parij.z : w;
    w = (w < parij.x) ? parij.x : w;
    dev_W[idx] = w;
  }
}
template <typename T>
void elemsat(const CudaContextPtr context, T *dev_W, const int size, float *dev_4params) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemSat<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_4params);
}
template void elemsat<float>(const CudaContextPtr, float *, const int, float *);
#ifdef RPU_USE_DOUBLE
template void elemsat<double>(const CudaContextPtr, double *, const int, float *);
#endif

// sat(W *= 1+(A-1)*alpha)
template <typename T>
__global__ void kernelElemScaleAlpha(
    T *dev_W,
    const int size,
    const T *dev_A,
    float *dev_4params,
    const T alpha,
    const T *dev_shift) {

  bool with_shift = dev_shift != nullptr;
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {

    T w = dev_W[idx];
    T a = dev_A[idx];
    T s = with_shift ? dev_shift[idx] : 0.0;
    float4 parij = reinterpret_cast<float4 *>(dev_4params)[idx];

    T scale = 1.0 + alpha * (a - 1.0);
    w = (w - s) * scale + s;

    // check bounds
    w = (w > parij.z) ? parij.z : w;
    w = (w < parij.x) ? parij.x : w;

    dev_W[idx] = w;
  }
}
template <typename T>
void elemscalealpha(
    const CudaContextPtr context,
    T *dev_W,
    const int size,
    const T *dev_A,
    float *dev_4params,
    const T alpha,
    const T *dev_shift) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemScaleAlpha<T><<<nblocks, nthreads, 0, context->getStream()>>>(
      dev_W, size, dev_A, dev_4params, alpha, dev_shift);
}
template void elemscalealpha<float>(
    const CudaContextPtr, float *, const int, const float *, float *, const float, const float *);
#ifdef RPU_USE_DOUBLE
template void elemscalealpha<double>(
    const CudaContextPtr,
    double *,
    const int,
    const double *,
    float *,
    const double,
    const double *);
#endif

// W += A, A = W
template <typename T> __global__ void kernelElemAddCopy(T *dev_W, T *dev_A, const int size) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = dev_W[idx];
    w += dev_A[idx];
    dev_W[idx] = w;
    dev_A[idx] = w;
  }
}
template <typename T>
void elemaddcopy(const CudaContextPtr context, T *dev_W, T *dev_A, const int size) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAddCopy<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, dev_A, size);
}
template void elemaddcopy<float>(const CudaContextPtr, float *, float *, const int);
#ifdef RPU_USE_DOUBLE
template void elemaddcopy<double>(const CudaContextPtr, double *, double *, const int);
#endif

// W = sat(W+A), A = W
template <typename T>
__global__ void kernelElemAddCopySat(T *dev_W, T *dev_A, const int size, const float *dev_4params) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {

    T w = dev_W[idx];
    T a = dev_A[idx];
    const float4 parij = reinterpret_cast<const float4 *>(dev_4params)[idx];
    w += a;
    // check bounds
    w = (w > parij.z) ? parij.z : w;
    w = (w < parij.x) ? parij.x : w;
    a = w;
    dev_W[idx] = w;
    dev_A[idx] = a;
  }
}
template <typename T>
void elemaddcopysat(
    const CudaContextPtr context, T *dev_W, T *dev_A, const int size, const float *dev_4params) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAddCopySat<T>
      <<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, dev_A, size, dev_4params);
}
template void
elemaddcopysat<float>(const CudaContextPtr, float *, float *, const int, const float *);
#ifdef RPU_USE_DOUBLE
template void
elemaddcopysat<double>(const CudaContextPtr, double *, double *, const int, const float *);
#endif

// MSK = P<thres
// W(MSK) = sat(A(MSK) + B(MSK))
template <typename T>
__global__ void kernelElemResetSat(
    T *dev_W,
    const int size,
    const T *dev_A,
    const float *dev_B,
    const float *dev_P,
    const T thres,
    const float *dev_4params) {

  bool with_A = dev_A != nullptr;
  bool with_B = dev_B != nullptr;
  bool with_P = dev_P != nullptr;

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T th = thres;
    T a = (with_A) ? dev_A[idx] : (float)0.0;
    float p = (with_P) ? dev_P[idx] : (float)0.0;
    float b = (with_B) ? dev_B[idx] : (float)0.0;
    T w = dev_W[idx];
    const float4 parij = reinterpret_cast<const float4 *>(dev_4params)[idx];

    if (p < th) {
      w = a + b;
      // check bounds [only those that changed]
      w = (w > parij.z) ? parij.z : w;
      w = (w < parij.x) ? parij.x : w;
      dev_W[idx] = w;
    }
  }
}
template <typename T>
void elemresetsat(
    const CudaContextPtr context,
    T *dev_W,
    const int size,
    const T *dev_A,
    const float *dev_B,
    const float *dev_P,
    T thres,
    const float *dev_4params) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemResetSat<T><<<nblocks, nthreads, 0, context->getStream()>>>(
      dev_W, size, dev_A, dev_B, dev_P, thres, dev_4params);
}
template void elemresetsat<float>(
    const CudaContextPtr,
    float *,
    const int,
    const float *,
    const float *,
    const float *,
    const float,
    const float *);
#ifdef RPU_USE_DOUBLE
template void elemresetsat<double>(
    const CudaContextPtr,
    double *,
    const int,
    const double *,
    const float *,
    const float *,
    const double,
    const float *);
#endif

// MSK != 0
// W(MSK) = sat(reset_bias(MSK) + std*randn())
#define RESET_TOLERANCE 1e-6
template <typename T>
__global__ void kernelElemResetSatMsk(
    T *weights,
    const int size,
    const char *msk,
    const T *reset_bias,
    const T reset_std_in,
    const float *dev_4params,
    hiprandState_t *random_states) {

  volatile unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
  T reset_std = reset_std_in;
  int total_threads = blockDim.x * gridDim.x;
  hiprandState_t local_state;
  bool with_bias = reset_bias != nullptr;

  if (reset_std) {
    local_state = random_states[tid];
  }

  for (int idx = tid; idx < size; idx += total_threads) {

    bool reset_if = msk[idx] != 0;

    if (reset_if) {
      // assume very sparse reset thus only read if reset
      T bias = with_bias ? reset_bias[idx] : (T)0.0;
      T w;
      const float4 parij = reinterpret_cast<const float4 *>(dev_4params)[tid];
      if (reset_std) {
        w = bias + reset_std * hiprand_normal(&local_state);
      } else {
        w = bias;
      }
      w = (w > parij.z) ? parij.z : w;
      w = (w < parij.x) ? parij.x : w;
      weights[idx] = w;
    }
  }
  if (reset_std) {
    random_states[tid] = local_state;
  }
}
#undef RESET_TOLERANCE
template <typename T>
void elemresetsatmsk(
    CudaContextPtr context,
    T *W,
    const int size,
    const char *msk,
    const T *reset_bias,
    const T reset_std,
    const float *dev_4params) {

  int nthreads = context->getNThreads();
  int nblocks_batch_max = context->getSMCount() * (context->maxThreadsPerBlock() / nthreads);
  int nblocks = MIN(context->getNBlocks(size, nthreads), nblocks_batch_max);
  kernelElemResetSatMsk<T><<<nblocks, nthreads, 0, context->getStream()>>>(
      W, size, msk, reset_bias, reset_std, dev_4params,
      context->getRandomStates(nblocks * nthreads));
}
template void elemresetsatmsk<float>(
    CudaContextPtr, float *, const int, const char *, const float *, const float, const float *x);
#ifdef RPU_USE_DOUBLE
template void elemresetsatmsk<double>(
    CudaContextPtr, double *, const int, const char *, const double *, const double, const float *);
#endif

// A = W - A_in; W = A_in;
template <typename T>
__global__ void kernelElemSubCopy(T *dev_W, T *dev_A, const int size, const T scale) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T new_w = dev_W[idx];
    T old_w = dev_A[idx];
    dev_W[idx] = old_w;
    dev_A[idx] = scale * (new_w - old_w); // dw
  }
}
template <typename T>
void elemsubcopy(const CudaContextPtr context, T *dev_W, T *dev_A, const int size, const T scale) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemSubCopy<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, dev_A, size, scale);
}
template void elemsubcopy<float>(const CudaContextPtr, float *, float *, const int, const float);
#ifdef RPU_USE_DOUBLE
template void
elemsubcopy<double>(const CudaContextPtr, double *, double *, const int, const double);
#endif

// set all elements to a
template <typename T> __global__ void kernelSetConstAlpha(T *values, int size, T alpha) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { values[idx] = alpha; }
}

template <typename T>
void elemconst(const CudaContextPtr context, T *X, const int size, const T alpha) {
  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelSetConstAlpha<<<nblocks, nthreads, 0, context->getStream()>>>(X, size, alpha);
}
template void elemconst<float>(const CudaContextPtr, float *, const int, const float);
#ifdef RPU_USE_DOUBLE
template void elemconst<double>(const CudaContextPtr, double *, const int, const double);
#endif
template void elemconst<uint32_t>(const CudaContextPtr, uint32_t *, const int, const uint32_t);
template void elemconst<uint64_t>(const CudaContextPtr, uint64_t *, const int, const uint64_t);
template void elemconst<int>(const CudaContextPtr, int *, const int, const int);
template void elemconst<char>(const CudaContextPtr, char *, const int, const char);
template void elemconst<int8_t>(const CudaContextPtr, int8_t *, const int, const int8_t);

template <typename T>
void elemconst(const CudaContext *context, T *X, const int size, const T alpha) {
  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelSetConstAlpha<<<nblocks, nthreads, 0, context->getStream()>>>(X, size, alpha);
}
template void elemconst<float>(const CudaContext *, float *, const int, const float);
#ifdef RPU_USE_DOUBLE
template void elemconst<double>(const CudaContext *, double *, const int, const double);
#endif
template void elemconst<uint32_t>(const CudaContext *, uint32_t *, const int, const uint32_t);
template void elemconst<uint64_t>(const CudaContext *, uint64_t *, const int, const uint64_t);
template void elemconst<int>(const CudaContext *, int *, const int, const int);
template void elemconst<char>(const CudaContext *, char *, const int, const char);

// w = max(min(w,|a|),-|a|)
template <typename T> __global__ void kernelAClip(T *values, int size, T abs_a) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { values[idx] = MIN(MAX(values[idx], -abs_a), abs_a); }
}

template <typename T> void aclip(const CudaContextPtr context, T *W, const int size, const T a) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelAClip<<<nblocks, nthreads, 0, context->getStream()>>>(W, size, fabs(a));
}
template void aclip<float>(const CudaContextPtr, float *, const int, const float);
#ifdef RPU_USE_DOUBLE
template void aclip<double>(const CudaContextPtr, double *, const int, const double);
#endif

// w = max(w,a)
template <typename T> __global__ void kernelElemmax(T *values, int size, T a, const T *in_values) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { values[idx] = MAX(in_values[idx], a); }
}

template <typename T>
void elemmax(const CudaContextPtr context, T *W, const int size, const T a, const T *in_values) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemmax<<<nblocks, nthreads, 0, context->getStream()>>>(
      W, size, a, in_values != nullptr ? in_values : W);
}
template void elemmax<float>(const CudaContextPtr, float *, const int, const float, const float *);
#ifdef RPU_USE_DOUBLE
template void
elemmax<double>(const CudaContextPtr, double *, const int, const double, const double *);
#endif

// w = min(w,a)
template <typename T> __global__ void kernelElemmin(T *values, int size, T a, const T *in_values) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { values[idx] = MIN(in_values[idx], a); }
}

template <typename T>
void elemmin(const CudaContextPtr context, T *W, const int size, const T a, const T *in_values) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemmin<<<nblocks, nthreads, 0, context->getStream()>>>(
      W, size, a, in_values != nullptr ? in_values : W);
}
template void elemmin<float>(const CudaContextPtr, float *, const int, const float, const float *);
#ifdef RPU_USE_DOUBLE
template void
elemmin<double>(const CudaContextPtr, double *, const int, const double, const double *);
#endif

// w = w<a?0:w elementwise
template <typename T> __global__ void kernelElemSetBelowZero(T *values, int size, T a) {
  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T v = values[idx];
    values[idx] = v < a ? (T)0.0 : v;
  }
}

template <typename T>
void elemsetbelowzero(const CudaContextPtr context, T *W, const int size, const T a) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemSetBelowZero<<<nblocks, nthreads, 0, context->getStream()>>>(W, size, a);
}
template void elemsetbelowzero<float>(const CudaContextPtr, float *, const int, const float);
#ifdef RPU_USE_DOUBLE
template void elemsetbelowzero<double>(const CudaContextPtr, double *, const int, const double);
#endif

// w[j] = a*A[j] + b*B[j]
// with fast paths for A-B and -B+A and A+B.
template <typename T>
__global__ void kernelElemAdd(T *dev_W, const int size, const T *dev_A, const T *dev_B) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_W[idx] = dev_A[idx] + dev_B[idx]; }
}
template <typename T>
__global__ void kernelElemSub(T *dev_W, const int size, const T *dev_A, const T *dev_B) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_W[idx] = dev_A[idx] - dev_B[idx]; }
}

template <typename T>
__global__ void kernelElemWeightedSum(
    T *dev_W, const int size, const T *dev_A, const T a, const T *dev_B, const T b) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) { dev_W[idx] = a * dev_A[idx] + b * dev_B[idx]; }
}

template <typename T>
void elemweightedsum(
    const CudaContextPtr context,
    T *dev_W,
    const int size,
    const T *dev_A,
    const T a,
    const T *dev_B,
    const T b) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  if (a == 1 && b == 1) {
    kernelElemAdd<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, dev_B);
  } else if (a == 1 && b == -1) {
    kernelElemSub<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, dev_B);
  } else if (a == -1 && b == 1) {
    kernelElemSub<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_B, dev_A);
  } else {
    kernelElemWeightedSum<T>
        <<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_A, a, dev_B, b);
  }
}
template void elemweightedsum<float>(
    const CudaContextPtr,
    float *,
    const int,
    const float *,
    const float,
    const float *,
    const float);
#ifdef RPU_USE_DOUBLE
template void elemweightedsum<double>(
    const CudaContextPtr,
    double *,
    const int,
    const double *,
    const double,
    const double *,
    const double);
#endif

// w[j] = sum_i^n(m_i[j])/n
template <typename T>
__global__ void kernelElemAverage(T *dev_W, const int size, T **dev_Ms, const int m) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T w = 0;
    for (int i = 0; i < m; i++) {
      const T *w1 = dev_Ms[i];
      w += w1[idx];
    }
    dev_W[idx] = w / (T)m;
  }
}

template <typename T>
void elemaverage(const CudaContextPtr context, T *dev_W, const int size, T **dev_Ms, const int m) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelElemAverage<T><<<nblocks, nthreads, 0, context->getStream()>>>(dev_W, size, dev_Ms, m);
}
template void elemaverage<float>(const CudaContextPtr, float *, const int, float **, const int);
#ifdef RPU_USE_DOUBLE
template void elemaverage<double>(const CudaContextPtr, double *, const int, double **, const int);
#endif

// permute (1,3,2)
template <typename T>
__global__ void kernelPermute132Bias(
    T *dev_X_out,
    const T *dev_X_in,
    const int size,
    const int d1_in,
    const int d2, // with bias added
    const int d3,
    const bool bias) {

  volatile unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  int d1 = d1_in;
  int b = (bias) ? 1 : 0;
  int sz = size;
  int d13 = d1 * d3;
  int d2wob = d2 - b;
  int szwob = (b == 1) ? sz - d13 : sz;
  int d12 = d1 * d2wob;
  int idx1 = tid % d1;
  int idx2 = (tid / d1) % d2wob;
  int idx3 = tid / d12;

  if (tid < szwob) {
    T x = dev_X_in[tid];
    dev_X_out[idx2 * d13 + idx3 * d1 + idx1] = x;
  } else if (tid < sz) {
    dev_X_out[tid] = 1.0;
  }
}

template <typename T>
void permute132(
    const CudaContextPtr context,
    T *X_out,
    const T *X_in,
    const int d1,
    const int d2, // with bias added
    const int d3,
    const bool bias) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(d1 * d2 * d3, nthreads);
  kernelPermute132Bias<T>
      <<<nblocks, nthreads, 0, context->getStream()>>>(X_out, X_in, d1 * d2 * d3, d1, d2, d3, bias);
}

template void permute132<float>(
    const CudaContextPtr, float *, const float *, const int, const int, const int, const bool);
#ifdef RPU_USE_DOUBLE
template void permute132<double>(
    const CudaContextPtr, double *, const double *, const int, const int, const int, const bool);
#endif

// copy with bias
template <typename T>
__global__ void
kernelCopyBiasTrans(T *dest_values, const T *source_values, int size, int size_without_bias) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < size_without_bias) {
    dest_values[tid] = source_values[tid];
  } else if (tid < size) {
    dest_values[tid] = (T)1.;
  }
}

template <typename T>
__global__ void kernelCopyBias(T *dest_values, const T *source_values, int size, int ld) {

  // might be somewhat inefficient because of offset memory block access...
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int ldmod = (tid + 1) % ld;
  int lddiv = tid / ld;
  int idx = tid - lddiv;
  if (tid < size) {
    if (ldmod == 0)
      dest_values[tid] = (T)1.0;
    else
      dest_values[tid] = source_values[idx];
  }
}

template <typename T>
void makeBias(
    const CudaContextPtr context,
    T *x_with_bias,
    const T *x_without_bias,
    const int x_size,
    const int m_batch,
    const bool trans) {

  hipStream_t s = context->getStream();
  int size = m_batch * x_size;
  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);

  if (trans) {
    int ld = m_batch;
    // put ld ones at end
    kernelCopyBiasTrans<T>
        <<<nblocks, nthreads, 0, s>>>(x_with_bias, x_without_bias, size, size - ld);
  } else {
    // put single one each ld-1 distance: [...(x_size-1)..,1,...(x_size-1)...,1,...]
    kernelCopyBias<T><<<nblocks, nthreads, 0, s>>>(x_with_bias, x_without_bias, size, x_size);
  }
}
template void
makeBias<float>(const CudaContextPtr, float *, const float *, const int, const int, const bool);
template void
makeBias<int>(const CudaContextPtr, int *, const int *, const int, const int, const bool);
#ifdef RPU_USE_DOUBLE
template void
makeBias<double>(const CudaContextPtr, double *, const double *, const int, const int, const bool);
#endif

// copy without bias (backward)
template <typename T>
void copyWithoutBias(
    const CudaContextPtr context,
    T *x_without_bias,
    const T *x_with_bias,
    const int x_size,
    const int m_batch,
    const bool trans) {

  hipStream_t s = context->getStream();
  // context_->synchronize(); // seems to be necessary ?!? (at least during nvprof)
  if (trans) {
    // m_batch first
    int sz = m_batch * (x_size - 1);

    CUDA_CALL(hipMemcpyAsync(
        (void *)x_without_bias, (const void *)x_with_bias, sizeof(T) * sz, hipMemcpyDeviceToDevice,
        s));
  } else {
    // x_size first
    int w = sizeof(T) * (x_size);
    int wm1 = w - sizeof(T);
    CUDA_CALL(hipMemcpy2DAsync(
        (void *)x_without_bias,    // dst
        wm1,                       // dpitch
        (const void *)x_with_bias, // src,
        w,                         // spitch,
        wm1,                       // width in bytes
        m_batch,                   // height
        hipMemcpyDeviceToDevice, s));
  }
}

template void copyWithoutBias<float>(
    const CudaContextPtr, float *, const float *, const int, const int, const bool);
#ifdef RPU_USE_DOUBLE
template void copyWithoutBias<double>(
    const CudaContextPtr, double *, const double *, const int, const int, const bool);
#endif

// addWithIterator
template <typename OutputIteratorT, typename T>
__global__ void kernelAddWithIterator(
    OutputIteratorT out_tensor, const T *in_tensor_a, const T *in_tensor_b, const int sz_all) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, sz_all) { out_tensor[idx] = in_tensor_a[idx] + in_tensor_b[idx]; }
}

template <typename OutputIteratorT, typename T>
void addWithIterator(
    const CudaContextPtr context,
    OutputIteratorT out_tensor,
    const T *in_tensor_a,
    const T *in_tensor_b,
    const int total_input_size) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(total_input_size, nthreads);

  kernelAddWithIterator<OutputIteratorT, T><<<nblocks, nthreads, 0, context->getStream()>>>(
      out_tensor, in_tensor_a, in_tensor_b, total_input_size);
}

#define RPU_CMU_DEFINE_AWI(OUTPUTITER, T)                                                          \
  template void addWithIterator<OUTPUTITER, T>(                                                    \
      const CudaContextPtr, OUTPUTITER, const T *, const T *, const int);

#define TRANS_FLOAT(TRANS) TRANS, float
RPU_CMU_DEFINE_AWI(float *, float);
RPU_CMU_DEFINE_AWI(PermuterTransOutputIterator<float>, float);
RPU_CMU_DEFINE_AWI(IndexReaderOutputIterator<float>, float);
RPU_CMU_DEFINE_AWI(IndexReaderTransOutputIterator<float>, float);
RPU_CMU_DEFINE_AWI(IndexReaderSliceOutputIterator<TRANS_FLOAT(true)>, float);
RPU_CMU_DEFINE_AWI(IndexReaderSliceOutputIterator<TRANS_FLOAT(false)>, float);
RPU_CMU_DEFINE_AWI(SliceOutputIterator<TRANS_FLOAT(true)>, float);
RPU_CMU_DEFINE_AWI(SliceOutputIterator<TRANS_FLOAT(false)>, float);

#undef TRANS_FLOAT

#ifdef RPU_USE_DOUBLE
#define TRANS_DOUBLE(TRANS) TRANS, double
RPU_CMU_DEFINE_AWI(double *, double);
RPU_CMU_DEFINE_AWI(PermuterTransOutputIterator<double>, double);
RPU_CMU_DEFINE_AWI(IndexReaderOutputIterator<double>, double);
RPU_CMU_DEFINE_AWI(IndexReaderTransOutputIterator<double>, double);
RPU_CMU_DEFINE_AWI(IndexReaderSliceOutputIterator<TRANS_DOUBLE(true)>, double);
RPU_CMU_DEFINE_AWI(IndexReaderSliceOutputIterator<TRANS_DOUBLE(false)>, double);
RPU_CMU_DEFINE_AWI(SliceOutputIterator<TRANS_DOUBLE(true)>, double);
RPU_CMU_DEFINE_AWI(SliceOutputIterator<TRANS_DOUBLE(false)>, double);
#undef TRANS_DOUBLE
#endif

#undef RPU_CMU_DEFINE_CWI

// copyWithIterator
template <typename OutputIteratorT, typename InputIteratorT>
__global__ void kernelCopyWithIterator(
    OutputIteratorT out_tensor, const InputIteratorT in_tensor, const int sz_all) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, sz_all) { out_tensor[idx] = in_tensor[idx]; }
}

template <typename OutputIteratorT, typename InputIteratorT>
void copyWithIterator(
    const CudaContextPtr context,
    OutputIteratorT out_tensor,
    InputIteratorT in_tensor,
    const int total_input_size) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(total_input_size, nthreads);

  kernelCopyWithIterator<OutputIteratorT, InputIteratorT>
      <<<nblocks, nthreads, 0, context->getStream()>>>(out_tensor, in_tensor, total_input_size);
}

#define RPU_CMU_DEFINE_CWI(OUTPUTITER, INPUTITER)                                                  \
  template void copyWithIterator<OUTPUTITER, INPUTITER>(                                           \
      const CudaContextPtr, OUTPUTITER, INPUTITER, const int);
#define TRANS_FLOAT(TRANS) TRANS, float
#define COMMA ,

RPU_CMU_DEFINE_CWI(float *, const float *);
RPU_CMU_DEFINE_CWI(float *, float *);
RPU_CMU_DEFINE_CWI(float *, IndexReaderInputIterator<float>);
RPU_CMU_DEFINE_CWI(float *, IndexReaderTransInputIterator<float>);
RPU_CMU_DEFINE_CWI(float *, PermuterTransInputIterator<float>);
RPU_CMU_DEFINE_CWI(float *, IndexReaderSliceInputIterator<TRANS_FLOAT(true)>);
RPU_CMU_DEFINE_CWI(float *, IndexReaderSliceInputIterator<TRANS_FLOAT(false)>);
RPU_CMU_DEFINE_CWI(float *, SliceInputIterator<TRANS_FLOAT(true)>);
RPU_CMU_DEFINE_CWI(float *, SliceInputIterator<TRANS_FLOAT(false)>);
RPU_CMU_DEFINE_CWI(float *, DiagInputIterator<float COMMA chop_t>);
RPU_CMU_DEFINE_CWI(float *, EyeInputIterator<float>);
RPU_CMU_DEFINE_CWI(PermuterTransOutputIterator<float>, const float *);
RPU_CMU_DEFINE_CWI(IndexReaderOutputIterator<float>, const float *);
RPU_CMU_DEFINE_CWI(IndexReaderTransOutputIterator<float>, const float *);
RPU_CMU_DEFINE_CWI(IndexReaderSliceOutputIterator<TRANS_FLOAT(true)>, const float *);
RPU_CMU_DEFINE_CWI(IndexReaderSliceOutputIterator<TRANS_FLOAT(false)>, const float *);
RPU_CMU_DEFINE_CWI(SliceOutputIterator<TRANS_FLOAT(true)>, const float *);
RPU_CMU_DEFINE_CWI(SliceOutputIterator<TRANS_FLOAT(false)>, const float *);

#undef TRANS_FLOAT

#ifdef RPU_USE_DOUBLE
#define TRANS_DOUBLE(TRANS) TRANS, double
RPU_CMU_DEFINE_CWI(double *, const double *);
RPU_CMU_DEFINE_CWI(double *, double *);
RPU_CMU_DEFINE_CWI(double *, IndexReaderInputIterator<double>);
RPU_CMU_DEFINE_CWI(double *, IndexReaderTransInputIterator<double>);
RPU_CMU_DEFINE_CWI(double *, PermuterTransInputIterator<double>);
RPU_CMU_DEFINE_CWI(double *, IndexReaderSliceInputIterator<TRANS_DOUBLE(true)>);
RPU_CMU_DEFINE_CWI(double *, IndexReaderSliceInputIterator<TRANS_DOUBLE(false)>);
RPU_CMU_DEFINE_CWI(double *, SliceInputIterator<TRANS_DOUBLE(true)>);
RPU_CMU_DEFINE_CWI(double *, SliceInputIterator<TRANS_DOUBLE(false)>);
RPU_CMU_DEFINE_CWI(double *, DiagInputIterator<double COMMA chop_t>);
RPU_CMU_DEFINE_CWI(double *, EyeInputIterator<double>);
RPU_CMU_DEFINE_CWI(PermuterTransOutputIterator<double>, const double *);
RPU_CMU_DEFINE_CWI(IndexReaderOutputIterator<double>, const double *);
RPU_CMU_DEFINE_CWI(IndexReaderTransOutputIterator<double>, const double *);
RPU_CMU_DEFINE_CWI(IndexReaderSliceOutputIterator<TRANS_DOUBLE(true)>, const double *);
RPU_CMU_DEFINE_CWI(IndexReaderSliceOutputIterator<TRANS_DOUBLE(false)>, const double *);
RPU_CMU_DEFINE_CWI(SliceOutputIterator<TRANS_DOUBLE(true)>, const double *);
RPU_CMU_DEFINE_CWI(SliceOutputIterator<TRANS_DOUBLE(false)>, const double *);

#undef TRANS_DOUBLE
#endif

#undef RPU_CMU_DEFINE_CWI

// fake cast to overcome constexpr lacking
template <> const float *fakeCastConst<float, const float *>(const float *X) { return X; };
template <> float *fakeCast<float, float *>(float *X) { return X; };
#ifdef RPU_USE_DOUBLE
template <> const double *fakeCastConst<double, const double *>(const double *X) { return X; };
template <> double *fakeCast<double, double *>(double *X) { return X; };
#endif

} // namespace math
} // namespace RPU
