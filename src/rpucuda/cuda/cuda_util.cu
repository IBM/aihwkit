#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021, 2022 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "cuda_math_util.h"
#include "cuda_util.h"
#include "utility_functions.h"
#include <algorithm>
#include <chrono>
#include <iostream>
#include <memory>

#define DISABLE_SHARED_MUTEX 1

#define IDX2F(i, j, ld) ((((j)-1) * (ld)) + ((i)-1))

// this should be not necesary, because device id is set individually
// per thread. However, if one would want to use 2 GPUs within one
// thread, one needs it.
#define RPU_EXPLICIT_ENFORCE_DEVICE_ID

#define SUBTRACTMEMCOUNTER(BYTES)                                                                  \
  {                                                                                                \
    std::lock_guard<std::mutex> lock(rpu_global_mem_counter_mutex);                                \
    rpu_global_mem_counter -= BYTES;                                                               \
  }
#define ADDTOMEMCOUNTER(BYTES)                                                                     \
  {                                                                                                \
    std::lock_guard<std::mutex> lock(rpu_global_mem_counter_mutex);                                \
    rpu_global_mem_counter += BYTES;                                                               \
  }

int64_t rpu_global_mem_counter = 0;
std::mutex rpu_global_mem_counter_mutex;

namespace RPU {

__global__ void kernelCurandSetup(unsigned long long rseed, hiprandState_t *state, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed, a different sequence
     number, no offset */
  if (id < n) {
    hiprand_init(rseed, id, 0, &state[id]);
  }
}

__global__ void kernelCurandSetupSameSeed(unsigned long long rseed, hiprandState_t *state, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    hiprand_init(rseed, 0, 0, &state[id]);
  }
}

void curandSetup(CudaArray<hiprandState_t> &dev_states, unsigned long long rseed, bool same_seed) {
  unsigned long long seed = rseed;

  if (rseed == 0) {
    seed = (unsigned long long)std::chrono::high_resolution_clock::now().time_since_epoch().count();
  } else {
    seed = rseed;
  }
  CudaContextPtr c = dev_states.getContext();
  int m = dev_states.getSize();
  int nthreads = c->getNThreads();
  int nblocks = c->getNBlocks(m, nthreads);
  if (same_seed) {
    kernelCurandSetupSameSeed<<<nblocks, nthreads, 0, c->getStream()>>>(
        seed, dev_states.getData(), m);
  } else {
    kernelCurandSetup<<<nblocks, nthreads, 0, c->getStream()>>>(seed, dev_states.getData(), m);
  }

  c->synchronize();
}

void curandSetup(
    CudaContextPtr c,
    std::unique_ptr<CudaArray<hiprandState_t>> &dev_states,
    int n,
    unsigned long long rseed,
    bool same_seed) {
  int m = (n + 31) / 32 * 32;
  c->synchronizeDevice();
  dev_states = RPU::make_unique<CudaArray<hiprandState_t>>(c, m);
  curandSetup(*dev_states, rseed, same_seed);
}

CublasEnvironment::~CublasEnvironment() {

  DEBUG_OUT("Destroy BLAS env.");
  // DEBUG_OUT("handle : " <<this->handle_);

  // destroy device

  // destroy host
  if (handle_ != nullptr) {
    hipblasDestroy(handle_);
    DEBUG_OUT("CUBLAS destroyed");
  }
#ifdef RPU_WITH_CUBLAS_DEVICE
  if (device_handle_created_) {
    DEBUG_OUT("destroy device handle");
    kernelCublasDestroy<<<1, 1>>>(device_handle_);
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipFree(device_handle_));
    DEBUG_OUT("CUBLAS device destroyed");
  }
#endif
  // hipDeviceReset();
}

CublasEnvironment::CublasEnvironment(int gpu_id) {

  DEBUG_OUT("GET BLAS env.");
  if (gpu_id >= 0) {
    CUDA_CALL(hipSetDevice(gpu_id));
  }

  // create host
  hipblasStatus_t stat = hipblasCreate(&handle_);
  CUDA_CALL(hipDeviceSynchronize());

  // DEBUG_CALL(this->test(););
  // DEBUG_OUT("handle : " <<handle_);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    RPU_FATAL("CUBLAS initialization failed");
  } else
    DEBUG_OUT("CUBLAS Host initialized.");

#ifdef RPU_WITH_CUBLAS_DEVICE
  device_handle_created_ = false;
#endif
}

void CublasEnvironment::test() {
  this->runTest();
#ifdef RPU_WITH_CUBLAS_DEVICE
  if (device_handle_created_) {
    this->runTestDevice();
  }
#endif
}

static __inline__ void
modifyS(hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta) {
  hipblasSscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}

int CublasEnvironment::runTest() {

  // make a test run
  hipblasStatus_t stat;

  int i, j;
  int M = 5;
  int N = 6;
  float *devPtrA;
  float *a = 0;
  a = (float *)malloc(M * N * sizeof(*a));
  if (!a) {
    std::cout << "CUBLAS test run failed (malloc)\n";
    return 1;
  }
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      a[IDX2F(i, j, M)] = (float)((i - 1) * M + j);
    }
  }

  if (hipMalloc((void **)&devPtrA, M * N * sizeof(*a)) != hipSuccess) {
    std::cerr << "CUBLAS test run failed (hipMalloc)\n";
    free(a);
    return 1;
  }

  modifyS(handle_, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
  stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS test run failed (data download)\n";
    CUDA_CALL(hipFree(devPtrA));
    free(a);
    return 1;
  }
  stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS test run failed (data upload)\n";
    CUDA_CALL(hipFree(devPtrA));
    free(a);
    return 1;
  }
  CUDA_CALL(hipFree(devPtrA));
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      std::cout << a[IDX2F(i, j, M)] << ",";
    }
    std::cout << std::endl;
  }

  free(a);
  std::cout << "CUBLAS test run successful.\n";

  return 0;
}

#ifdef RPU_WITH_CUBLAS_DEVICE

__global__ void kernelCublasDestroy(hipblasHandle_t *device_handle) {

  hipblasStatus_t status = hipblasDestroy(*device_handle);
  hipDeviceSynchronize();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("ERROR in destroying cublas device!\n");
  }
}

__global__ void kernelCublasCreateDevice(hipblasHandle_t *device_handle) {

  hipblasStatus_t status = hipblasCreate(device_handle);

  hipDeviceSynchronize();

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("ERROR in creating cublas device!\n");
    return;
  }
}

void CublasEnvironment::createDeviceHandle() {

  if (device_handle_created_)
    return;

  CUDA_CALL(hipMalloc(&device_handle_, sizeof(hipblasHandle_t)));
  CUDA_CALL(hipDeviceSynchronize());

  kernelCublasCreateDevice<<<1, 1>>>(device_handle_);
  CUDA_CALL(hipDeviceSynchronize());

  DEBUG_OUT("Created device handle");

  device_handle_created_ = true;
}

hipblasHandle_t *CublasEnvironment::getDeviceHandle() {
  if (!device_handle_created_) {
    this->createDeviceHandle();
  }
  return device_handle_;
}

__global__ void kernelCublasTest(hipblasHandle_t *device_handle, float *source, float *dest) {

  hipblasStatus_t status = hipblasScopy(*device_handle, 1, source, 1, dest, 1);
  hipDeviceSynchronize();

  if ((status != HIPBLAS_STATUS_SUCCESS)) {
    printf("Some problems with the CuBLAS device test.\n");
  }
}

int CublasEnvironment::runTestDevice() {

  float one = 1;
  float zero = 0;
  float *a;
  float *b;

  CUDA_CALL(hipMalloc(&a, sizeof(float)));
  CUDA_CALL(hipMalloc(&b, sizeof(float)));
  CUDA_CALL(hipMemcpy(a, &one, sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(b, &zero, sizeof(float), hipMemcpyHostToDevice));

  kernelCublasTest<<<1, 1>>>(device_handle_, a, b);
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(&zero, b, sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(a));
  CUDA_CALL(hipFree(b));

  if (zero == 1) {
    std::cout << "CuBLAS device test succeded\n";
    return 0;
  } else {
    std::cerr << "ERROR in CuBLAS device test\n";
    return 1;
  }
}

#endif

//**********************************************************************//

template <typename T> T *CudaBuffer<T>::get(CudaContextPtr c, int size) {
  mutex_.lock(); // need to be explicitely released to avoid multi-threading issues
  if (buffer_ == nullptr || buffer_->getSize() < size || &*(buffer_->getContext()) != &*c) {
    if (buffer_ != nullptr) {
      buffer_->synchronize();
    }
    buffer_ = RPU::make_unique<CudaArray<T>>(c, size);
    c->synchronize();
  }
  return buffer_->getData();
}

template <typename T> void CudaBuffer<T>::release() { mutex_.unlock(); }

// copy constructor
template <typename T> CudaBuffer<T>::CudaBuffer(const CudaBuffer<T> &other) {
  if (other.buffer_ != nullptr) {
    buffer_ = RPU::make_unique<CudaArray<T>>(*other.buffer_);
    buffer_->synchronize();
  }
}

// copy assignment
template <typename T> CudaBuffer<T> &CudaBuffer<T>::operator=(const CudaBuffer &other) {
  CudaBuffer tmp(other);
  swap(*this, tmp);
  if (tmp.buffer_ != nullptr) {
    tmp.buffer_->context_.synchronize();
  }
  return *this;
}

// move constructor
template <typename T> CudaBuffer<T>::CudaBuffer(CudaBuffer<T> &&other) {
  { const std::lock_guard<std::recursive_mutex> lock(other.mutex_); }
  *this = std::move(other);
}

// move assignment
template <typename T> CudaBuffer<T> &CudaBuffer<T>::operator=(CudaBuffer<T> &&other) {

  const std::lock_guard<std::recursive_mutex> lock(other.mutex_);
  buffer_ = std::move(other.buffer_);
  return *this;
}

//**********************************************************************//
void CudaContext::init() {
  DEBUG_OUT("Init context...");

  if (gpu_id_ >= 0) {
    CUDA_CALL(hipSetDevice(gpu_id_));
  } else {
    CUDA_CALL(hipGetDevice(&gpu_id_));
  }
  CUDA_CALL(hipDeviceSynchronize());
  DEBUG_OUT("Create context on GPU " << gpu_id_);
  env_ = new CublasEnvironment(gpu_id_);
  stream_id_ = 0;
  rng_created_ = false;
  shared_ = false;
  non_blocking_ = true;

  CUDA_CALL(hipEventCreate(&event_));

  prop_ = new hipDeviceProp_t();
  CUDA_CALL(hipGetDeviceProperties(prop_, gpu_id_));
  CUDA_CALL(hipDeviceSynchronize());
}

CudaContext::CudaContext(int gpu_id, bool non_blocking)
    : gpu_id_(gpu_id), non_blocking_(non_blocking) {
  this->init();
  this->getStream(0);
}

CudaContext::CudaContext(hipStream_t shared_stream, int gpu_id) : gpu_id_(gpu_id) {
  DEBUG_OUT("Create context on GPU " << gpu_id << " with shared stream (on id 0)\n");
  this->init();

  shared_ = true;
  streams_.push_back(shared_stream);
  stream_id_ = 0;

  shared_stream_id_ = 0;
  shared_streams_.push_back(shared_stream);
}

CudaContext::~CudaContext() {
  DEBUG_OUT("Destroy CudaContext...");

  shared_random_states_.clear();
  shared_float_buffer_.clear();
  shared_double_buffer_.clear();

  random_states_.clear();
  float_buffer_.clear();
  double_buffer_.clear();

  int i_start = shared_ ? 1 : 0;
  for (int i = i_start; i < streams_.size(); i++) {
    hipStreamSynchronize(streams_[i]);
    hipStreamDestroy(streams_[i]);
  }

  if (event_ != nullptr) {
    hipEventDestroy(event_);
    event_ = nullptr;
  }

  if (rng_created_) {
    hiprandDestroyGenerator(rng_);
    rng_created_ = false;
  }

  if (prop_ != nullptr) {
    delete prop_;
    prop_ = nullptr;
  }
  if (env_ != nullptr) {
    delete env_;
    env_ = nullptr;
  }
  DEBUG_OUT("Destroyed CudaContext.");
}

// copy constructor
CudaContext::CudaContext(const CudaContext &other) {
  // NOTE: changed to non-shared copy

  gpu_id_ = other.gpu_id_;
  this->init();

  shared_ = other.shared_;
  non_blocking_ = other.non_blocking_;

  if (other.shared_ && other.streams_.size() > 0) {
    streams_.push_back(other.streams_[0]);
  }
  shared_streams_ = other.shared_streams_;
  shared_stream_id_ = other.shared_stream_id_;

  for (int i = other.shared_ ? 1 : 0; i < other.streams_.size(); i++) {
    // rest are new streams!!
    this->getStream(i);
  }

  stream_id_ = other.stream_id_;

  if (other.rng_created_) {
    this->createRandomGenerator();
  }

  // random states and buffers won't be copied. They will be created a new

  DEBUG_OUT("CudaContext copy constructed [but only first stream shared. New streams and event!].");
}

// copy assignment
CudaContext &CudaContext::operator=(const CudaContext &other) {
  CudaContext tmp(other);
  swap(*this, tmp);
  synchronize();
  return *this;
}

// move constructor
CudaContext::CudaContext(CudaContext &&other) { *this = std::move(other); }

// move assignment
CudaContext &CudaContext::operator=(CudaContext &&other) {

  gpu_id_ = other.gpu_id_;
  stream_id_ = other.stream_id_;
  shared_stream_id_ = other.shared_stream_id_;
  shared_ = other.shared_;
  non_blocking_ = other.non_blocking_;

  prop_ = other.prop_;
  other.prop_ = nullptr;

  streams_ = std::move(other.streams_);
  shared_streams_ = std::move(other.shared_streams_);

  env_ = other.env_;
  other.env_ = nullptr;

  rng_ = other.rng_;
  other.rng_ = nullptr;

  rng_created_ = other.rng_created_;

  event_ = other.event_;
  other.event_ = nullptr;

  random_states_ = std::move(other.random_states_);
  shared_random_states_ = std::move(other.shared_random_states_);

  shared_float_buffer_ = std::move(other.shared_float_buffer_);
  shared_double_buffer_ = std::move(other.shared_double_buffer_);

  float_buffer_ = std::move(other.float_buffer_);
  double_buffer_ = std::move(other.double_buffer_);

  return *this;
}

void CudaContext::synchronizeContext() const {
  enforceDeviceId();
  for (int i = 0; i < streams_.size(); i++) {
    CUDA_CALL(hipStreamSynchronize(streams_[i]));
  }
}

void CudaContext::enforceDeviceId() const {
#ifdef RPU_EXPLICIT_ENFORCE_DEVICE_ID
  int gpu_id;
  CUDA_CALL(hipGetDevice(&gpu_id));
  if (gpu_id != gpu_id_) {
    std::cout << "WARNING wrong device detected: " << gpu_id << " vs. " << gpu_id_ << std::endl;
    CUDA_CALL(hipSetDevice(gpu_id_));
  }
#endif
}

void CudaContext::synchronizeDevice() const {
  enforceDeviceId();
  CUDA_CALL(hipDeviceSynchronize());
}

void CudaContext::synchronizeWith(CudaContextPtr c) const {

  if (this->getStream() == c->getStream()) {
    // do nothing since work on the same stream
  } else {
    this->synchronize();
    c->synchronize();
  }
}

void CudaContext::synchronizeWith(CudaContextPtr ca, CudaContextPtr cb) const {

  if (ca->getStream() != cb->getStream()) {
    ca->synchronizeWith(cb);
  }
  if (ca->getStream() != this->getStream()) {
    this->synchronize();
  }
}

void CudaContext::synchronizeStream(int idx) const {
  DEBUG_OUT("Synchronize stream idx " << idx);
  enforceDeviceId();
  if ((idx >= 0) && (idx < streams_.size())) {
    CUDA_CALL(hipStreamSynchronize(streams_[idx]));
  }
}
void CudaContext::synchronizeStream() const {
  DEBUG_OUT("Synchronize stream id " << stream_id_);
  enforceDeviceId();
  CUDA_CALL(hipStreamSynchronize(streams_[stream_id_]));
}

int CudaContext::getNBlocks(int size, int nthreads) const {
  DEBUG_OUT("get NBlocks for  size " << size);
  return (size + nthreads - 1) / nthreads;
}

int CudaContext::getNStrideBlocks(int size, int nthreads) const {
  DEBUG_OUT("get N Stride Blocks for  size " << size);
  int max_blocks = getSMCount() * maxThreadsPerBlock() / nthreads;
  return MIN(getNBlocks(size, nthreads), max_blocks);
}

hipStream_t CudaContext::getStream(int idx) {

  enforceDeviceId();

  DEBUG_OUT("Try to get streams " << idx);
  if ((idx >= 0) && (idx < streams_.size())) {
    if (stream_id_ != idx) {
      stream_id_ = idx;
      CUBLAS_CALL(hipblasSetStream(this->getBlasHandle(), streams_[idx]));
    }
    return streams_[idx];
  } else if (streams_.size() == idx) {

    hipStream_t s;
    if (non_blocking_) {
      CUDA_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    } else {
      CUDA_CALL(hipStreamCreate(&s));
    }
    streams_.push_back(s);

    stream_id_ = idx;
    CUBLAS_CALL(hipblasSetStream(this->getBlasHandle(), streams_[idx]));

    DEBUG_OUT("Created stream id " << idx << " at : " << streams_[idx] << " ( s: " << s << ")");
    return streams_[idx];
  } else {
    RPU_FATAL("Requested stream size mismatch.");
  }
}

void CudaContext::setExternalStream(hipStream_t s) {
  if (shared_) {

#ifndef DISABLE_SHARED_MUTEX
    shared_mutex_.lock();
#endif

    enforceDeviceId();
    if (s != streams_[stream_id_]) {
      if (stream_id_ != 0) {
        this->synchronizeDevice();
      } else {
        this->synchronizeStream();
      }
      CUBLAS_CALL(hipblasSetStream(this->getBlasHandle(), s));
    }
    streams_[0] = s;
    stream_id_ = 0;

    auto it = std::find(shared_streams_.begin(), shared_streams_.end(), s);
    if (it != shared_streams_.end()) {
      shared_stream_id_ = it - shared_streams_.begin();
    } else {
      shared_streams_.push_back(s);
      shared_stream_id_ = shared_streams_.size() - 1;
    }
  } else {
    RPU_FATAL("setExternalStream: must be a shared context.");
  }
}

void CudaContext::releaseExternalStream() {
  if (shared_) {

#ifndef DISABLE_SHARED_MUTEX
    shared_mutex_.unlock();
#endif
  }
}

void CudaContext::createRandomGenerator() {
  if (!rng_created_) {
    enforceDeviceId();
    CURAND_CALL(hiprandCreateGenerator(&rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetStream(rng_, this->getStream()));
    rng_created_ = true;
  }
}

void CudaContext::setRandomSeed(unsigned long long rseed) {
  enforceDeviceId();

  if (!rng_created_) {
    this->createRandomGenerator();
  }

  unsigned long long seed = rseed;
  if (rseed == 0) {
    seed = (unsigned long long)std::chrono::high_resolution_clock::now().time_since_epoch().count();
  } else {
    seed = rseed;
  }
  CURAND_CALL(hiprandSetStream(rng_, this->getStream()));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(rng_, seed));
  this->synchronizeStream();
}

void CudaContext::randNormal(float *dev_array, int size, float mean, float stddev) {
  if (!rng_created_) {
    setRandomSeed(0); // will create random generator on the fly
  }

  if (stddev > 0) {
    CURAND_CALL(hiprandSetStream(rng_, this->getStream()));
    CURAND_CALL(hiprandGenerateNormal(rng_, dev_array, size, mean, stddev));
  } else {
    RPU::math::elemconst(this, dev_array, size, mean);
  }
}

void CudaContext::randUniform(float *dev_array, int size) {

  if (!rng_created_) {
    setRandomSeed(0);
  }
  CURAND_CALL(hiprandSetStream(rng_, this->getStream()));
  CURAND_CALL(hiprandGenerateUniform(rng_, dev_array, size));
}

hiprandState_t *CudaContext::getRandomStates(int size) {

  int n = size;
  if (n <= 0) {
    n = getSMCount() * maxThreadsPerBlock();
  }

  auto *rs = &random_states_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    rs = &shared_random_states_;
    stream_id = shared_stream_id_;
  }

  if (rs->size() <= stream_id) {
    rs->resize(stream_id + 1);
  }
  if (!(*rs)[stream_id] || (n > (*rs)[stream_id]->getSize())) {
    curandSetup(this, (*rs)[stream_id], n, 0, false);
  }
  return (*rs)[stream_id]->getData();
}

template <> float *CudaContext::getSharedBuffer<float>(int id, int size) {

  auto *buffer = &float_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_float_buffer_;
    stream_id = shared_stream_id_;
    DEBUG_OUT("Get SHARED float buffer ID " << id << ", size " << size << ", stream " << stream_id);
  } else {
    DEBUG_OUT("Get float buffer ID " << id << ", size " << size << ", stream " << stream_id);
  }

  while (buffer->size() <= stream_id) {
    buffer->push_back(std::vector<CudaBuffer<float>>{RPU_MAX_BUFFER});
  }
  return (*buffer)[stream_id][id].get(this, size);
}

template <> void CudaContext::releaseSharedBuffer<float>(int id) {

  auto *buffer = &float_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_float_buffer_;
    stream_id = shared_stream_id_;
    DEBUG_OUT("Release SHARED float buffer ID " << id << ", stream " << stream_id);
  } else {
    DEBUG_OUT("Release float buffer ID " << id << ", stream " << stream_id);
  }

  (*buffer)[stream_id][id].release();
}

template <> void CudaContext::printSharedBuffer<float>(int id, int size) {

  auto *buffer = &float_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_float_buffer_;
    stream_id = shared_stream_id_;
  }
  const auto *array = (*buffer)[stream_id][id].getCudaArray();
  if (!array) {
    return;
  }
  RPU_INFO("Float buffer " << id);
  array->printValues(size);
}

template <> double *CudaContext::getSharedBuffer<double>(int id, int size) {
  // somehow this needs to be a MAX_BUFFER vector to avoid dynamical
  // resizing. Not sure why, but dynamical allocation of the
  // CudaBuffer vector elements does not work without uniptr (which
  // then has sync problems)

  auto *buffer = &double_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_double_buffer_;
    stream_id = shared_stream_id_;
  }

  while (buffer->size() <= stream_id) {
    buffer->push_back(std::vector<CudaBuffer<double>>{RPU_MAX_BUFFER});
  }
  return (*buffer)[stream_id][id].get(this, size);
}

template <> void CudaContext::releaseSharedBuffer<double>(int id) {

  auto *buffer = &double_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_double_buffer_;
    stream_id = shared_stream_id_;
  }

  (*buffer)[stream_id][id].release();
}

template <> void CudaContext::printSharedBuffer<double>(int id, int size) {

  auto *buffer = &double_buffer_;
  auto stream_id = stream_id_;
  if (shared_ && stream_id_ == 0) {
    buffer = &shared_double_buffer_;
    stream_id = shared_stream_id_;
  }

  const auto *array = (*buffer)[stream_id][id].getCudaArray();
  if (!array) {
    return;
  }
  RPU_INFO("Double buffer " << id);
  array->printValues(size);
}

void CudaContext::recordWaitEvent(CudaContextPtr wait_on_context) {
  this->recordWaitEvent(wait_on_context->getStream(), wait_on_context->getEvent());
}
void CudaContext::recordEvent() { CUDA_CALL(hipEventRecord(event_, streams_[stream_id_])); }
void CudaContext::waitEvent(hipEvent_t wait_on_event) {
  CUDA_CALL(hipStreamWaitEvent(streams_[stream_id_], wait_on_event, 0));
}

void CudaContext::waitEvent(CudaContextPtr wait_on_context) {
  waitEvent(wait_on_context->getEvent());
}

void CudaContext::recordWaitEvent(hipStream_t s) { this->recordWaitEvent(s, event_); }

void CudaContext::recordWaitEvent(hipStream_t s, hipEvent_t e) {
  if (streams_[stream_id_] != s) {
    CUDA_CALL(hipEventRecord(e, s));
    CUDA_CALL(hipStreamWaitEvent(streams_[stream_id_], e, 0));
  }
}

//**********************************************************************//

template <typename T>
CudaArray<T>::CudaArray(CudaContextPtr c)
    : size_(0), width_(0), height_(1), pitch_(0), context_(c) {}

template <typename T> CudaArray<T>::CudaArray(CudaContextPtr c, int n) : CudaArray(c) {
  size_ = n;
  width_ = n;
  height_ = 1; // this needs to be one! No height>1 supported yet
  if (n > 0) {
    context_->enforceDeviceId();
    CUDA_CALL(hipMallocPitch(&values_, &pitch_, n * sizeof(T), height_));
    ADDTOMEMCOUNTER(size_ * sizeof(T));
  }
}

template <typename T>
CudaArray<T>::CudaArray(CudaContextPtr c, int n, const T *host_array) : CudaArray(c, n) {
  if (n > 0) {
    this->assign(host_array);
    context_->synchronize(); // better syncrhonize. Constructing is considered slow anyway
    ADDTOMEMCOUNTER(size_ * sizeof(T));
  }
}

template <typename T>
CudaArray<T>::CudaArray(CudaContextPtr c, const std::vector<T> &host_vector)
    : CudaArray(c, host_vector.size()) {
  size_t n = host_vector.size();
  if (n > 0) {
    this->assign(host_vector.data());
    context_->synchronize(); // better syncrhonize. Constructing is considered slow anyway
    ADDTOMEMCOUNTER(size_ * sizeof(T));
  }
}

template <typename T> CudaArray<T>::~CudaArray() {
  if (values_ == nullptr) {
    return;
  }

  if ((size_ > 0) && (values_ != nullptr) && (!shared_if_)) {
    // hipDeviceSynchronize(); // too much?
    SUBTRACTMEMCOUNTER(size_ * sizeof(T));
    hipFree(values_);
    values_ = nullptr;
    size_ = 0;
    width_ = 0;
  }

  values_ = nullptr;
}

// copy constructor
template <typename T> CudaArray<T>::CudaArray(const CudaArray<T> &other) {
  size_ = other.size_;
  width_ = other.width_;
  height_ = other.height_;
  pitch_ = other.pitch_;
  context_ = other.context_;
  values_ = nullptr;

  if (size_ > 0) {
    context_->enforceDeviceId();

    if (other.shared_if_) {
      this->setShared(other.values_);
    } else {
      CUDA_CALL(hipMallocPitch(&values_, &pitch_, size_ * sizeof(T), height_));
      this->assign(other);
      ADDTOMEMCOUNTER(size_ * sizeof(T));
    }
    context_->synchronize(); // better synchronize. Constructing is slow anyway
  }

  DEBUG_OUT("CudaArray copy constructed.");
}

// copy assignment
template <typename T> CudaArray<T> &CudaArray<T>::operator=(const CudaArray<T> &other) {
  CudaArray<T> tmp(other);
  swap(*this, tmp);
  if (size_ > 0) {
    context_->synchronize(); // need sync because of tmp
  }
  ADDTOMEMCOUNTER(size_ * sizeof(T));
  return *this;
}

// move constructor
template <typename T> CudaArray<T>::CudaArray(CudaArray<T> &&other) { *this = std::move(other); }

// move assignment
template <typename T> CudaArray<T> &CudaArray<T>::operator=(CudaArray<T> &&other) {

  size_ = other.size_;
  other.size_ = 0;

  width_ = other.width_;
  other.width_ = 0;

  height_ = other.height_;
  other.height_ = 0;

  pitch_ = other.pitch_;
  other.pitch_ = 0;

  context_ = other.context_;
  other.context_ = nullptr;

  values_ = other.values_;
  other.values_ = nullptr;

  shared_if_ = other.shared_if_;

  return *this;
}

template <typename T> void CudaArray<T>::setConst(T set_value) {

  DEBUG_OUT(
      "Set (hsize,P,W,H): " << size_ << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                            << height_);
  if (size_ > 0) {
    context_->enforceDeviceId();
    if (set_value != 0) {
      RPU::math::elemconst(context_, values_, size_, set_value);
    } else {
      CUDA_CALL(hipMemset2DAsync(
          values_, pitch_, 0, this->getWidthBytes(), height_, context_->getStream()));
    }
  }
}

template <> void CudaArray<hiprandStateXORWOW>::setConst(hiprandStateXORWOW set_value) {
  RPU_FATAL("Cannot set curandstates to some values.");
}

template <> void CudaArray<double *>::setConst(double *set_value) {
  RPU_FATAL("Cannot set pointer types to some values.");
}

template <> void CudaArray<float *>::setConst(float *set_value) {
  RPU_FATAL("Cannot set pointer types to some values.");
}

template <typename T> void CudaArray<T>::printValues(int nmax) const {
  T *values = new T[size_];
  this->copyTo(values); // will synchronize
  int n = nmax > 0 ? MIN(nmax, size_) : size_;
  for (int i = 0; i < n; ++i) {
    std::cout << "[" << i << "]:" << values[i] << ", ";
  }
  if (n < size_) {
    std::cout << "...";
  }
  std::cout << std::endl;
  delete[] values;
}

template <typename T> void CudaArray<T>::printNZValues(int nmax) const {
  T *values = new T[size_];
  this->copyTo(values); // will synchronize
  int n = nmax > 0 ? MIN(nmax, size_) : size_;
  for (int i = 0; i < n; ++i) {
    if (values[i] != (T)0) {
      std::cout << "[" << i << "]:" << values[i] << ", ";
    }
  }
  if (n < size_) {
    std::cout << "...";
  }
  std::cout << std::endl;
  delete[] values;
}

template <> void CudaArray<hiprandStateXORWOW>::printValues(int nmax) const {
  RPU_FATAL("Cannot print curandstates.");
}
template <> void CudaArray<hiprandStateXORWOW>::printNZValues(int nmax) const {
  RPU_FATAL("Cannot print curandstates.");
}

template <> void CudaArray<int8_t>::printValues(int nmax) const {
  int8_t *values = new int8_t[size_];
  this->copyTo(values); // will synchronize
  int n = nmax > 0 ? MIN(nmax, size_) : size_;
  for (int i = 0; i < n; ++i) {
    std::cout << "[" << i << "]:" << static_cast<int>(values[i]) << ", ";
  }
  if (n < size_) {
    std::cout << "...";
  }
  std::cout << std::endl;
  delete[] values;
}

template <> void CudaArray<int8_t>::printNZValues(int nmax) const {
  int8_t *values = new int8_t[size_];
  this->copyTo(values); // will synchronize
  int n = nmax > 0 ? MIN(nmax, size_) : size_;
  for (int i = 0; i < n; ++i) {
    if (values[i] != 0) {
      std::cout << "[" << i << "]:" << static_cast<int>(values[i]) << ", ";
    }
  }
  if (n < size_) {
    std::cout << "...";
  }
  std::cout << std::endl;
  delete[] values;
}

template <typename T> void CudaArray<T>::assign(const T *host_array) {
  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Assign host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                    << height_);
  if (size_ > 0) {
    context_->enforceDeviceId();
    context_->synchronize();
    CUDA_CALL(hipMemcpy2DAsync(
        values_, pitch_, host_array, sz, sz, 1, hipMemcpyHostToDevice, context_->getStream()));
  }
}

template <typename T>
void CudaArray<T>::assignTranspose(const T *host_array, const int m, const int n) {
  // col major to row major
  if (size_ <= 0) {
    return;
  }

  if (m * n != size_) {
    RPU_FATAL("Size mismatch");
  }
  T *transposed_array = new T[size_];
  for (int i = 0; i < size_; i++) {
    int i_col = (i % n);
    int i_row = (i / n);
    transposed_array[i_col * m + i_row] = host_array[i];
  }
  context_->enforceDeviceId();
  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Assign host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                    << height_);
  context_->synchronize();
  CUDA_CALL(hipMemcpy2D(
      values_, pitch_, transposed_array, sz, sz, 1, hipMemcpyHostToDevice)); // no async
  delete[] transposed_array;
}

template <typename T> void CudaArray<T>::assign(const CudaArray<T> &source) {
  DEBUG_OUT(
      "Assign from CudaArray (S,P,W,H): " << size_ << ", " << pitch_ << ", " << width_ * sizeof(T)
                                          << ", " << height_);
  if (source.getSize() != size_) {
    RPU_FATAL("Assignment of Cuda Array failed. Size mismatch.");
  }
  if ((size_ > 0) && (source.getSize() > 0)) {
    hipStream_t s = context_->getStream();
    context_->synchronizeWith(source.getContext());
    CUDA_CALL(hipMemcpy2DAsync(
        values_, pitch_, source.getDataConst(), source.getPitch(), source.getWidthBytes(), 1,
        hipMemcpyDeviceToDevice, s));
  }
}

template <typename T> void CudaArray<T>::assignFromDevice(const T *device_array) {
  DEBUG_OUT(
      "Assign device (S, P,W,H): " << size_ << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                   << height_);
  if ((size_ > 0)) {
    int sz = size_ * sizeof(T);
    hipStream_t s = context_->getStream();
    context_->synchronizeDevice(); // better do device-wide. Not clear where the device array lives
    CUDA_CALL(
        hipMemcpy2DAsync(values_, pitch_, device_array, sz, sz, 1, hipMemcpyDeviceToDevice, s));
  }
}

template <typename T> void CudaArray<T>::setShared(T *device_array) {

  if (device_array == nullptr || size_ <= 0 || values_ == nullptr) {
    RPU_FATAL("Cannot setShared of empty or to nullptr.");
  }

  // destruct
  if (!shared_if_ && values_ != nullptr) {
    context_->synchronize();
    context_->enforceDeviceId();
    CUDA_CALL(hipFree(values_));
    values_ = nullptr;
    shared_if_ = true;
  }
  values_ = device_array; // assign memory shared (memory is governed from outside)

  // Caution: does not CHECK THE SIZE OF THE GIVEN ARRAY!
}

template <typename T> void CudaArray<T>::copyTo(T *host_array) const {

  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Copy to host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                     << height_);

  if (size_ > 0) {
    context_->enforceDeviceId();
    CUDA_CALL(hipMemcpy2DAsync(
        host_array, sz, values_, pitch_, this->getWidthBytes(), height_, hipMemcpyDeviceToHost,
        context_->getStream()));

    context_->synchronizeStream();
  }
}

template <typename T> void CudaArray<T>::copyTo(std::vector<T> &host_vector) const {
  host_vector.resize(size_);
  copyTo(host_vector.data());
}

template <typename T> T *CudaArray<T>::getDataSafe(CudaContextPtr c) {
  context_->synchronizeWith(c);
  return values_;
}

#ifdef RPU_USE_DOUBLE
template class CudaArray<double>;
template class CudaArray<double *>;
#endif

template class CudaArray<float>;
template class CudaArray<float *>;

template class CudaArray<int>;
template class CudaArray<char>;
template class CudaArray<int8_t>;
template class CudaArray<uint32_t>;
template class CudaArray<uint64_t>;
template class CudaArray<hiprandStateXORWOW>;

// reset
void resetCuda(int gpu_id) {
  if (gpu_id >= 0) {
    CUDA_CALL(hipSetDevice(gpu_id));
  }
  CUDA_CALL(hipDeviceReset());
  CUDA_CALL(hipFree(0));
  CUDA_CALL(hipDeviceSynchronize());
}

} // namespace RPU
