#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "cuda_math_util.h"
#include "cuda_util.h"
#include "utility_functions.h"
#include <chrono>
#include <iostream>
#include <memory>

#define IDX2F(i, j, ld) ((((j)-1) * (ld)) + ((i)-1))

// this should be not necesary, because device id is set individually
// per thread. However, if one would want to use 2 GPUs within one
// thread, one needs it.
#define RPU_EXPLICIT_ENFORCE_DEVICE_ID

namespace RPU {

__global__ void kernelCurandSetup(unsigned long long rseed, hiprandState_t *state, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  /* Each thread gets same seed, a different sequence
     number, no offset */
  if (id < n) {
    hiprand_init(rseed, id, 0, &state[id]);
  }
}

__global__ void kernelCurandSetupSameSeed(unsigned long long rseed, hiprandState_t *state, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  if (id < n) {
    hiprand_init(rseed, 0, 0, &state[id]);
  }
}

void curandSetup(CudaArray<hiprandState_t> &dev_states, unsigned long long rseed, bool same_seed) {
  unsigned long long seed = rseed;

  if (rseed == 0) {
    seed = (unsigned long long)std::chrono::high_resolution_clock::now().time_since_epoch().count();
  } else {
    seed = rseed;
  }
  CudaContext *c = dev_states.getContext();
  int m = dev_states.getSize();
  int nthreads = c->getNThreads();
  int nblocks = c->getNBlocks(m, nthreads);
  if (same_seed) {
    kernelCurandSetupSameSeed<<<nblocks, nthreads, 0, c->getStream()>>>(
        seed, dev_states.getData(), m);
  } else {
    kernelCurandSetup<<<nblocks, nthreads, 0, c->getStream()>>>(seed, dev_states.getData(), m);
  }

  c->synchronize();
}

void curandSetup(
    CudaContext *c,
    std::unique_ptr<CudaArray<hiprandState_t>> &dev_states,
    int n,
    unsigned long long rseed,
    bool same_seed) {
  int m = (n + 31) / 32 * 32;
  c->synchronizeDevice();
  dev_states = std::unique_ptr<CudaArray<hiprandState_t>>(new CudaArray<hiprandState_t>(c, m));
  curandSetup(*dev_states, rseed, same_seed);
}

CublasEnvironment::~CublasEnvironment() {

  DEBUG_OUT("Destroy BLAS env.");
  // DEBUG_OUT("handle : " <<this->handle_);

  // destroy device

  // destroy host
  if (handle_ != nullptr) {
    hipblasDestroy(handle_);
    DEBUG_OUT("CUBLAS destroyed");
  }
#ifdef RPU_WITH_CUBLAS_DEVICE
  if (device_handle_created_) {
    DEBUG_OUT("destroy device handle");
    kernelCublasDestroy<<<1, 1>>>(device_handle_);
    hipDeviceSynchronize();
    hipFree(device_handle_);
    DEBUG_OUT("CUBLAS device destroyed");
  }
#endif
  // hipDeviceReset();
}

CublasEnvironment::CublasEnvironment(int gpu_id) {

  DEBUG_OUT("GET BLAS env.");
  if (gpu_id >= 0)
    CUDA_CALL(hipSetDevice(gpu_id));

  // create host
  hipblasStatus_t stat = hipblasCreate(&handle_);
  CUDA_CALL(hipDeviceSynchronize());

  // DEBUG_CALL(this->test(););
  // DEBUG_OUT("handle : " <<handle_);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    RPU_FATAL("CUBLAS initialization failed");
  } else
    DEBUG_OUT("CUBLAS Host initialized.");

#ifdef RPU_WITH_CUBLAS_DEVICE
  device_handle_created_ = false;
#endif
}

void CublasEnvironment::test() {
  this->runTest();
#ifdef RPU_WITH_CUBLAS_DEVICE
  if (device_handle_created_) {
    this->runTestDevice();
  }
#endif
}

static __inline__ void
modifyS(hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta) {
  hipblasSscal(handle, n - p + 1, &alpha, &m[IDX2F(p, q, ldm)], ldm);
  hipblasSscal(handle, ldm - p + 1, &beta, &m[IDX2F(p, q, ldm)], 1);
}

int CublasEnvironment::runTest() {

  // make a test run
  hipblasStatus_t stat;

  int i, j;
  int M = 5;
  int N = 6;
  float *devPtrA;
  float *a = 0;
  a = (float *)malloc(M * N * sizeof(*a));
  if (!a) {
    std::cout << "CUBLAS test run failed (malloc)\n";
    return 1;
  }
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      a[IDX2F(i, j, M)] = (float)((i - 1) * M + j);
    }
  }

  if (hipMalloc((void **)&devPtrA, M * N * sizeof(*a)) != hipSuccess) {
    std::cerr << "CUBLAS test run failed (hipMalloc)\n";
    free(a);
    return 1;
  }

  modifyS(handle_, devPtrA, M, N, 2, 3, 16.0f, 12.0f);
  stat = hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS test run failed (data download)\n";
    hipFree(devPtrA);
    free(a);
    return 1;
  }
  stat = hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "CUBLAS test run failed (data upload)\n";
    hipFree(devPtrA);
    free(a);
    return 1;
  }
  hipFree(devPtrA);
  for (j = 1; j <= N; j++) {
    for (i = 1; i <= M; i++) {
      std::cout << a[IDX2F(i, j, M)] << ",";
    }
    std::cout << std::endl;
  }

  free(a);
  std::cout << "CUBLAS test run successful.\n";

  return 0;
}

#ifdef RPU_WITH_CUBLAS_DEVICE

__global__ void kernelCublasDestroy(hipblasHandle_t *device_handle) {

  hipblasStatus_t status = hipblasDestroy(*device_handle);
  hipDeviceSynchronize();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("ERROR in destroying cublas device!\n");
  }
}

__global__ void kernelCublasCreateDevice(hipblasHandle_t *device_handle) {

  hipblasStatus_t status = hipblasCreate(device_handle);

  hipDeviceSynchronize();

  if (status != HIPBLAS_STATUS_SUCCESS) {
    printf("ERROR in creating cublas device!\n");
    return;
  }
}

void CublasEnvironment::createDeviceHandle() {

  if (device_handle_created_)
    return;

  CUDA_CALL(hipMalloc(&device_handle_, sizeof(hipblasHandle_t)));
  CUDA_CALL(hipDeviceSynchronize());

  kernelCublasCreateDevice<<<1, 1>>>(device_handle_);
  CUDA_CALL(hipDeviceSynchronize());

  DEBUG_OUT("Created device handle");

  device_handle_created_ = true;
}

hipblasHandle_t *CublasEnvironment::getDeviceHandle() {
  if (!device_handle_created_) {
    this->createDeviceHandle();
  }
  return device_handle_;
}

__global__ void kernelCublasTest(hipblasHandle_t *device_handle, float *source, float *dest) {

  hipblasStatus_t status = hipblasScopy(*device_handle, 1, source, 1, dest, 1);
  hipDeviceSynchronize();

  if ((status != HIPBLAS_STATUS_SUCCESS)) {
    printf("Some problems with the CuBLAS device test.\n");
  }
}

int CublasEnvironment::runTestDevice() {

  float one = 1;
  float zero = 0;
  float *a;
  float *b;

  CUDA_CALL(hipMalloc(&a, sizeof(float)));
  CUDA_CALL(hipMalloc(&b, sizeof(float)));
  CUDA_CALL(hipMemcpy(a, &one, sizeof(float), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(b, &zero, sizeof(float), hipMemcpyHostToDevice));

  kernelCublasTest<<<1, 1>>>(device_handle_, a, b);
  CUDA_CALL(hipDeviceSynchronize());

  CUDA_CALL(hipMemcpy(&zero, b, sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CALL(hipFree(a));
  CUDA_CALL(hipFree(b));

  if (zero == 1) {
    std::cout << "CuBLAS device test succeded\n";
    return 0;
  } else {
    std::cerr << "ERROR in CuBLAS device test\n";
    return 1;
  }
}

#endif

//**********************************************************************//
void CudaContext::init() {
  DEBUG_OUT("Init context...");

  if (gpu_id_ >= 0) {
    CUDA_CALL(hipSetDevice(gpu_id_));
  } else {
    CUDA_CALL(hipGetDevice(&gpu_id_));
  }
  DEBUG_OUT("Create context on GPU " << gpu_id_);
  env_ = new CublasEnvironment(gpu_id_);
  stream_id_ = 0;
  rng_created_ = false;
  shared_ = false;
  non_blocking_ = true;

  CUDA_CALL(hipEventCreate(&event_));

  prop_ = new hipDeviceProp_t();
  CUDA_CALL(hipGetDeviceProperties(prop_, gpu_id_));
}

CudaContext::CudaContext(int gpu_id, bool non_blocking)
    : gpu_id_(gpu_id), non_blocking_(non_blocking) {
  this->init();
  this->getStream(0);
}

CudaContext::CudaContext(hipStream_t shared_stream, int gpu_id) : gpu_id_(gpu_id) {
  DEBUG_OUT("Create context on GPU " << gpu_id << " with shared stream (on id 0)\n");
  this->init();

  // ignore the test for shared stream 0. Pytorch seem to like 0
  // if (!shared_stream) {
  //  RPU_FATAL("Shared stream should not be NULL!");
  //} else {
  shared_ = true;
  streams_.push_back(shared_stream);
  // }
}

CudaContext::~CudaContext() {
  DEBUG_OUT("Destroy Cuda Context...");
  if (env_ != nullptr) {
    enforceDeviceId();
    int i_start = shared_ ? 1 : 0;
    for (int i = i_start; i < streams_.size(); i++) {
      hipStreamSynchronize(streams_[i]);
      hipStreamDestroy(streams_[i]);
    }
  }
  if (event_ != nullptr) {
    hipEventDestroy(event_);
    event_ = nullptr;
  }

  if (env_ != nullptr) {
    delete env_;
    env_ = nullptr;
  }
  if (rng_created_) {
    hiprandDestroyGenerator(rng_);
    rng_created_ = false;
  }

  if (prop_ != nullptr) {
    delete prop_;
    prop_ = nullptr;
  }

  DEBUG_OUT("Destroyed.");
}

// copy constructor
CudaContext::CudaContext(const CudaContext &other) {
  // only stream idx 0 is ever shared !
  // copy construction will share the stream.
  // random generator etc are NOT shared !

  gpu_id_ = other.gpu_id_;
  this->init();

  shared_ = true;
  non_blocking_ = other.non_blocking_;

  // only stream 0 is ever shared !!
  if (other.streams_.size() > 0) {
    streams_.push_back(other.streams_[0]);
  }

  for (int i = 1; i < other.streams_.size(); i++) {
    // rest are new streams!!
    this->getStream(i);
  }

  stream_id_ = other.stream_id_;

  if (other.rng_created_) {
    this->createRandomGenerator();
  }

  // random states won't be copied. They will be created a new

  DEBUG_OUT("CudaContext copy constructed [but only first stream shared. New streams and event!].");
}

// copy assignment
CudaContext &CudaContext::operator=(const CudaContext &other) {
  DEBUG_OUT("Copy assignment ");
  CudaContext tmp(other);
  swap(*this, tmp);
  return *this;
}

// move constructor
CudaContext::CudaContext(CudaContext &&other) {
  *this = std::move(other);
  DEBUG_OUT("Move constructor ");
}

// move assignment
CudaContext &CudaContext::operator=(CudaContext &&other) {

  gpu_id_ = other.gpu_id_;
  stream_id_ = other.stream_id_;
  shared_ = other.shared_;
  non_blocking_ = other.non_blocking_;

  prop_ = other.prop_;
  other.prop_ = nullptr;

  streams_ = std::move(other.streams_);

  env_ = other.env_;
  other.env_ = nullptr;

  rng_ = other.rng_;
  other.rng_ = nullptr;

  rng_created_ = other.rng_created_;

  event_ = other.event_;
  other.event_ = nullptr;

  shared_random_states_ = std::move(other.shared_random_states_);

  DEBUG_OUT("Move assignment ");
  return *this;
}

void CudaContext::synchronizeContext() const {
  enforceDeviceId();
  for (int i = 0; i < streams_.size(); i++) {
    CUDA_CALL(hipStreamSynchronize(streams_[i]));
  }
}

void CudaContext::enforceDeviceId() const {
#ifdef RPU_EXPLICIT_ENFORCE_DEVICE_ID
  int gpu_id;
  CUDA_CALL(hipGetDevice(&gpu_id));
  if (gpu_id != gpu_id_) {
    std::cout << "WARNING wrong device detected " << gpu_id << " versus " << gpu_id_ << "!"
              << std::endl;
    CUDA_CALL(hipSetDevice(gpu_id_));
  }
#endif
}

void CudaContext::synchronizeDevice() const {
  enforceDeviceId();
  CUDA_CALL(hipDeviceSynchronize());
}

void CudaContext::synchronizeWith(CudaContext *c) const {

  if (this->getStream() == c->getStream()) {
    // do nothing since work on the same stream
  } else {
    this->synchronize();
    c->synchronize();
  }
}

void CudaContext::synchronizeWith(CudaContext *ca, CudaContext *cb) const {

  if (ca->getStream() != cb->getStream()) {
    ca->synchronizeWith(cb);
  }
  if (ca->getStream() != this->getStream()) {
    this->synchronize();
  }
}

void CudaContext::synchronizeStream(int idx) const {
  DEBUG_OUT("Synchronize stream idx " << idx);
  enforceDeviceId();
  if ((idx >= 0) && (idx < streams_.size())) {
    CUDA_CALL(hipStreamSynchronize(streams_[idx]));
  }
}
void CudaContext::synchronizeStream() const {
  DEBUG_OUT("Synchronize stream id " << stream_id_);
  enforceDeviceId();
  CUDA_CALL(hipStreamSynchronize(streams_[stream_id_]));
}

int CudaContext::getNBlocks(int size, int nthreads) const {
  DEBUG_OUT("get NBlocks for  size " << size);
  return (size + nthreads - 1) / nthreads;
}

int CudaContext::getNStrideBlocks(int size, int nthreads) const {
  DEBUG_OUT("get N Stride Blocks for  size " << size);
  int max_blocks = getSMCount() * maxThreadsPerBlock() / nthreads;
  return MIN(getNBlocks(size, nthreads), max_blocks);
}

hipStream_t CudaContext::getStream(int idx) {

  enforceDeviceId();

  DEBUG_OUT("Try to get streams " << idx);
  if ((idx >= 0) && (idx < streams_.size())) {
    if (stream_id_ != idx) {
      stream_id_ = idx;
      CUBLAS_CALL(hipblasSetStream(this->getBlasHandle(), streams_[idx]));
    }
    return streams_[idx];
  } else if (streams_.size() == idx) {

    hipStream_t s;
    if (non_blocking_) {
      CUDA_CALL(hipStreamCreateWithFlags(&s, hipStreamNonBlocking));
    } else {
      CUDA_CALL(hipStreamCreate(&s));
    }
    streams_.push_back(s);

    stream_id_ = idx;
    CUBLAS_CALL(hipblasSetStream(this->getBlasHandle(), streams_[idx]));

    DEBUG_OUT("Created stream id " << idx << " at : " << streams_[idx] << " ( s: " << s << ")");
    return streams_[idx];
  } else {
    RPU_FATAL("Requested stream size mismatch.");
  }
}

void CudaContext::setStream(hipStream_t s) {
  if (shared_) {
    enforceDeviceId();
    if (s != streams_[stream_id_]) {
      if (stream_id_ != 0) {
        this->synchronizeDevice();
      } else {
        this->synchronizeStream();
      }
    }
    streams_[0] = s;
    stream_id_ = 0;
  } else {
    RPU_FATAL("setStream: must be shared context.");
  }
}

void CudaContext::createRandomGenerator() {
  if (!rng_created_) {
    enforceDeviceId();
    CURAND_CALL(hiprandCreateGenerator(&rng_, HIPRAND_RNG_PSEUDO_DEFAULT));
    CURAND_CALL(hiprandSetStream(rng_, this->getStream()));
    rng_created_ = true;
  }
}

void CudaContext::setRandomSeed(unsigned long long rseed) {
  enforceDeviceId();

  if (!rng_created_) {
    this->createRandomGenerator();
  }

  unsigned long long seed = rseed;
  if (rseed == 0) {
    seed = (unsigned long long)std::chrono::high_resolution_clock::now().time_since_epoch().count();
  } else {
    seed = rseed;
  }
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(rng_, seed));
  this->synchronizeStream();
}

void CudaContext::randNormal(float *dev_array, int size, float mean, float stddev) {
  if (!rng_created_) {
    setRandomSeed(0); // will create random generator on the fly
  }

  if (stddev > 0) {
    CURAND_CALL(hiprandGenerateNormal(rng_, dev_array, size, mean, stddev));
  } else {
    RPU::math::elemconst(this, dev_array, size, mean);
  }
}

void CudaContext::randUniform(float *dev_array, int size) {

  if (!rng_created_) {
    setRandomSeed(0);
  }

  CURAND_CALL(hiprandGenerateUniform(rng_, dev_array, size));
}

hiprandState_t *CudaContext::getRandomStates(int size) {

  int n = size;
  if (n <= 0) {
    n = getSMCount() * maxThreadsPerBlock();
  }

  if (shared_random_states_.size() <= stream_id_) {
    shared_random_states_.resize(stream_id_ + 1);
  }
  if (!shared_random_states_[stream_id_] || (n > shared_random_states_[stream_id_]->getSize())) {
    curandSetup(this, shared_random_states_[stream_id_], n, 0, false);
  }
  return shared_random_states_[stream_id_]->getData();
}

void CudaContext::recordWaitEvent(CudaContext *wait_on_context) {
  this->recordWaitEvent(wait_on_context->getStream(), wait_on_context->getEvent());
}
void CudaContext::recordEvent() { CUDA_CALL(hipEventRecord(event_, streams_[stream_id_])); }
void CudaContext::waitEvent(hipEvent_t wait_on_event) {
  CUDA_CALL(hipStreamWaitEvent(streams_[stream_id_], wait_on_event, 0));
}

void CudaContext::waitEvent(CudaContext *wait_on_context) {
  waitEvent(wait_on_context->getEvent());
}

void CudaContext::recordWaitEvent(hipStream_t s) { this->recordWaitEvent(s, event_); }

void CudaContext::recordWaitEvent(hipStream_t s, hipEvent_t e) {
  if (streams_[stream_id_] != s) {
    CUDA_CALL(hipEventRecord(e, s));
    CUDA_CALL(hipStreamWaitEvent(streams_[stream_id_], e, 0));
  }
}

//**********************************************************************//

template <typename T>
CudaArray<T>::CudaArray(CudaContext *c) : size_(0), width_(0), height_(1), pitch_(0), context_(c) {}

template <typename T> CudaArray<T>::CudaArray(CudaContext *c, int n) : CudaArray(c) {
  size_ = n;
  width_ = n;
  height_ = 1; // this needs to be one! No height>1 supported yet
  if (n > 0) {
    context_->enforceDeviceId();
    CUDA_CALL(hipMallocPitch(&values_, &pitch_, n * sizeof(T), height_));
  }
}

template <typename T>
CudaArray<T>::CudaArray(CudaContext *c, int n, const T *host_array) : CudaArray(c, n) {
  if (n > 0) {
    this->assign(host_array);
    context_->synchronize(); // better syncrhonize. Constructing is considered slow anyway
  }
}

template <typename T> CudaArray<T>::~CudaArray() {

  // no sync because no ownership of context !! (might be already destructed)
  if ((size_ > 0) && (values_ != nullptr) && (!shared_if_)) {
    hipFree(values_);
    values_ = nullptr;
  }
}

// copy constructor
template <typename T> CudaArray<T>::CudaArray(const CudaArray<T> &other) {
  size_ = other.size_;
  width_ = other.width_;
  height_ = other.height_;
  pitch_ = other.pitch_;
  context_ = other.context_;
  values_ = nullptr;

  if (size_ > 0) {
    context_->enforceDeviceId();
    CUDA_CALL(hipMallocPitch(&values_, &pitch_, size_ * sizeof(T), height_));

    if (other.shared_if_) {
      this->setShared(other.values_);
    } else {
      this->assign(other);
    }
    context_->synchronize(); // better synchronize. Constructing is slow anyway
  }

  DEBUG_OUT("CudaArray copy constructed.");
}

// copy assignment
template <typename T> CudaArray<T> &CudaArray<T>::operator=(const CudaArray<T> &other) {
  context_->enforceDeviceId();
  CudaArray<T> tmp(other); // seems a bit inefficient...
  swap(*this, tmp);
  context_->synchronize(); // need sync because of tmp
  return *this;
}

// move constructor
template <typename T> CudaArray<T>::CudaArray(CudaArray<T> &&other) {
  context_->enforceDeviceId();
  *this = std::move(other);
}

// move assignment
template <typename T> CudaArray<T> &CudaArray<T>::operator=(CudaArray<T> &&other) {

  size_ = other.size_;
  other.size_ = 0;

  width_ = other.width_;
  other.width_ = 0;

  height_ = other.height_;
  other.height_ = 0;

  pitch_ = other.pitch_;
  other.pitch_ = 0;

  context_ = other.context_;
  other.context_ = nullptr;

  values_ = other.values_;
  other.values_ = nullptr;

  shared_if_ = other.shared_if_;

  return *this;
}

template <typename T> void CudaArray<T>::setConst(T set_value) {

  DEBUG_OUT(
      "Set (hsize,P,W,H): " << size_ << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                            << height_);
  if (size_ > 0) {
    context_->enforceDeviceId();
    if (set_value != 0) {
      RPU::math::elemconst(context_, values_, size_, set_value);
    } else {
      CUDA_CALL(hipMemset2DAsync(
          values_, pitch_, 0, this->getWidthBytes(), height_, context_->getStream()));
    }
  }
}

template <> void CudaArray<hiprandStateXORWOW>::setConst(hiprandStateXORWOW set_value) {
  RPU_FATAL("Cannot set curandstates to some values.");
}

template <> void CudaArray<double *>::setConst(double *set_value) {
  RPU_FATAL("Cannot set pointer types to some values.");
}

template <> void CudaArray<float *>::setConst(float *set_value) {
  RPU_FATAL("Cannot set pointer types to some values.");
}

template <typename T> void CudaArray<T>::printValues(int nmax) const {
  T *values = new T[size_];
  this->copyTo(values); // will synchronize
  int n = nmax > 0 ? MIN(nmax, size_) : size_;
  for (int i = 0; i < n; ++i) {
    std::cout << "[" << i << "]:" << values[i] << ", ";
  }
  if (n < size_) {
    std::cout << "...";
  }
  std::cout << std::endl;
  delete[] values;
}

template <> void CudaArray<hiprandStateXORWOW>::printValues(int nmax) const {
  RPU_FATAL("Cannot print curandstates.");
}

template <typename T> void CudaArray<T>::assign(const T *host_array) {
  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Assign host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                    << height_);
  context_->enforceDeviceId();
  context_->synchronize();
  CUDA_CALL(hipMemcpy2DAsync(
      values_, pitch_, host_array, sz, sz, 1, hipMemcpyHostToDevice, context_->getStream()));
}

template <typename T>
void CudaArray<T>::assignTranspose(const T *host_array, const int m, const int n) {
  // col major to row major
  if (m * n != size_) {
    RPU_FATAL("Size mismatch");
  }
  T *transposed_array = new T[size_];
  for (int i = 0; i < size_; i++) {
    int i_col = (i % n);
    int i_row = (i / n);
    transposed_array[i_col * m + i_row] = host_array[i];
  }
  context_->enforceDeviceId();
  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Assign host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                    << height_);
  context_->synchronize();
  CUDA_CALL(hipMemcpy2D(
      values_, pitch_, transposed_array, sz, sz, 1, hipMemcpyHostToDevice)); // no async
  delete[] transposed_array;
}

template <typename T> void CudaArray<T>::assign(const CudaArray<T> &source) {
  DEBUG_OUT(
      "Assign device (P,W,H): "
      << ", " << pitch_ << ", " << width_ * sizeof(T) << ", " << height_);
  if (source.getSize() != size_) {
    RPU_FATAL("Assignment of Cuda Array failed. Size mismatch.");
  }
  if ((size_ > 0) && (source.getSize() > 0)) {
    hipStream_t s = context_->getStream();
    context_->synchronizeWith(source.getContext());
    CUDA_CALL(hipMemcpy2DAsync(
        values_, pitch_, source.getDataConst(), source.getPitch(), source.getWidthBytes(), 1,
        hipMemcpyDeviceToDevice, s));
  }
}

template <typename T> void CudaArray<T>::assignFromDevice(const T *device_array) {
  DEBUG_OUT(
      "Assign device (P,W,H): "
      << ", " << pitch_ << ", " << width_ * sizeof(T) << ", " << height_);
  if ((size_ > 0)) {
    int sz = size_ * sizeof(T);
    hipStream_t s = context_->getStream();
    context_->synchronizeDevice(); // better do device-wide. Not clear where the device array lives
    CUDA_CALL(
        hipMemcpy2DAsync(values_, pitch_, device_array, sz, sz, 1, hipMemcpyDeviceToDevice, s));
  }
}

template <typename T> void CudaArray<T>::setShared(T *device_array) {

  if (device_array == nullptr || size_ <= 0 || values_ == nullptr) {
    RPU_FATAL("Cannot setShared of empty or to nullptr.");
  }

  // destruct
  if (!shared_if_) {
    context_->synchronize();
    context_->enforceDeviceId();
    CUDA_CALL(hipFree(values_));
    values_ = nullptr;
    shared_if_ = true;
  }
  values_ = device_array; // assign memory shared (memory is governed from outside)

  // Caution: does not CHECK THE SIZE OF THE GIVEN ARRAY!
}

template <typename T> void CudaArray<T>::copyTo(T *host_array) const {

  int sz = size_ * sizeof(T);
  DEBUG_OUT(
      "Copy to host (hsize,P,W,H): " << sz << ", " << pitch_ << ", " << width_ * sizeof(T) << ", "
                                     << height_);

  if (size_ > 0) {
    context_->enforceDeviceId();
    CUDA_CALL(hipMemcpy2DAsync(
        host_array, sz, values_, pitch_, this->getWidthBytes(), height_, hipMemcpyDeviceToHost,
        context_->getStream()));

    context_->synchronizeStream();
  }
}

template <typename T> T *CudaArray<T>::getDataSafe(CudaContext *c) {
  context_->synchronizeWith(c);
  return values_;
}

#ifdef RPU_USE_DOUBLE
template class CudaArray<double>;
template class CudaArray<double *>;
#endif

template class CudaArray<float>;
template class CudaArray<float *>;

template class CudaArray<int>;
template class CudaArray<char>;
template class CudaArray<uint32_t>;
template class CudaArray<uint64_t>;
template class CudaArray<hiprandStateXORWOW>;

// reset
void resetCuda(int gpu_id) {
  if (gpu_id >= 0) {
    CUDA_CALL(hipSetDevice(gpu_id));
  }
  CUDA_CALL(hipDeviceReset());
  CUDA_CALL(hipFree(0));
  CUDA_CALL(hipDeviceSynchronize());
}

} // namespace RPU
