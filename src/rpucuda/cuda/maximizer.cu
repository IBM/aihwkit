#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "maximizer.h"

#include <chrono>
#include <cmath>
#include <iostream>
#include <memory>
#include <random>

#include "cuda_math_util.h"
#include "cuda_util.h"
#include <hipcub/hipcub.hpp>

#include "io_iterator.h"

namespace RPU {

template <typename InputIteratorT, bool abs_if = true>
__global__ void kernelMaximizeBatchTrans(
    InputIteratorT input,
    const int total_size_in,
    const int m_batch_in,
    float *max_values,
    float *max_values0) {

  // -- only use this version if m_batch < blockDim.x !!!
  // -- probably: strided version would be faster...

  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  extern __shared__ int block_max_values[]; // assumes that shared is of size nthreads*sizeof(int)

  const int size = total_size_in;
  const int m_batch = m_batch_in;

  if (abs_if) {
    block_max_values[threadIdx.x] = 0;
  } else {
    block_max_values[threadIdx.x] = INT_MIN;
  }
  __syncthreads();

  if (tid < m_batch) {
    if (abs_if) {
      max_values0[tid] = 0; // for next round
    } else {
      max_values0[tid] = -FLT_MAX; // for next round
    }
  }

  if (tid < size) {

    float value = input[tid]; // typecast to float. (because float to int)

    int midx = tid % m_batch;

    if (abs_if) {
      value = (value >= 0) ? value : -value;
    }

    atomicMax(&(block_max_values[midx]), __float_as_int(value));
  }
  __syncthreads();

  int bidx = threadIdx.x;
  if (bidx < m_batch) {
    atomicMax((int *)&(max_values[bidx]), block_max_values[bidx]);
  }
}

template <typename InputIteratorT, bool abs_if = true>
__global__ void kernelMaximizeBatchTrans_LargeBatch(
    InputIteratorT input,
    const int total_size_in,
    const int m_batch_in,
    float *max_values,
    float *max_values0) {

  // -- use this version if m_batch >= blockDim.x
  // -- just uses atomic on global memory

  int tid = blockDim.x * blockIdx.x + threadIdx.x;

  const int size = total_size_in;
  const int m_batch = m_batch_in;

  if (tid < m_batch) {
    if (abs_if) {
      max_values0[tid] = 0; // for next round
    } else {
      max_values0[tid] = -FLT_MAX; // for next round
    }
  }

  if (tid < size) {

    float value = input[tid];

    int midx = tid % m_batch;

    if (abs_if) {
      value = (value >= 0) ? value : -value;
    }

    atomicMax((int *)&max_values[midx], __float_as_int(value));
  }
}

template <typename T> struct IndexReader {
  __host__ __device__ IndexReader(T *data_in) { data = data_in; }
  __host__ __device__ __forceinline__ T operator()(const int &idx) const {
    return (idx > 0) ? data[idx - 1] : 0;
  }

  __host__ __device__ __forceinline__ void setData(T *data_in) { data = data_in; }

  T *data;
};

template <typename T> struct BatchTransposer {
  __host__ __device__ BatchTransposer(T *data_in, int size_in, int m_batch_in) {
    m_batch = m_batch_in;
    size = size_in;
    data = data_in;
  }
  __host__ __device__ __forceinline__ T operator()(const int &idx) const {
    return data[(idx / size) + (idx % size) * m_batch];
  }

  __host__ __device__ __forceinline__ void setSizeAndBatch(int size_in, int m_batch_in) {
    m_batch = m_batch_in;
    size = size_in;
  }

  __host__ __device__ __forceinline__ void setData(T *data_in) { data = data_in; }

  T *data;
  int size;
  int m_batch;
};

namespace test_helper {

template <typename T>
void debugMaxBatched(const T *indata, int size, int m_batch, bool trans, float *max_values) {

  int *offsets = new int[m_batch + 1];

  for (int i = 0; i <= m_batch; i++) {
    offsets[i] = i * size;
  }

  CudaContext c(-1, false);
  CudaArray<T> dev_in(&c, size * m_batch, indata);
  CudaArray<float> dev_max_values(&c, m_batch);
  dev_max_values.setConst(0);
  CudaArray<float> dev_max_values0(&c, m_batch);

  CudaArray<int> dev_offsets(&c, m_batch + 1, offsets);

  CUDA_CALL(hipPeekAtLastError());
  CUDA_CALL(hipDeviceSynchronize());

  // test transform input iterator
  int *tmp = new int[size * m_batch];
  for (int i = 0; i < size * m_batch; i++) {
    tmp[i] = i + 1;
  }
  CudaArray<int> dev_in_index(&c, size * m_batch, tmp);
  CUDA_CALL(hipDeviceSynchronize());

  IndexReader<T> idx_reader(dev_in.getData());
  hipcub::TransformInputIterator<T, IndexReader<T>, int *> in_itr(dev_in_index.getData(), idx_reader);

  hipcub::CountingInputIterator<int> index(0);
  BatchTransposer<T> batch_transposer(dev_in.getData(), size, m_batch);
  hipcub::TransformInputIterator<T, BatchTransposer<T>, hipcub::CountingInputIterator<int>> in_trans_itr(
      index, batch_transposer);

  IndexReader<int> idx_reader_host(tmp);
  hipcub::TransformInputIterator<int, IndexReader<int>, int *> test_host(tmp, idx_reader_host);
  std::cout << test_host[0] << std::endl;

  CustomMaxAbs max_abs;
  // Determine temporary device storage requirements
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedReduce::Reduce(
      d_temp_storage, temp_storage_bytes, in_itr, dev_max_values.getData(), m_batch,
      dev_offsets.getData(), dev_offsets.getData() + 1, max_abs, 0, c.getStream());
  // Allocate temporary storage
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  CUDA_CALL(hipDeviceSynchronize());

  int nthreads = c.getNThreads();
  int nblocks = c.getNBlocks(size * m_batch, nthreads);
  hipStream_t s = c.getStream();

  CUDA_TIMING_INIT;
  CUDA_TIMING_START(c);

  if (trans) {

    // this works, too, but has some performance hit, because of non-aligned memory reads
    // hipcub::DeviceSegmentedReduce::Reduce(d_temp_storage, temp_storage_bytes,
    // 				   in_trans_itr, dev_max_values.getData(),
    // 				   m_batch, dev_offsets.getData(),
    // 				   dev_offsets.getData()+1, max_abs,0,c.getStream());

    if (m_batch > nthreads) {
      kernelMaximizeBatchTrans_LargeBatch<<<nblocks, nthreads, 0, s>>>(
          in_itr, size * m_batch, m_batch, dev_max_values.getData(), dev_max_values0.getData());

    } else {
      kernelMaximizeBatchTrans<<<nblocks, nthreads, nthreads * sizeof(int), s>>>(
          in_itr, size * m_batch, m_batch, dev_max_values.getData(), dev_max_values0.getData());
    }

  } else {
    // only trans==false
    // Fast Segmented reduction (much faster than loop from outside)
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, temp_storage_bytes, in_itr, dev_max_values.getData(), m_batch,
        dev_offsets.getData(), dev_offsets.getData() + 1, max_abs, 0, c.getStream());
  }

  CUDA_TIMING_STOP(c, "Max Batch");

  CUDA_CALL(hipPeekAtLastError());
  CUDA_CALL(hipDeviceSynchronize());
  dev_max_values.copyTo(max_values);
  CUDA_CALL(hipDeviceSynchronize());
  hipFree(d_temp_storage);
  delete[] offsets;
  delete[] tmp;
}
#ifdef RPU_USE_DOUBLE
template void debugMaxBatched<double>(double const *, int, int, bool, float *);
#endif
template void debugMaxBatched<float>(float const *, int, int, bool, float *);
} // namespace test_helper

/****************************************************************************************************************/
/* MAXIMIZER */
/******************************************************************************************************************/
#define LAUNCH_MAX_KERNEL(KNAME, SHARED_MEM, ARGS)                                                 \
  if (abs_if_) {                                                                                   \
    KNAME<InputIteratorT, true><<<nblocks, nthreads, SHARED_MEM, s>>> ARGS;                        \
  } else {                                                                                         \
    KNAME<InputIteratorT, false><<<nblocks, nthreads, SHARED_MEM, s>>> ARGS;                       \
  }

template <typename T>
Maximizer<T>::Maximizer(CudaContext *c, int size, bool abs_if)
    : size_{size}, context_{c}, buffer_m_batch_{0}, abs_if_{abs_if} {
  // initialize for m_batch=1
  dev_max_values_ = RPU::make_unique<CudaArray<float>>(context_, 1);
  size_t temp_storage_bytes = 0;
  if (abs_if_) {
    hipcub::DeviceReduce::Reduce(
        nullptr, temp_storage_bytes, dev_max_values_->getData(), dev_max_values_->getData(), size_,
        max_abs_op_, 0, context_->getStream());
  } else {
    hipcub::DeviceReduce::Max(
        nullptr, temp_storage_bytes, dev_max_values_->getData(), dev_max_values_->getData(), size_,
        context_->getStream());
  }

  dev_v_temp_storage_ = RPU::make_unique<CudaArray<char>>(context_, temp_storage_bytes);
}

template <typename T> void Maximizer<T>::initializeBatchBuffer(int m_batch) {

  if ((m_batch > 1) && (buffer_m_batch_ != m_batch)) {
    buffer_m_batch_ = m_batch;

    dev_max_values_ = RPU::make_unique<CudaArray<float>>(context_, m_batch);
    dev_max_values0_ = RPU::make_unique<CudaArray<float>>(context_, m_batch);
    dev_max_values0_->setConst(abs_if_ ? 0 : std::numeric_limits<T>::min());

    int *offsets = new int[m_batch + 1];

    // not trans
    for (int i = 0; i <= m_batch; i++) {
      offsets[i] = i * size_;
    }

    dev_offsets_ = RPU::make_unique<CudaArray<int>>(context_, m_batch + 1, offsets);

    size_t temp_storage_bytes = 0;
    if (abs_if_) {
      hipcub::DeviceSegmentedReduce::Reduce(
          nullptr, temp_storage_bytes, dev_max_values_->getData(), dev_max_values_->getData(),
          m_batch, dev_offsets_->getData(), dev_offsets_->getData() + 1, max_abs_op_, 0,
          context_->getStream());
    } else {
      hipcub::DeviceSegmentedReduce::Max(
          nullptr, temp_storage_bytes, dev_max_values_->getData(), dev_max_values_->getData(),
          m_batch, dev_offsets_->getData(), dev_offsets_->getData() + 1, context_->getStream());
    }
    dev_m_temp_storage_ = RPU::make_unique<CudaArray<char>>(context_, temp_storage_bytes);

    context_->synchronize();
    delete[] offsets;
    // dev_offsets_->printValues();
  }
}

template <typename T> void Maximizer<T>::setZeroBelow(T thres) {
  RPU::math::elemsetbelowzero(
      context_, dev_max_values_->getData(), dev_max_values_->getSize(), (float)thres);
}

template <typename T> void Maximizer<T>::saturateAbove(T thres) {
  RPU::math::elemmin<float>(
      context_, dev_max_values_->getData(), dev_max_values_->getSize(), (float)thres);
}

template <typename T>
template <typename InputIteratorT>
void Maximizer<T>::compute(InputIteratorT dev_input, int m_batch, bool trans) {

  // does not check for positive m_batch!
  hipStream_t s = context_->getStream();

  if (m_batch == 1) {
    size_t ssz = dev_v_temp_storage_->getSize();
    if (abs_if_) {
      hipcub::DeviceReduce::Reduce(
          (void *)dev_v_temp_storage_->getData(), ssz, dev_input, dev_max_values_->getData(), size_,
          max_abs_op_, (T)0, s);
    } else {
      hipcub::DeviceReduce::Max(
          (void *)dev_v_temp_storage_->getData(), ssz, dev_input, dev_max_values_->getData(), size_,
          s);
    }

  } else {

    if (trans) {

      if (buffer_m_batch_ < m_batch) {
        this->initializeBatchBuffer(m_batch);
      }

      std::swap(dev_max_values_, dev_max_values0_);
      int nthreads = context_->getNThreads();
      int n = size_ * m_batch;
      int nblocks = context_->getNBlocks(n, nthreads);
      if (m_batch <= nthreads) {
        int shared_mem = nthreads * sizeof(int);

        LAUNCH_MAX_KERNEL(
            kernelMaximizeBatchTrans, shared_mem,
            (dev_input, n, m_batch, dev_max_values_->getData(), dev_max_values0_->getData()));

      } else {
        // simple atomic global memory version
        LAUNCH_MAX_KERNEL(
            kernelMaximizeBatchTrans_LargeBatch, 0,
            (dev_input, n, m_batch, dev_max_values_->getData(), dev_max_values0_->getData()));
      }

    } else {
      if (buffer_m_batch_ != m_batch) { // !! need to reinitilize offsets when batch changes !
        this->initializeBatchBuffer(m_batch);
      }

      // Fast Segmented reduction (much faster than loop from outside)
      size_t ssz = dev_m_temp_storage_->getSize();
      if (abs_if_) {
        hipcub::DeviceSegmentedReduce::Reduce(
            (void *)dev_m_temp_storage_->getData(), ssz, dev_input, dev_max_values_->getData(),
            m_batch, dev_offsets_->getData(), dev_offsets_->getData() + 1, max_abs_op_, (T)0.0, s);
      } else {
        hipcub::DeviceSegmentedReduce::Max(
            (void *)dev_m_temp_storage_->getData(), ssz, dev_input, dev_max_values_->getData(),
            m_batch, dev_offsets_->getData(), dev_offsets_->getData() + 1, s);
      }
    }
  }
}

#define ARGS1 , int, bool

template class Maximizer<float>;
RPU_GEN_IITER_TEMPLATES(float, void, Maximizer<float>::compute, ARGS1);
template void Maximizer<float>::compute(NegateInputIterator<float> ARGS1);

#ifdef RPU_USE_DOUBLE
template class Maximizer<double>;
RPU_GEN_IITER_TEMPLATES(double, void, Maximizer<double>::compute, ARGS1);
template void Maximizer<double>::compute(NegateInputIterator<double> ARGS1);
#endif

#undef RPU_MX_TEMPLATE
#undef LAUNCH_MAX_KERNEL
#undef ARGS1

} // namespace RPU
