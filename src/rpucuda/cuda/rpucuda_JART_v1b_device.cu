#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021, 2022 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "pwu_kernel_parameter.h"
#include "rpu_pulsed_meta_parameter.h"
#include "rpucuda_JART_v1b_device.h"
#include <memory>
#include <stdio.h>

namespace RPU {

template <typename T>
__device__ __forceinline__ T map_Ndisc_to_weight(
    const double &Ndisc,
    const T &current_min,
    const T &weight_min_bound,
    const T &current_to_weight_ratio,
    const T &g_read,
    const T &h_read,
    const T &j_0,
    const T &k0,
    const T &Original_Ndiscmin) {
  T read_current = g_read/(pow((1+h_read*pow((Ndisc/Original_Ndiscmin),-j_0)),1/k0));
  T weight = (read_current-current_min)*current_to_weight_ratio+weight_min_bound;
  return weight;
}

template <typename T>
__device__ __forceinline__ void apply_cycle_to_cycle_noise(
    const T &ratio,
    T &Ndiscmax,
    T &Ndiscmin,
    T &ldet,
    T &A,
    const T &Ndiscmax_std,
    const T &Ndiscmin_std,
    const T &ldet_std,
    const T &rdet_std,
    const T &ldet_std_slope,
    const T &rdet_std_slope,
    hiprandState &local_state,
    const T &Ndiscmax_upper_bound,
    const T &Ndiscmax_lower_bound,
    const T &Ndiscmin_upper_bound,
    const T &Ndiscmin_lower_bound,
    const T &ldet_upper_bound,
    const T &ldet_lower_bound,
    const T &rdet_upper_bound,
    const T &rdet_lower_bound) {
  if (Ndiscmax_std > (T)0.0) {
    T stoch_value = hiprand_normal(&local_state);
    Ndiscmax = Ndiscmax * (1 + Ndiscmax_std * stoch_value);
    if ((Ndiscmax_upper_bound > (T)0.0) && (Ndiscmax_lower_bound >= (T)0.0)) {
      Ndiscmax = MIN(Ndiscmax, Ndiscmax_upper_bound);
      Ndiscmax = MAX(Ndiscmax, Ndiscmax_lower_bound);
    }
  }
  if (Ndiscmin_std > (T)0.0) {
    T stoch_value = hiprand_normal(&local_state);
    Ndiscmin = Ndiscmin * (1 + Ndiscmin_std * stoch_value);
    if ((Ndiscmin_upper_bound > (T)0.0) && (Ndiscmin_lower_bound >= (T)0.0)) {
      Ndiscmin = MIN(Ndiscmin, Ndiscmin_upper_bound);
      Ndiscmin = MAX(Ndiscmin, Ndiscmin_lower_bound);
    }
  }
  if ((ldet_std > (T)0.0)||(ldet_std_slope > (T)0.0)) {
    T stoch_value_1 = hiprand_normal(&local_state);
    T stoch_value_2 = hiprand_normal(&local_state);
    ldet = ldet * (1 + ldet_std * stoch_value_1 + ratio * ldet_std_slope * stoch_value_2);
    if ((ldet_upper_bound > (T)0.0) && (ldet_lower_bound >= (T)0.0)) {
      ldet = MIN(ldet, ldet_upper_bound);
      ldet = MAX(ldet, ldet_lower_bound);
    }
  }
  if ((rdet_std > (T)0.0)||(rdet_std_slope > (T)0.0)) {
    T stoch_value_1 = hiprand_normal(&local_state);
    T stoch_value_2 = hiprand_normal(&local_state);
    T rdet = pow(A/M_PI, 0.5) * (1 + rdet_std * stoch_value_1 + ratio * rdet_std_slope * stoch_value_2);
    if ((rdet_upper_bound > (T)0.0) && (rdet_lower_bound >= (T)0.0)) {
      rdet = MIN(rdet, rdet_upper_bound);
      rdet = MAX(rdet, rdet_lower_bound);
    }
    A = M_PI*pow(rdet,2.0);
  }
}

template <typename T> struct UpdateFunctorJARTv1b {

  __device__ __forceinline__ void operator()(
      T &apparent_weight,
      uint32_t n,
      uint32_t negative,
      float4 &par_4,
      float2 &par_2,
      T &persistent_weight,
      const T *global_pars,
      const int global_params_count,
      T noise_std_dw,
      hiprandState &local_state) {

    UNUSED(global_params_count); // fixed

    const T pulse_voltage_SET       = global_pars[0];
    const T pulse_voltage_RESET     = global_pars[1];
    const T pulse_length            = global_pars[2];
    const T base_time_step          = global_pars[3];
    const T alpha_SET               = global_pars[4];
    const T beta_SET                = global_pars[5];
    const T c_SET                   = global_pars[6];
    const T d_SET                   = global_pars[7];
    const T f_SET                   = global_pars[8];
    const T g_RESET                 = global_pars[9];
    const T h_RESET                 = global_pars[10];
    const T g_read                  = global_pars[11];
    const T h_read                  = global_pars[12];
    const T j_0                     = global_pars[13];
    const T k0                      = global_pars[14];
    const T T0                      = global_pars[15];
    const T Ndiscmin                = global_pars[16];
    const T Nplug                   = global_pars[17];
    const T a_ny0                   = global_pars[18];
    const T dWa                     = global_pars[19];
    const T Rth_negative            = global_pars[20];
    const T Rth_positive            = global_pars[21];
    const T RseriesTiOx             = global_pars[22];
    const T R0                      = global_pars[23];
    const T V_series_coefficient    = global_pars[24];
    const T V_disk_coefficient      = global_pars[25];
    const T gamma_coefficient       = global_pars[26];
    const T lcell                   = global_pars[27];
    const T current_min             = global_pars[28];
    const T current_to_weight_ratio = global_pars[29];
    const T weight_to_current_ratio = global_pars[30];
    const T w_min                   = global_pars[31];
    // TODO: BUG: Use device variable bounds will result in PyTorch not receving the updated weights.
    const T Ndisc_max_bound         = global_pars[32];
    const T Ndisc_min_bound         = global_pars[33];
    const T Ndiscmax_std            = global_pars[34];
    const T Ndiscmax_upper_bound    = global_pars[35];
    const T Ndiscmax_lower_bound    = global_pars[36];
    const T Ndiscmin_std            = global_pars[37];
    const T Ndiscmin_upper_bound    = global_pars[38];
    const T Ndiscmin_lower_bound    = global_pars[39];
    const T ldet_std                = global_pars[40];
    const T ldet_std_slope          = global_pars[41];
    const T ldet_upper_bound        = global_pars[42];
    const T ldet_lower_bound        = global_pars[43];
    const T rdet_std                = global_pars[44];
    const T rdet_std_slope          = global_pars[45];
    const T rdet_upper_bound        = global_pars[46];
    const T rdet_lower_bound        = global_pars[47];
    
    /* NOTE: These values does not do random walk,
             so the original values are not supposed to change.
             Do not use refference pointer on these values.
    */ 
    T device_specific_Ndisc_min_bound_cuda = par_4.x;                          // [0]
    T device_specific_Ndisc_max_bound_cuda = par_4.z;                          // [2]

    /* NOTE: These values do random walks,
             use refference to change the recorded values.
    */ 
    T &device_specific_Ndiscmin_cuda = par_4.y; // [1]
    T &device_specific_Ndiscmax_cuda = par_4.w; // [3]
    T &device_specific_ldet_cuda = par_2.x; // [0]
    T &device_specific_A_cuda = par_2.y; // [1]

    T &w = apparent_weight;
    T &Ndisc = persistent_weight;

    uint32_t pulse_counter = uint32_t (pulse_length/base_time_step);
    // n is larger 0 in any case
    pulse_counter = pulse_counter *n;
    double Ndisc_double = Ndisc;
    
    // TODO: BUG: Use device variable bounds will result in PyTorch not receving the updated weights.
    // T max_bound = MIN(device_specific_Ndisc_max_bound_cuda, device_specific_Ndiscmax_cuda);
    // T min_bound = MAX(device_specific_Ndisc_min_bound_cuda, device_specific_Ndiscmin_cuda);
    T max_bound = MIN(Ndisc_max_bound, device_specific_Ndiscmax_cuda);
    T min_bound = MAX(Ndisc_min_bound, device_specific_Ndiscmin_cuda);

    if (negative > 0) {
      if (Ndisc_double < max_bound){
        for (int i_updates = 0; i_updates < pulse_counter; i_updates++) {
          T I_mem = -alpha_SET-beta_SET/(pow((1.0+pow((c_SET/Ndisc),d_SET)),f_SET));

          T V_disk = I_mem*(device_specific_ldet_cuda/(V_disk_coefficient*device_specific_A_cuda*Ndisc_double));

          // NOTE: T gamma = gamma_coefficient*Eion
          T gamma = gamma_coefficient*V_disk/device_specific_ldet_cuda;
          
          // NOTE: V - V_series = V_disk+V_plug+V_Schottky
          T V_other_than_series = pulse_voltage_SET - (I_mem*(RseriesTiOx + R0 + V_series_coefficient*I_mem*I_mem));

          T Treal = T0 + I_mem*V_other_than_series*Rth_negative;

          // NOTE: dWamin = dWa_f = dWa*(sqrt(1.0-pow(gamma,2.0))-(gamma*M_PI)/2+gamma*asin(gamma)) = dWa_mean - dWa_difference
          // NOTE: dWamax = dWa_r = dWa*(sqrt(1.0-pow(gamma,2.0))+(gamma*M_PI)/2+gamma*asin(gamma)) = dWa_mean + dWa_difference
          T dWa_mean = dWa*(sqrt(1.0-pow(gamma,2.0))+gamma*asin(gamma));
          T dWa_difference = dWa*((gamma*M_PI)/2.0);

          T denominator = PHYSICAL_PARAMETER_kb_over_e*Treal;

          T c_v0 = (Nplug+Ndisc_double)/2.0;
          T F1 = 1.0-pow((Ndisc_double/device_specific_Ndiscmax_cuda),10.0);
          T dNdt = -(c_v0*a_ny0*F1*(exp(-(dWa_mean - dWa_difference)/denominator)-exp(-(dWa_mean + dWa_difference)/denominator)))/device_specific_ldet_cuda;

          Ndisc_double = Ndisc_double + dNdt*base_time_step;
        }
        // TODO: BUG: applying these noise will result in PyTorch not receving the updated weights.
        // T ratio = Ndisc_double;
        // ratio = (ratio-Ndisc)/(Ndiscmax-Ndisc);
        // apply_cycle_to_cycle_noise(ratio, Ndiscmax, Ndiscmin, ldet, A, Ndiscmax_std, Ndiscmin_std, ldet_std, rdet_std, ldet_std_slope, rdet_std_slope, local_state,
        //                            Ndiscmax_upper_bound, Ndiscmax_lower_bound, Ndiscmin_upper_bound, Ndiscmin_lower_bound,
        //                            ldet_upper_bound, ldet_lower_bound, rdet_upper_bound, rdet_lower_bound);
        Ndisc_double = MIN(Ndisc_double, max_bound);
        w = map_Ndisc_to_weight(Ndisc_double, current_min, w_min, current_to_weight_ratio, g_read, h_read, j_0, k0, Ndiscmin);
        Ndisc = Ndisc_double;
      }
    
    }else{
      if (Ndisc_double > min_bound){
        for (int i_updates = 0; i_updates < pulse_counter; i_updates++) {
          T I_mem = g_RESET/(pow((1+h_RESET*pow((Ndisc/Ndiscmin),-j_0)),1.0/k0));
          
          // NOTE: V - V_series = V_disk+V_plug+V_Schottky
          T V_other_than_series = pulse_voltage_RESET - (I_mem*(RseriesTiOx + R0 + V_series_coefficient*I_mem*I_mem));

          // NOTE: T gamma = gamma_coefficient*Eion
          T gamma = gamma_coefficient*V_other_than_series/lcell;

          T Treal = T0 + I_mem*V_other_than_series*Rth_positive;

          // NOTE: dWamin = dWa_f = dWa*(sqrt(1.0-pow(gamma,2.0))-(gamma*M_PI)/2+gamma*asin(gamma)) = dWa_mean - dWa_difference
          // NOTE: dWamax = dWa_r = dWa*(sqrt(1.0-pow(gamma,2.0))+(gamma*M_PI)/2+gamma*asin(gamma)) = dWa_mean + dWa_difference
          T dWa_mean = dWa*(sqrt(1.0-pow(gamma,2.0))+gamma*asin(gamma));
          T dWa_difference = dWa*((gamma*M_PI)/2.0);

          T denominator = PHYSICAL_PARAMETER_kb_over_e*Treal;

          T c_v0 = (Nplug+Ndisc_double)/2.0;
          T F1 = 1.0-pow((device_specific_Ndiscmin_cuda/Ndisc_double),10.0);
          T dNdt = -(c_v0*a_ny0*F1*(exp(-(dWa_mean - dWa_difference)/denominator)-exp(-(dWa_mean + dWa_difference)/denominator)))/device_specific_ldet_cuda;

          Ndisc_double = Ndisc_double + dNdt*base_time_step;
        }
        // TODO: BUG: applying these noise will result in PyTorch not receving the updated weights.
        // T ratio = Ndisc_double;
        // ratio = (Ndisc-ratio)/(Ndisc-Ndiscmin);
        // apply_cycle_to_cycle_noise(ratio, Ndiscmax, Ndiscmin, ldet, A, Ndiscmax_std, Ndiscmin_std, ldet_std, rdet_std, ldet_std_slope, rdet_std_slope, local_state,
        //                            Ndiscmax_upper_bound, Ndiscmax_lower_bound, Ndiscmin_upper_bound, Ndiscmin_lower_bound,
        //                            ldet_upper_bound, ldet_lower_bound, rdet_upper_bound, rdet_lower_bound);
        Ndisc_double = MAX(Ndisc_double, min_bound);
        w = map_Ndisc_to_weight(Ndisc_double, current_min, w_min, current_to_weight_ratio, g_read, h_read, j_0, k0, Ndiscmin);
        Ndisc = Ndisc_double;
      }
    }
    // TODO: BUG: Removing this delay or the print line will result in PyTorch not receving the updated weights.
    // printf("w after update %.20f\n", apparent_weight);
    uint32_t ns = 1;
    __nanosleep(ns);
  }
};

#define ARGS                                                                                       \
  (this->context_, this->x_size_, this->d_size_, m_batch, nK32, use_bo64, out_trans, up,           \
   par.getName())

template <typename T>
pwukpvec_t<T> JARTv1bRPUDeviceCuda<T>::getUpdateKernels(
    int m_batch, int nK32, int use_bo64, bool out_trans, const PulsedUpdateMetaParameter<T> &up) {

  pwukpvec_t<T> v;
  const auto &par = getPar();
  v.push_back(
      RPU::make_unique<PWUKernelParameterSingleFunctor<T, UpdateFunctorJARTv1b<T>, DEVICE_PARAMETER_COUNT>>
          ARGS);
  v.push_back(
      RPU::make_unique<PWUKernelParameterBatchFunctor<T, UpdateFunctorJARTv1b<T>, DEVICE_PARAMETER_COUNT>>
          ARGS);
  v.push_back(
      RPU::make_unique<PWUKernelParameterBatchSharedFunctor<T, UpdateFunctorJARTv1b<T>, DEVICE_PARAMETER_COUNT>>
          ARGS);

  return v;
}

#undef ARGS

template <typename T>
__global__ void kernelMapWeightToNdisc(
  T *weights, 
  T *Ndiscs, 
  int size, 
  T current_min, 
  T weight_min_bound,
  T weight_to_current_ratio,
  T g_read,
  T h_read,
  T j_0,
  T k0,
  T Ndiscmin) {

  RPU_CUDA_1D_KERNEL_LOOP(idx, size) {
    T current = (weights[idx]-weight_min_bound)*weight_to_current_ratio+current_min;
    Ndiscs[idx] = pow(((pow((g_read/current), k0)-1.0)/(h_read)),1.0/(-j_0))*Ndiscmin;
    }
}

template <typename T>
void map_weight_to_Ndisc(
  const CudaContext *context,
  T *w,
  T *Ndiscs,
  const int size,
  const T current_min, 
  const T weight_min_bound,
  const T weight_to_current_ratio,
  const T g_read,
  const T h_read,
  const T j_0,
  const T k0,
  const T Ndiscmin) {

  int nthreads = context->getNThreads();
  int nblocks = context->getNBlocks(size, nthreads);
  kernelMapWeightToNdisc<T><<<nblocks, nthreads, 0, context->getStream()>>>(w, Ndiscs, size, current_min, weight_min_bound, weight_to_current_ratio, g_read, h_read, j_0, k0, Ndiscmin);
}
template void map_weight_to_Ndisc<float>(const CudaContext *, float *, float *, const int, const float, const float, const float, const float, const float, const float, const float, const float);
#ifdef RPU_USE_DOUBLE
template void map_weight_to_Ndisc<double>(const CudaContext *, double *, double *, const int, const double, const double, const double, const double, const double, const double, const double, const double);
#endif

template <typename T>
void JARTv1bRPUDeviceCuda<T>::applyWeightUpdate(T *weights, T *dw_and_current_weight_out) {

  if (getPar().real_write_noise_std > 0) {
    RPU_FATAL("ApplyWeightUpdate is not supported with write_noise_std>0!");
  }
  RPU::math::elemaddcopysat<T>(
      this->context_, weights, dw_and_current_weight_out, this->size_,
      this->dev_4params_->getDataConst());
  
  const auto &par = getPar();
  T *Ndisc = get1ParamsData();

  map_weight_to_Ndisc<T>(
      this->context_, weights, Ndisc, this->size_,
      par.current_min, par.w_min, par.weight_to_current_ratio,
      par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);
}

// template <typename T>
// void JARTv1bRPUDeviceCuda<T>::decayWeights(T *weights, T alpha, bool bias_no_decay) {

//   RPU::math::elemscalealpha<T>(
//       this->context_, weights, bias_no_decay ? MAX(this->size_ - this->d_size_, 0) : this->size_,
//       this->dev_decay_scale_->getData(), this->dev_4params_->getData(), alpha,
//       this->dev_reset_bias_ != nullptr ? this->dev_reset_bias_->getData() : nullptr);
  
//   const auto &par = getPar();
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, weights, Ndisc, this->size_,
      // par.current_min, par.w_min, par.weight_to_current_ratio,
      // par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

// template <typename T> void JARTv1bRPUDeviceCuda<T>::decayWeights(T *weights, bool bias_no_decay) {

//   const auto &par = getPar();

//   RPU::math::elemscale<T>(
//       this->context_, weights, bias_no_decay ? MAX(this->size_ - this->d_size_, 0) : this->size_,
//       this->dev_decay_scale_->getData(), this->dev_4params_->getData(),
//       this->dev_reset_bias_ != nullptr ? this->dev_reset_bias_->getData() : nullptr);
  
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, weights, Ndisc, this->size_,
      // par.current_min, par.w_min, par.weight_to_current_ratio,
      // par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

// template <typename T>
// void JARTv1bRPUDeviceCuda<T>::driftWeights(T *weights, T time_since_last_call) {

//   PulsedRPUDeviceCudaBase<T>::driftWeights(weights, time_since_last_call);
//   this->wdrifter_cuda_->saturate(weights, this->dev_4params_->getData());
  
//   const auto &par = getPar();
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, weights, Ndisc, this->size_,
//       par.current_min, par.w_min, par.weight_to_current_ratio,
//       par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

// template <typename T> void JARTv1bRPUDeviceCuda<T>::diffuseWeights(T *weights) {

//   if (this->dev_diffusion_rate_ == nullptr) {
//     return; // no diffusion
//   }

//   if (this->dev_diffusion_nrnd_ == nullptr) {
//     this->initDiffusionRnd();
//     this->rnd_context_->randNormal(
//         this->dev_diffusion_nrnd_->getData(), this->dev_diffusion_nrnd_->getSize());
//   }
//   this->rnd_context_->synchronize();

//   RPU::math::elemasb02<T>(
//       this->context_, weights, this->size_, this->dev_diffusion_nrnd_->getData(),
//       this->dev_diffusion_rate_->getData(), this->dev_4params_->getData());

//   this->rnd_context_->recordWaitEvent(this->context_->getStream());
//   this->rnd_context_->randNormal(
//       this->dev_diffusion_nrnd_->getData(), this->dev_diffusion_nrnd_->getSize());

//   // Note: write noise will use the same rand to save memory. If
//   // diffusion + writenoise is often needed one might want to add an
//   // extra variable for the random numbers
  
//   const auto &par = getPar();
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, weights, Ndisc, this->size_,
      // par.current_min, par.w_min, par.weight_to_current_ratio,
      // par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

template <typename T> void JARTv1bRPUDeviceCuda<T>::clipWeights(T *weights, T clip) {

  RPU::math::elemsat<T>(this->context_, weights, this->size_, this->dev_4params_->getData());
  if (clip >= 0) {
    RPU::math::aclip<T>(this->context_, weights, this->size_, clip);
  }
  
  const auto &par = getPar();

  if (par.real_write_noise_std > 0) {
    // re-uses the diffusion rnd
    if (this->dev_diffusion_nrnd_ == nullptr) {
      this->initDiffusionRnd();
      this->rnd_context_->randNormal(
          this->dev_diffusion_nrnd_->getData(), this->dev_diffusion_nrnd_->getSize());
    }
    this->rnd_context_->synchronize();

    RPU::math::elemweightedsum<T>(
        this->context_, weights, this->size_, weights, (T)1.0,
        this->dev_diffusion_nrnd_->getData(), par.real_write_noise_std);

    this->rnd_context_->recordWaitEvent(this->context_->getStream());
    this->rnd_context_->randNormal(
        this->dev_diffusion_nrnd_->getData(), this->dev_diffusion_nrnd_->getSize());
  }
  T *Ndisc = get1ParamsData();

  map_weight_to_Ndisc<T>(
      this->context_, weights, Ndisc, this->size_,
      par.current_min, par.w_min, par.weight_to_current_ratio,
      par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

}

// template <typename T>
// void JARTv1bRPUDeviceCuda<T>::resetAt(T *dev_weights, const char *dev_non_zero_msk) {

//   const auto &par = getPar();

//   // if (par.usesPersistentWeight()) {
//   //   RPU_FATAL("ResetAt is not supported with write_noise_std>0!");
//   // }

//   RPU::math::elemresetsatmsk<T>(
//       this->context_, dev_weights, this->size_, dev_non_zero_msk,
//       this->dev_reset_bias_ == nullptr ? nullptr : this->dev_reset_bias_->getDataConst(), par.reset_std,
//       this->dev_4params_->getData());
  
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, dev_weights, Ndisc, this->size_,
      // par.current_min, par.w_min, par.weight_to_current_ratio,
      // par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

// template <typename T>
// void JARTv1bRPUDeviceCuda<T>::resetCols(T *weights, int start_col, int n_cols, T reset_prob) {
//   // col-major in CUDA.

//   if (this->dev_reset_bias_ == nullptr) {
//     return; // no reset
//   }

//   // if (getPar().usesPersistentWeight()) {
//   //   RPU_FATAL("ResetCols is not supported with write_noise_std>0!");
//   // }

//   if (this->dev_reset_nrnd_ == nullptr) {
//     PulsedRPUDeviceCuda<T>::initResetRnd();
//   }
//   int n = n_cols * this->d_size_;
//   int offset = start_col * this->d_size_;
//   this->rnd_context_->randNormal(
//       this->dev_reset_nrnd_->getData(), n_cols * this->d_size_, 0.0, getPar().reset_std);
//   if (reset_prob < 1) {
//     this->rnd_context_->randUniform(this->dev_reset_flag_->getData(), n_cols * this->d_size_);
//   }
//   this->context_->recordWaitEvent(this->rnd_context_->getStream());

//   if (n >= this->size_) {
//     // reset whole matrix
//     RPU::math::elemresetsat<T>(
//         this->context_, weights, this->size_, this->dev_reset_bias_->getDataConst(),
//         this->dev_reset_nrnd_->getDataConst(), this->dev_reset_flag_->getDataConst(), reset_prob,
//         this->dev_4params_->getData());

//   } else if (offset + n <= this->size_) {
//     // one pass enough
//     RPU::math::elemresetsat<T>(
//         this->context_, weights + offset, n, this->dev_reset_bias_->getDataConst() + offset,
//         this->dev_reset_nrnd_->getDataConst(), this->dev_reset_flag_->getDataConst(), reset_prob,
//         this->dev_4params_->getData() + 4 * offset);
//   } else {
//     // two passes
//     int m = this->size_ - offset;

//     RPU::math::elemresetsat<T>(
//         this->context_, weights + offset, m, this->dev_reset_bias_->getDataConst() + offset,
//         this->dev_reset_nrnd_->getDataConst(), this->dev_reset_flag_->getDataConst(), reset_prob,
//         this->dev_4params_->getData() + 4 * offset);

//     RPU::math::elemresetsat<T>(
//         this->context_, weights, n - m, this->dev_reset_bias_->getDataConst(),
//         this->dev_reset_nrnd_->getDataConst() + m, this->dev_reset_flag_->getDataConst() + m, reset_prob,
//         this->dev_4params_->getData());
//   }
  
//   const auto &par = getPar();
//   T *Ndisc = get1ParamsData();

//   map_weight_to_Ndisc<T>(
//       this->context_, weights, Ndisc, this->size_,
      // par.current_min, par.w_min, par.weight_to_current_ratio,
      // par.g_read, par.h_read, par.j_0, par.k0, par.Ndiscmin);

// }

template class JARTv1bRPUDeviceCuda<float>;
#ifdef RPU_USE_DOUBLE
template class JARTv1bRPUDeviceCuda<double>;
#endif

} // namespace RPU
