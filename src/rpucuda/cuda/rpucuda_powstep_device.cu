#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020, 2021, 2022 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "pwu_kernel_parameter.h"
#include "rpu_pulsed_meta_parameter.h"
#include "rpucuda_powstep_device.h"
#include <memory>

namespace RPU {

template <typename T> struct UpdateFunctorPowStep {

  __device__ __forceinline__ void operator()(
      T &apparent_weight,
      uint32_t n,
      uint32_t negative,
      const float4 par_4,         // min_bound, scale_down, max_bound, scale_up,
      const float2 gamma_down_up, // gamma_down, gamma_up
      T &persistent_weight,
      const T *write_noise_std,
      const int global_params_count,
      T noise_std_dw,
      hiprandState &local_state) {

    UNUSED(global_params_count); // fixed

    T wmin = par_4.x; // [0]
    T wmax = par_4.z; // [2]
    T range = wmax - wmin;
    if (range == 0) {
      return;
    }
    T uw_std = *write_noise_std;
    T &w = uw_std > 0 ? persistent_weight : apparent_weight;
    // negative > 0 means sign < 0 and thus up-direction
    T scale = (negative > 0) ? (par_4.w) : (-par_4.y);                //[3] (up), [1] (down)
    T gamma = (negative > 0) ? (gamma_down_up.y) : (gamma_down_up.x); // [1] (up), [0] (down)

    // up direction: ((wmax - w) / range) ^ gamma
    // down direction:  ((w - wmin) / range) ^ gamma  == (1 - (wmax - w)/range) ^ gamma

    // n is larger 0 in any case
    if (n == 1) {
      T x = (wmax - w) / range;
      T dw = scale * ((negative > 0) ? __powf(x, gamma) : __powf((T)1.0 - x, gamma));

      if (noise_std_dw > 0) {
        T stoch_value = hiprand_normal(&local_state);
        stoch_value *= noise_std_dw;
        w += dw * ((T)1.0 + stoch_value);
      } else {
        w += dw;
      }
      w = (w > wmax) ? wmax : w;
      w = (w < wmin) ? wmin : w;

    } else {
      if (noise_std_dw > 0) {
        for (int i_updates = 0; i_updates < n; i_updates++) {
          T stoch_value = hiprand_normal(&local_state);
          stoch_value *= noise_std_dw;
          T x = (wmax - w) / range;
          T dw = scale * ((negative > 0) ? __powf(x, gamma) : __powf((T)1.0 - x, gamma));
          w += dw * ((T)1.0 + stoch_value);
          // better always check both bounds
          w = (w > wmax) ? wmax : w;
          w = (w < wmin) ? wmin : w;
        }
      } else {
        for (int i_updates = 0; i_updates < n; i_updates++) {
          T x = (wmax - w) / range;
          w += scale * ((negative > 0) ? __powf(x, gamma) : __powf((T)1.0 - x, gamma));
          // better always check both bounds
          w = (w > wmax) ? wmax : w;
          w = (w < wmin) ? wmin : w;
        }
      }
    }

    // add update write noise onto apparent weight
    if (uw_std > 0) {
      T stoch_value = hiprand_normal(&local_state);
      apparent_weight = persistent_weight + uw_std * stoch_value;
    }
  }
};

#define ARGS                                                                                       \
  (this->context_, this->x_size_, this->d_size_, m_batch, nK32, use_bo64, out_trans, up,           \
   par.getName())

template <typename T>
pwukpvec_t<T> PowStepRPUDeviceCuda<T>::getUpdateKernels(
    int m_batch, int nK32, int use_bo64, bool out_trans, const PulsedUpdateMetaParameter<T> &up) {

  pwukpvec_t<T> v;
  const auto &par = getPar();
  v.push_back(
      RPU::make_unique<PWUKernelParameterSingleFunctor<T, UpdateFunctorPowStep<T>, 1>> ARGS);
  v.push_back(RPU::make_unique<PWUKernelParameterBatchFunctor<T, UpdateFunctorPowStep<T>, 1>> ARGS);
  v.push_back(
      RPU::make_unique<PWUKernelParameterBatchSharedFunctor<T, UpdateFunctorPowStep<T>, 1>> ARGS);
  v.push_back(
      RPU::make_unique<
          PWUKernelParameterBatchSharedWeightOutputFunctor<T, UpdateFunctorPowStep<T>, 1>> ARGS);
  return v;
}

#undef ARGS

template class PowStepRPUDeviceCuda<float>;
#ifdef RPU_USE_DOUBLE
template class PowStepRPUDeviceCuda<double>;
#endif

} // namespace RPU
