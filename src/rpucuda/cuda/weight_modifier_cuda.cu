#include "hip/hip_runtime.h"
/**
 * (C) Copyright 2020 IBM. All Rights Reserved.
 *
 * This code is licensed under the Apache License, Version 2.0. You may
 * obtain a copy of this license in the LICENSE.txt file in the root directory
 * of this source tree or at http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Any modifications or derivative works of this code must retain this
 * copyright notice, and modified files need to carry a notice indicating
 * that they have been altered from the originals.
 */

#include "cuda_math_util.h"
#include "weight_modifier_cuda.h"

namespace RPU {

#define RPU_WM_KERNEL_LOOP(STOCH_IF, BODY)                                                         \
  int tid = blockDim.x * blockIdx.x + threadIdx.x;                                                 \
  int total_threads = blockDim.x * gridDim.x;                                                      \
  int size = size_in;                                                                              \
  const bool stoch_if = STOCH_IF;                                                                  \
                                                                                                   \
  hiprandState local_state;                                                                         \
  if (stoch_if && tid < size) {                                                                    \
    local_state = random_states[tid];                                                              \
  }                                                                                                \
                                                                                                   \
  for (int i_stride = 0; i_stride < size; i_stride += total_threads) {                             \
    int i = i_stride + tid;                                                                        \
    if (i < size) {                                                                                \
      {                                                                                            \
        BODY;                                                                                      \
      }                                                                                            \
    }                                                                                              \
  }                                                                                                \
                                                                                                   \
  if (stoch_if && tid < size) {                                                                    \
    random_states[tid] = local_state;                                                              \
  }

template <typename T>
__global__ void kernelModifyWeightsDiscretize(
    int size_in,
    T *new_weights,
    const T *weights,
    const T res_in, // need to larger than zero!!
    const bool sto_round,
    const float assumed_wmax,
    const float *wmax,
    hiprandState_t *random_states) {
  const T res = res_in;
  T amax = (wmax) ? (*wmax) : assumed_wmax;
  amax = amax > 0.0 ? amax : (T)1.0;

  RPU_WM_KERNEL_LOOP(
      sto_round,

      T value = weights[i] / amax;
      value /= res; if (stoch_if) {
        T stoch_value = hiprand_uniform(&local_state);
        value += stoch_value - 0.5;
      } new_weights[i] = amax * res * round(value););
}

template <typename T>
__global__ void kernelModifyWeightsDoReFa(
    int size_in,
    T *new_weights,
    const T *weights,
    const T res_in, // need to larger than zero!!
    const bool sto_round,
    const T dorefa_clip,
    float assumed_wmax,
    float *wmax,
    hiprandState_t *random_states) {
  T amax = (wmax) ? (*wmax) : assumed_wmax;
  amax = amax > 0.0 ? amax : (T)1.0;

  const T res = res_in;
  const T scale = fabs(dorefa_clip / tanhf(amax));

  RPU_WM_KERNEL_LOOP(
      sto_round,

      T value = weights[i];
      value = tanhf(value) * scale;

      value /= res; if (stoch_if) {
        T stoch_value = hiprand_uniform(&local_state);
        value += stoch_value - 0.5;
      } new_weights[i] = res * round(value););
}

template <typename T>
__global__ void kernelModifyWeightsDiscretizeAddNormal(
    int size_in,
    T *new_weights,
    const T *weights,
    const T res_in, // need to larger than zero!!
    const bool sto_round_in,
    const T stddev_in,
    const float assumed_wmax,
    const float *wmax,
    hiprandState_t *random_states) {
  const T res = res_in;
  const T stddev = stddev_in;
  const bool sto_round = sto_round_in;
  T amax = (wmax) ? (*wmax) : assumed_wmax;
  amax = amax > 0.0 ? amax : (T)1.0;

  RPU_WM_KERNEL_LOOP(
      true,

      T value = weights[i] / amax;
      value /= res;
      if (sto_round) { value += hiprand_uniform(&local_state) - 0.5; } value = res * round(value);
      T stoch_value = hiprand_normal(&local_state);
      new_weights[i] = amax * (value + stddev * stoch_value););
}

template <typename T>
__global__ void kernelModifyWeightsAddNormal(
    int size_in,
    T *new_weights,
    const T *weights,
    const T stddev_in,
    const float assumed_wmax,
    const float *wmax,
    hiprandState_t *random_states) {
  T amax = (wmax) ? (*wmax) : assumed_wmax;
  amax = amax > 0.0 ? amax : (T)1.0;

  const T stddev = amax * stddev_in;

  RPU_WM_KERNEL_LOOP(true,

                     T stoch_value = hiprand_normal(&local_state);
                     new_weights[i] = weights[i] + stddev * stoch_value;);
}

template <typename T>
__global__ void kernelModifyWeightsMultNormal(
    int size_in,
    T *new_weights,
    const T *weights,
    const T stddev_in,
    const float assumed_wmax,
    const float *wmax,
    hiprandState_t *random_states) {
  T amax = (wmax) ? (*wmax) : assumed_wmax;
  amax = amax > 0.0 ? amax : (T)1.0;

  const T stddev = stddev_in * amax;

  RPU_WM_KERNEL_LOOP(true,

                     T w = weights[i];
                     T stoch_value = hiprand_normal(&local_state);

                     new_weights[i] = w * (1 + stddev * stoch_value););
}

template <typename T>
__global__ void kernelModifyWeightsDropConnections(
    int size_in, T *new_weights, const T prob_in, hiprandState_t *random_states) {
  const T prob = prob_in;

  RPU_WM_KERNEL_LOOP(
      true,

      T stoch_value = hiprand_uniform(&local_state);
      if (stoch_value < prob) { new_weights[i] = (T)0.0; });
}

// ctor
template <typename T>
WeightModifierCuda<T>::WeightModifierCuda(CudaContext *context, int x_size, int d_size)
    : context_(context), x_size_(x_size), d_size_(d_size), size_(x_size * d_size),
      enable_during_test_(false) {}

template <typename T>
void WeightModifierCuda<T>::apply(
    T *new_weights, const T *weights, const WeightModifierParameter &wmpar) {

  int nthreads = context_->getNThreads();
  auto s = context_->getStream();
  int nblocks = context_->getNStrideBlocks(size_, nthreads);

  bool done = false;
  enable_during_test_ = wmpar.enable_during_test;

  float *amax = nullptr;
  if (wmpar.rel_to_actual_wmax && wmpar.type != WeightModifierType::Copy) {
    if (!amaximizer_) {
      amaximizer_ = RPU::make_unique<Maximizer<T>>(context_, size_, true);
    }
    amaximizer_->compute(weights, 1, false);
    amax = amaximizer_->getMaxValues();
  }

  // note: all methods need to work in
  switch (wmpar.type) {
  case WeightModifierType::Copy: {

    if (new_weights == weights) {
      RPU_FATAL("cannot use WeightModifierType::Copy with in-place weights.");
    }
    // copies below
    break; // maybe dropping below though
  }

  case WeightModifierType::Discretize: {

    if (wmpar.res > 0) {

      kernelModifyWeightsDiscretize<T><<<nblocks, nthreads, 0, s>>>(
          size_, new_weights, weights, wmpar.res, wmpar.sto_round, wmpar.assumed_wmax, amax,
          wmpar.sto_round ? context_->getRandomStates(nblocks * nthreads) : nullptr);
      done = true;
    }
    break;
  }
  case WeightModifierType::DoReFa: {
    if (wmpar.res > 0) {

      kernelModifyWeightsDoReFa<T><<<nblocks, nthreads, 0, s>>>(
          size_, new_weights, weights, wmpar.res, wmpar.sto_round, wmpar.dorefa_clip,
          wmpar.assumed_wmax, amax,
          wmpar.sto_round ? context_->getRandomStates(nblocks * nthreads) : nullptr);
      done = true;
    }
    break;
  }

  case WeightModifierType::MultNormal: {
    if (wmpar.std_dev > 0) {

      kernelModifyWeightsMultNormal<T><<<nblocks, nthreads, 0, s>>>(
          size_, new_weights, weights, wmpar.std_dev, wmpar.assumed_wmax, amax,
          context_->getRandomStates(nblocks * nthreads));
      done = true;
    }
    break;
  }
  case WeightModifierType::AddNormal: {
    if (wmpar.std_dev > 0) {

      kernelModifyWeightsAddNormal<T><<<nblocks, nthreads, 0, s>>>(
          size_, new_weights, weights, wmpar.std_dev, wmpar.assumed_wmax, amax,
          context_->getRandomStates(nblocks * nthreads));
      done = true;
    }
    break;
  }
  case WeightModifierType::DiscretizeAddNormal: {
    if (wmpar.res > 0 || wmpar.std_dev > 0) {

      kernelModifyWeightsDiscretizeAddNormal<T><<<nblocks, nthreads, 0, s>>>(
          size_, new_weights, weights, wmpar.res, wmpar.sto_round, wmpar.std_dev,
          wmpar.assumed_wmax, amax, context_->getRandomStates(nblocks * nthreads));
      done = true;
    }
    break;
  }

  default:
    RPU_FATAL("Requested WeightModifierType not implemented.");
  }

  // need to copy in case some parameters were set to 0
  if (!done && new_weights != weights) {
    RPU::math::copy<T>(context_, size_, weights, 1, new_weights, 1);
  }

  if (wmpar.pdrop > 0.0) {

    if (new_weights == weights) {
      RPU_FATAL("cannot use pdrop>0 with in-place weights.");
    }

    kernelModifyWeightsDropConnections<T><<<nblocks, nthreads, 0, s>>>(
        size_, new_weights, wmpar.pdrop, context_->getRandomStates(nblocks * nthreads));
  }
}

template class WeightModifierCuda<float>;
#ifdef RPU_USE_DOUBLE
template class WeightModifierCuda<double>;
#endif

#undef RPU_WM_KERNEL_LOOP
} // namespace RPU
